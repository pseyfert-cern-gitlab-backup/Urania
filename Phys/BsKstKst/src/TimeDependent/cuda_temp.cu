__device__ double KpiKpiSpectrumNW::P_trueBs(iev) {

  set_buffer_integral_vars();
  double yield_Bs = (1.+DCP_prod)*int_Bs(iev);
  double yield_Bsbar = (1.-DCP_prod)*int_Bsbar(iev);

  return yield_Bs/(yield_Bs+yield_Bsbar);

 }

 void KpiKpiSpectrumNW::Randomize6D() {

   if (wide_window == 1) {
      m1_ran = 750.+ran.Rndm()*(1600.-750.);
      m2_ran = 750.+ran.Rndm()*(1600.-750.);
      }
   else {
      m1_ran = 750.+ran.Rndm()*(1050.-750.);
      m2_ran = 750.+ran.Rndm()*(1050.-750.);
      }
   cos1_ran = -1.+ran.Rndm()*2.;
   cos2_ran = -1.+ran.Rndm()*2.;
   phi_ran = ran.Rndm()*2.*pi;
   t_ran = ran.Rndm()*12.;

   return;

 }

void KpiKpiSpectrumNW::SetGenerator(int compute_max_fun, int sample_size_7D) {

  // Setting of the mKpi generation range.
  Wide_Window_Gen = wide_window;

  // Randomization of the random number generator seed.
  ran.SetSeed(0);

  // Variable inicialization.
  dec_flavour = 0.;
  dec_SSK_tagged = 0.;
  dec_OS_tagged = 0.;
  dec_accepted = 0.;
  N_accepted = 0;

  // Computation of the maximum value for the considered distributions, if requested.
  if (compute_max_fun == 1) {

    cout << "Computing p.d.f.'s maxima.\n";

    max_fun_deltat = 0.;
    max_fun_etaSSK = 0.;
    max_fun_etaOS = 0.;
    max_fun_6DBs = 0.;
    max_fun_6DBsbar = 0.;

    // Decay time error distribution.
    for (int i=0; i<10000; i++) {
      t_err_ran = 0.01+ran.Rndm()*(0.04-0.01);
      fun_ran = P_deltat(t_err_ran);
      if (fun_ran > max_fun_deltat) {max_fun_deltat = fun_ran;}
      }

    // Mistag probability distributions.
    for (int i=0; i<10000; i++) {
      etamistag_SSK_ran = 0.3+ran.Rndm()*(0.5-0.3);
      fun_ran = P_eta_SSK(etamistag_SSK_ran);
      if (fun_ran > max_fun_etaSSK) {max_fun_etaSSK = fun_ran;}
      etamistag_OS_ran = 0.3+ran.Rndm()*(0.5-0.3);
      fun_ran = P_eta_OS(etamistag_OS_ran);
      if (fun_ran > max_fun_etaOS) {max_fun_etaOS = fun_ran;}
      }

    // Decay variable distributions.
    cout << "(Using a randomized sample of " << sample_size_7D << " 7D points to find the maximum of the Bs and Bs-bar PDFs).\n";
    for (int i=0; i<sample_size_7D; i++) {
      Randomize7D_fun_max();
      set_buffer_differential_vars(m1_ran,m2_ran,cos1_ran,cos2_ran,phi_ran,t_ran,t_err_ran,0,0,0.5,0.5);
      fun_ran = fun_Bs()*accGen(t_ran,m1_ran,m2_ran,cos1_ran,cos2_ran,phi_ran);
      if (fun_ran > max_fun_6DBs) {max_fun_6DBs = fun_ran;}
      Randomize7D_fun_max();
      set_buffer_differential_vars(m1_ran,m2_ran,cos1_ran,cos2_ran,phi_ran,t_ran,t_err_ran,0,0,0.5,0.5);
      fun_ran = fun_Bsbar()*accGen(t_ran,m1_ran,m2_ran,cos1_ran,cos2_ran,phi_ran);
      if (fun_ran > max_fun_6DBsbar) {max_fun_6DBsbar = fun_ran;}
      }

    // Providing the computed maxima with an extra safety range.
    max_fun_deltat *= 1.2;
    max_fun_etaSSK *= 1.2;
    max_fun_etaOS *= 1.2;
    max_fun_6DBs *= 1.2;
    max_fun_6DBsbar *= 1.2;

    cout << "max_fun_deltat = " << max_fun_deltat << "\n" << "max_fun_etaSSK = " << max_fun_etaSSK << "\n" << "max_fun_etaOS = " << max_fun_etaOS << "\n" << "max_fun_6DBs = " << max_fun_6DBs << "\n" << "max_fun_6DBsbar = " << max_fun_6DBsbar << "\n";

    }

  else {

    if (Wide_Window_Gen == 0) {
      cout << "Using previously obtained values for the maxima of the PDFs in the narrow mKpi window.\n";
      max_fun_deltat = 68.8064;
      max_fun_etaSSK = 0.276388;
      max_fun_etaOS = 1.03582;
      max_fun_6DBs = 202.432;
      max_fun_6DBsbar = 194.987;
      }

    else {
      cout << "Using previously obtained values for the maxima of the PDFs in the wide mKpi window.\n";
      max_fun_deltat = 68.8064;
      max_fun_etaSSK = 0.276483;
      max_fun_etaOS = 1.03582;
      max_fun_6DBs = 760.726;
      max_fun_6DBsbar = 790.697;
      }

    }

  return;

 }

int KpiKpiSpectrumNW::getGenerator(const RooArgSet& directVars, RooArgSet &generateVars, Bool_t /*staticInitOK*/) {

  // 11-D generation, corresponding to the full set of observables.
  if ( matchArgs(directVars,generateVars,m1,m2,cos1,cos2) && matchArgs(directVars,generateVars,phi,t,decision_SSK,decision_OS) && matchArgs(directVars,generateVars,etamistag_SSK,etamistag_OS,t_err) ) return 1 ;

  return 0 ;

 }

void KpiKpiSpectrumNW::generateEvent(int code)
{

  assert(code==1);

  // Generation is performed in four steps:
  //    1. The per event decay time error is determined.
  //    2. A flavour, Bs or Bsbar, is assigned to the event.
  //    3. Tagging decisions and mistag probabilities are obtained.
  //    4. Decay variables (angles, invariant masses and decay time) are determined.

  // Determination of the per event decay time error.
  event_accepted = 0;
  max_fun = max_fun_deltat;
  while (event_accepted == 0) {
    t_err_ran = ran.Rndm()*0.1;
    fun_ran = P_deltat(t_err_ran);
    dec_accepted = max_fun*ran.Rndm();
    if (fun_ran > dec_accepted) {event_accepted = 1;}
  }
  t_err = t_err_ran;

  // Generation of a Bs or a Bsbar event.
  dec_flavour = ran.Rndm();
  if (dec_flavour < P_trueBs()) {true_ID = 1;} // Bs-like event.
  else {true_ID = -1;} // Bsbar-like event.

  // Determination of the SSK mistag probability.
  dec_SSK_tagged = ran.Rndm();
  if (dec_SSK_tagged < tag_eff_SSK) {
    // If the event is tagged, the accept-reject method is used to generate eta.
    event_accepted = 0;
    max_fun = max_fun_etaSSK;
    while (event_accepted == 0) {
      etamistag_SSK_ran = 0.5*ran.Rndm();
      fun_ran = P_eta_SSK(etamistag_SSK_ran);
      dec_accepted = max_fun*ran.Rndm();
      if (fun_ran > dec_accepted) {event_accepted = 1;}
      }
    etamistag_SSK = etamistag_SSK_ran;
    }
  else {
    // If the event is not tagged, eta is set to 0.5.
    etamistag_SSK = 0.5;
    }

  // Determination of the SSK tagging decision.
  if (etamistag_SSK < 0.5) {
    dec_right_tagged = ran.Rndm();
    if (true_ID == 1) {
      prob_right_tagged = 1.-omega_SSK(etamistag_SSK);
      if (dec_right_tagged < prob_right_tagged) {decision_SSK = 1;} // Right tagged Bs.
      else {decision_SSK = -1;} // Wrong tagged Bs.
      }
    else {
      prob_right_tagged = 1.-omegabar_SSK(etamistag_SSK);
      if (dec_right_tagged < prob_right_tagged) {decision_SSK = -1;} // Right tagged Bsbar.
      else {decision_SSK = 1;} // Wrong tagged Bsbar.
      }
    }
  else {
    decision_SSK = 0;
    }

  // Determination of the OS mistag probability.
  dec_OS_tagged = ran.Rndm();
  if (dec_OS_tagged < tag_eff_OS) {
    // If the event is tagged, the accept-reject method is used to generate eta.
    event_accepted = 0;
    max_fun = max_fun_etaOS;
    while (event_accepted == 0) {
      etamistag_OS_ran = 0.5*ran.Rndm();
      fun_ran = P_eta_OS(etamistag_OS_ran);
      dec_accepted = max_fun*ran.Rndm();
      if (fun_ran > dec_accepted) {event_accepted = 1;}
      }
    etamistag_OS = etamistag_OS_ran;
    }
  else {
    // If the event is not tagged, eta is set to 0.5.
    etamistag_OS = 0.5;
    }

  // Determination of the OS tagging decision.
  if (etamistag_OS < 0.5) {
    dec_right_tagged = ran.Rndm();
    if (true_ID == 1) {
      prob_right_tagged = 1.-omega_OS(etamistag_OS);
      if (dec_right_tagged < prob_right_tagged) {decision_OS = 1;} // Right tagged Bs.
      else {decision_OS = -1;} // Wrong tagged Bs.
      }
    else {
      prob_right_tagged = 1.-omegabar_OS(etamistag_OS);
      if (dec_right_tagged < prob_right_tagged) {decision_OS = -1;} // Right tagged Bsbar.
      else {decision_OS = 1;} // Wrong tagged Bsbar.
      }
    }
  else {
    decision_OS = 0;
    }

  // Determination of the decay observables, using the accept-reject method in 6-D, taking t_err as a conditional variable.
  event_accepted = 0;
  if (true_ID == 1) {max_fun = max_fun_6DBs;}
  else {max_fun = max_fun_6DBsbar;}
  while (event_accepted == 0) {
    Randomize6D(Wide_Window_Gen);
    set_buffer_differential_vars(m1_ran,m2_ran,cos1_ran,cos2_ran,phi_ran,t_ran,t_err,0,0,0.5,0.5);
    dec_accepted = max_fun*ran.Rndm();
    if (true_ID == 1) {
      fun_ran = fun_Bs()*accGen(t_ran,m1_ran,m2_ran,cos1_ran,cos2_ran,phi_ran);
      }
    else {
      fun_ran = fun_Bsbar()*accGen(t_ran,m1_ran,m2_ran,cos1_ran,cos2_ran,phi_ran);
      }
    if (fun_ran > dec_accepted) {event_accepted = 1;}
    }
  m1 = m1_ran;
  m2 = m2_ran;
  cos1 = cos1_ran;
  cos2 = cos2_ran;
  phi = phi_ran;
  t = t_ran;

  N_accepted += 1;
  if (N_accepted%100 == 0) {cout << N_accepted << " events generated\n";}

  return;

 }
