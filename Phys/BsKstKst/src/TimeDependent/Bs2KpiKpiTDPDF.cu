#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <pycuda-complex.hpp>
#include<hiprand.h>
#include<hiprand/hiprand_kernel.h>

extern "C" {

// ##########################################
// Global variables

#define pi 3.14159265359
#define MPion 139.57018
#define MKaon 493.667
#define MKst_1_1410 1414.
#define GKst_1_1410 232.
#define MKst_1_1680 1717.
#define GKst_1_1680 322.
#define MBs 5366.77

__device__ double DCP_prod;

__device__ double MEta = 547.;
__device__ double sAdler_Stheo = 0.236;
__device__ double B0_Stheo = 0.411;
__device__ double B1_Stheo = 0.162;
__device__ double alpha_Stheo = 1.15;
__device__ double s0_Stheo = 1.21;
__device__ double phi0_Stheo = -0.19;
__device__ double phi1_Stheo = 5.03;
__device__ double a_Stheo = -5.2;
__device__ double b_Stheo = 7.6;
__device__ double c_Stheo = -1.73;
__device__ double sqrtsr1_Stheo = 1.401;
__device__ double sqrtsr2_Stheo = 1.817;
__device__ double e1_Stheo = 1.;
__device__ double e2_Stheo = 0.184;
__device__ double G1_Stheo = 0.497;
__device__ double G2_Stheo = 0.28;

__device__ double s_Kpi_palano = (139.57018*139.57018+493.667*493.667)/1000./1000.;
__device__ double s_A_palano = 0.87753*(139.57018*139.57018+493.667*493.667)/1000./1000.;
__device__ double s_a_palano = 1.7991;
__device__ double g_1_a_palano = 0.3139;
__device__ double g_2_a_palano = -0.00775;
__device__ double s_b_palano = 8.3627;
__device__ double g_1_b_palano = 1.1804;
__device__ double g_2_b_palano = -0.22335;
__device__ double C_11_0_palano = -0.1553;
__device__ double C_11_1_palano = 0.0909;
__device__ double C_11_2_palano = 0.8618;
__device__ double C_11_3_palano = 0.0629;
__device__ double C_12_0_palano = 0.0738;
__device__ double C_12_1_palano = 0.3866;
__device__ double C_12_2_palano = 1.2195;
__device__ double C_12_3_palano = 0.8390;
__device__ double C_22_0_palano = -0.0036;
__device__ double C_22_1_palano = 0.2590;
__device__ double C_22_2_palano = 1.6950;
__device__ double C_22_3_palano = 2.2300;
__device__ double A_1_0_palano = 1.;
__device__ double A_1_1_palano = 0.00491636810678;
__device__ double A_1_2_palano = 2.12489529189;
__device__ double A_1_3_palano = 0.56004179484;
__device__ double A_1_4_palano = 0.;
__device__ double A_2_0_palano = -4.20943829183;
__device__ double A_2_1_palano = -1.2110147687;
__device__ double A_2_2_palano = 2.28474898994;
__device__ double A_2_3_palano = 5.93332582489;
__device__ double A_2_4_palano = 0.;


// ##########################################
// Auxiliar elements

__device__ double fjjphhpindexdict[3][3][5][5];
__device__ double ghhpindexdict[5][5];
__device__ double reNj1j2hdict[3][3][5];
__device__ double imNj1j2hdict[3][3][5];
__device__ double spl_knot_vector[2][2][6];

__device__ pycuda::complex<double> Nj1j2hdict(int j1, int j2, int h) {

   return pycuda::complex<double>(reNj1j2hdict[j1][j2][h],imNj1j2hdict[j1][j2][h]);

 }


// ##########################################
// Initializer

__global__ void initialize() {

   fjjphhpindexdict[0][0][0][0] = 1;
   fjjphhpindexdict[0][1][0][0] = 2;
   fjjphhpindexdict[0][1][0][1] = 3;
   fjjphhpindexdict[0][1][0][2] = 3;
   fjjphhpindexdict[0][2][0][0] = 4;
   fjjphhpindexdict[0][2][0][1] = 5;
   fjjphhpindexdict[0][2][0][2] = 5;
   fjjphhpindexdict[1][0][0][0] = 2;
   fjjphhpindexdict[1][1][0][0] = 6;
   fjjphhpindexdict[1][1][0][1] = 5;
   fjjphhpindexdict[1][1][0][2] = 5;
   fjjphhpindexdict[1][2][0][0] = 7;
   fjjphhpindexdict[1][2][0][1] = 8;
   fjjphhpindexdict[1][2][0][2] = 8;
   fjjphhpindexdict[1][0][1][0] = 3;
   fjjphhpindexdict[1][0][2][0] = 3;
   fjjphhpindexdict[1][1][1][0] = 5;
   fjjphhpindexdict[1][1][2][0] = 5;
   fjjphhpindexdict[1][1][1][1] = 9;
   fjjphhpindexdict[1][1][1][2] = 9;
   fjjphhpindexdict[1][1][2][1] = 9;
   fjjphhpindexdict[1][1][2][2] = 9;
   fjjphhpindexdict[1][2][1][0] = 10;
   fjjphhpindexdict[1][2][2][0] = 10;
   fjjphhpindexdict[1][2][1][1] = 11;
   fjjphhpindexdict[1][2][1][2] = 11;
   fjjphhpindexdict[1][2][2][1] = 11;
   fjjphhpindexdict[1][2][2][2] = 11;
   fjjphhpindexdict[2][0][0][0] = 4;
   fjjphhpindexdict[2][1][0][0] = 7;
   fjjphhpindexdict[2][1][0][1] = 10;
   fjjphhpindexdict[2][1][0][2] = 10;
   fjjphhpindexdict[2][2][0][0] = 12;
   fjjphhpindexdict[2][2][0][1] = 13;
   fjjphhpindexdict[2][2][0][2] = 13;
   fjjphhpindexdict[2][0][1][0] = 5;
   fjjphhpindexdict[2][0][2][0] = 5;
   fjjphhpindexdict[2][1][1][0] = 8;
   fjjphhpindexdict[2][1][2][0] = 8;
   fjjphhpindexdict[2][1][1][1] = 11;
   fjjphhpindexdict[2][1][1][2] = 11;
   fjjphhpindexdict[2][1][2][1] = 11;
   fjjphhpindexdict[2][1][2][2] = 11;
   fjjphhpindexdict[2][2][1][0] = 13;
   fjjphhpindexdict[2][2][2][0] = 13;
   fjjphhpindexdict[2][2][1][1] = 14;
   fjjphhpindexdict[2][2][1][2] = 14;
   fjjphhpindexdict[2][2][2][1] = 14;
   fjjphhpindexdict[2][2][2][2] = 14;
   fjjphhpindexdict[0][2][0][3] = 9;
   fjjphhpindexdict[0][2][0][4] = 9;
   fjjphhpindexdict[1][2][0][3] = 11;
   fjjphhpindexdict[1][2][0][4] = 11;
   fjjphhpindexdict[1][2][1][3] = 15;
   fjjphhpindexdict[1][2][1][4] = 15;
   fjjphhpindexdict[1][2][2][3] = 15;
   fjjphhpindexdict[1][2][2][4] = 15;
   fjjphhpindexdict[2][2][0][3] = 16;
   fjjphhpindexdict[2][2][0][4] = 16;
   fjjphhpindexdict[2][2][1][3] = 17;
   fjjphhpindexdict[2][2][1][4] = 17;
   fjjphhpindexdict[2][2][2][3] = 17;
   fjjphhpindexdict[2][2][2][4] = 17;
   fjjphhpindexdict[2][0][3][0] = 9;
   fjjphhpindexdict[2][0][4][0] = 9;
   fjjphhpindexdict[2][1][3][0] = 11;
   fjjphhpindexdict[2][1][4][0] = 11;
   fjjphhpindexdict[2][1][3][1] = 15;
   fjjphhpindexdict[2][1][3][2] = 15;
   fjjphhpindexdict[2][1][4][1] = 15;
   fjjphhpindexdict[2][1][4][2] = 15;
   fjjphhpindexdict[2][2][3][0] = 16;
   fjjphhpindexdict[2][2][4][0] = 16;
   fjjphhpindexdict[2][2][3][1] = 17;
   fjjphhpindexdict[2][2][3][2] = 17;
   fjjphhpindexdict[2][2][4][1] = 17;
   fjjphhpindexdict[2][2][4][2] = 17;
   fjjphhpindexdict[2][2][3][3] = 18;
   fjjphhpindexdict[2][2][3][4] = 18;
   fjjphhpindexdict[2][2][4][3] = 18;
   fjjphhpindexdict[2][2][4][4] = 18;

   ghhpindexdict[0][0] = 1;
   ghhpindexdict[0][1] = 2;
   ghhpindexdict[0][2] = 3;
   ghhpindexdict[1][0] = 2;
   ghhpindexdict[1][1] = 4;
   ghhpindexdict[1][2] = 5;
   ghhpindexdict[2][0] = 3;
   ghhpindexdict[2][1] = 5;
   ghhpindexdict[2][2] = 6;
   ghhpindexdict[0][3] = 7;
   ghhpindexdict[0][4] = 8;
   ghhpindexdict[1][3] = 9;
   ghhpindexdict[1][4] = 10;
   ghhpindexdict[2][3] = 11;
   ghhpindexdict[2][4] = 12;
   ghhpindexdict[3][0] = 7;
   ghhpindexdict[3][1] = 9;
   ghhpindexdict[3][2] = 11;
   ghhpindexdict[3][3] = 13;
   ghhpindexdict[3][4] = 14;
   ghhpindexdict[4][0] = 8;
   ghhpindexdict[4][1] = 10;
   ghhpindexdict[4][2] = 12;
   ghhpindexdict[4][3] = 14;
   ghhpindexdict[4][4] = 15;

   reNj1j2hdict[0][0][0] = pycuda::real(pycuda::complex<double>(1./(2.*sqrt(2.*pi)),0.));
   reNj1j2hdict[0][1][0] = pycuda::real(pycuda::complex<double>(-sqrt(3.)/(2.*sqrt(2.*pi)),0.));
   reNj1j2hdict[0][2][0] = pycuda::real(pycuda::complex<double>(sqrt(5.)/(4.*sqrt(2.*pi)),0.));
   reNj1j2hdict[1][0][0] = pycuda::real(pycuda::complex<double>(sqrt(3.)/(2.*sqrt(2.*pi)),0.));
   reNj1j2hdict[1][1][0] = pycuda::real(pycuda::complex<double>(-3./(2.*sqrt(2.*pi)),0.));
   reNj1j2hdict[1][1][1] = pycuda::real(pycuda::complex<double>(-3./(4.*sqrt(pi)),0.));
   reNj1j2hdict[1][1][2] = pycuda::real(pycuda::complex<double>(0.,-3./(4.*sqrt(pi))));
   reNj1j2hdict[1][2][0] = pycuda::real(pycuda::complex<double>(sqrt(15.)/(4.*sqrt(2.*pi)),0.));
   reNj1j2hdict[1][2][1] = pycuda::real(pycuda::complex<double>(3.*sqrt(5.)/(4.*sqrt(pi)),0.));
   reNj1j2hdict[1][2][2] = pycuda::real(pycuda::complex<double>(0.,3.*sqrt(5.)/(4.*sqrt(pi))));
   reNj1j2hdict[2][0][0] = pycuda::real(pycuda::complex<double>(sqrt(5.)/(4.*sqrt(2.*pi)),0.));
   reNj1j2hdict[2][1][0] = pycuda::real(pycuda::complex<double>(-sqrt(15.)/(4.*sqrt(2.*pi)),0.));
   reNj1j2hdict[2][1][1] = pycuda::real(pycuda::complex<double>(-3.*sqrt(5.)/(4.*sqrt(pi)),0.));
   reNj1j2hdict[2][1][2] = pycuda::real(pycuda::complex<double>(0.,-3.*sqrt(5.)/(4.*sqrt(pi))));
   reNj1j2hdict[2][2][0] = pycuda::real(pycuda::complex<double>(5./(8.*sqrt(2.*pi)),0.));
   reNj1j2hdict[2][2][1] = pycuda::real(pycuda::complex<double>(15./(4.*sqrt(pi)),0.));
   reNj1j2hdict[2][2][2] = pycuda::real(pycuda::complex<double>(0.,15./(4.*sqrt(pi))));
   reNj1j2hdict[2][2][3] = pycuda::real(pycuda::complex<double>(15./(16.*sqrt(pi)),0.));
   reNj1j2hdict[2][2][4] = pycuda::real(pycuda::complex<double>(0.,15./(16.*sqrt(pi))));
   imNj1j2hdict[0][0][0] = pycuda::imag(pycuda::complex<double>(1./(2.*sqrt(2.*pi)),0.));
   imNj1j2hdict[0][1][0] = pycuda::imag(pycuda::complex<double>(-sqrt(3.)/(2.*sqrt(2.*pi)),0.));
   imNj1j2hdict[0][2][0] = pycuda::imag(pycuda::complex<double>(sqrt(5.)/(4.*sqrt(2.*pi)),0.));
   imNj1j2hdict[1][0][0] = pycuda::imag(pycuda::complex<double>(sqrt(3.)/(2.*sqrt(2.*pi)),0.));
   imNj1j2hdict[1][1][0] = pycuda::imag(pycuda::complex<double>(-3./(2.*sqrt(2.*pi)),0.));
   imNj1j2hdict[1][1][1] = pycuda::imag(pycuda::complex<double>(-3./(4.*sqrt(pi)),0.));
   imNj1j2hdict[1][1][2] = pycuda::imag(pycuda::complex<double>(0.,-3./(4.*sqrt(pi))));
   imNj1j2hdict[1][2][0] = pycuda::imag(pycuda::complex<double>(sqrt(15.)/(4.*sqrt(2.*pi)),0.));
   imNj1j2hdict[1][2][1] = pycuda::imag(pycuda::complex<double>(3.*sqrt(5.)/(4.*sqrt(pi)),0.));
   imNj1j2hdict[1][2][2] = pycuda::imag(pycuda::complex<double>(0.,3.*sqrt(5.)/(4.*sqrt(pi))));
   imNj1j2hdict[2][0][0] = pycuda::imag(pycuda::complex<double>(sqrt(5.)/(4.*sqrt(2.*pi)),0.));
   imNj1j2hdict[2][1][0] = pycuda::imag(pycuda::complex<double>(-sqrt(15.)/(4.*sqrt(2.*pi)),0.));
   imNj1j2hdict[2][1][1] = pycuda::imag(pycuda::complex<double>(-3.*sqrt(5.)/(4.*sqrt(pi)),0.));
   imNj1j2hdict[2][1][2] = pycuda::imag(pycuda::complex<double>(0.,-3.*sqrt(5.)/(4.*sqrt(pi))));
   imNj1j2hdict[2][2][0] = pycuda::imag(pycuda::complex<double>(5./(8.*sqrt(2.*pi)),0.));
   imNj1j2hdict[2][2][1] = pycuda::imag(pycuda::complex<double>(15./(4.*sqrt(pi)),0.));
   imNj1j2hdict[2][2][2] = pycuda::imag(pycuda::complex<double>(0.,15./(4.*sqrt(pi))));
   imNj1j2hdict[2][2][3] = pycuda::imag(pycuda::complex<double>(15./(16.*sqrt(pi)),0.));
   imNj1j2hdict[2][2][4] = pycuda::imag(pycuda::complex<double>(0.,15./(16.*sqrt(pi))));

 }


// ##########################################
// Normalisation weights
 
__device__ int indexdictcpp[4050] = {0,1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,2,
3,-1,-1,-1,-1,-1,-1,-1,-1,4,5,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,6,7,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,8,9,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,10,
11,-1,-1,-1,-1,-1,-1,-1,-1,12,13,-1,-1,-1,-1,-1,-1,-1,-1,14,15,-1,-1,-1,-1,-1,-1,-1,-1,16,17,-1,-1,-1,-1,-1,-1,-1,-1,18,19,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,20,21,-1,-1,-1,-1,-1,-1,-1,-1,22,23,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,24,
25,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,26,27,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,28,
29,-1,-1,-1,-1,-1,-1,-1,-1,30,31,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,32,33,-1,-1,-1,-1,-1,-1,-1,-1,34,35,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,36,37,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,38,39,-1,-1,-1,-1,-1,-1,-1,-1,40,
41,-1,-1,-1,-1,-1,-1,-1,-1,42,43,-1,-1,-1,-1,-1,-1,-1,-1,44,45,-1,-1,-1,-1,-1,-1,-1,-1,46,47,48,49,-1,-1,-1,-1,-1,-1,50,51,-1,-1,-1,-1,-1,-1,-1,-1,52,
53,-1,-1,-1,-1,-1,-1,-1,-1,54,55,-1,-1,-1,-1,-1,-1,-1,-1,56,57,-1,-1,-1,-1,-1,-1,-1,-1,58,59,-1,-1,-1,-1,-1,-1,-1,-1,60,61,-1,-1,-1,-1,-1,-1,-1,-1,62,
63,-1,-1,-1,-1,-1,-1,-1,-1,64,65,-1,-1,-1,-1,-1,-1,-1,-1,66,67,68,69,70,71,-1,-1,-1,-1,72,73,74,75,-1,-1,-1,-1,-1,-1,76,77,-1,-1,-1,-1,-1,-1,-1,-1,78,
79,80,81,-1,-1,-1,-1,-1,-1,82,83,84,85,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,86,
87,-1,-1,-1,-1,-1,-1,-1,-1,88,89,-1,-1,-1,-1,-1,-1,-1,-1,90,91,-1,-1,-1,-1,-1,-1,-1,-1,92,93,-1,-1,-1,-1,-1,-1,-1,-1,94,95,-1,-1,-1,-1,-1,-1,-1,-1,96,
97,-1,-1,-1,-1,-1,-1,-1,-1,98,99,-1,-1,-1,-1,-1,-1,-1,-1,100,101,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,102,103,-1,-1,-1,-1,-1,-1,-1,-1,104,
105,-1,-1,-1,-1,-1,-1,-1,-1,106,107,-1,-1,-1,-1,-1,-1,-1,-1,108,109,-1,-1,-1,-1,-1,-1,-1,-1,110,111,112,113,-1,-1,-1,-1,-1,-1,114,115,116,117,-1,-1,-1,-1,-1,-1,118,
119,-1,-1,-1,-1,-1,-1,-1,-1,120,121,122,123,-1,-1,-1,-1,-1,-1,124,125,-1,-1,-1,-1,-1,-1,-1,-1,126,127,-1,-1,-1,-1,-1,-1,-1,-1,128,129,-1,-1,-1,-1,-1,-1,-1,-1,130,
131,-1,-1,-1,-1,-1,-1,-1,-1,132,133,-1,-1,-1,-1,-1,-1,-1,-1,134,135,136,137,138,139,-1,-1,-1,-1,140,141,142,143,144,145,-1,-1,-1,-1,146,147,-1,-1,-1,-1,-1,-1,-1,-1,148,
149,150,151,152,153,-1,-1,-1,-1,154,155,156,157,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,158,
159,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,160,161,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,162,163,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,164,
165,-1,-1,-1,-1,-1,-1,-1,-1,166,167,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,168,169,-1,-1,-1,-1,-1,-1,-1,-1,170,171,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,172,173,-1,-1,-1,-1,-1,-1,-1,-1,174,175,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,176,177,-1,-1,-1,-1,-1,-1,-1,-1,178,
179,-1,-1,-1,-1,-1,-1,-1,-1,180,181,-1,-1,-1,-1,-1,-1,-1,-1,182,183,-1,-1,-1,-1,-1,-1,-1,-1,184,185,186,187,-1,-1,-1,-1,-1,-1,188,189,-1,-1,-1,-1,-1,-1,-1,-1,190,
191,-1,-1,-1,-1,-1,-1,-1,-1,192,193,194,195,-1,-1,-1,-1,-1,-1,196,197,-1,-1,-1,-1,-1,-1,-1,-1,198,199,-1,-1,-1,-1,-1,-1,-1,-1,200,201,-1,-1,-1,-1,-1,-1,-1,-1,202,
203,-1,-1,-1,-1,-1,-1,-1,-1,204,205,-1,-1,-1,-1,-1,-1,-1,-1,206,207,208,209,210,211,-1,-1,-1,-1,212,213,214,215,-1,-1,-1,-1,-1,-1,216,217,-1,-1,-1,-1,-1,-1,-1,-1,218,
219,220,221,222,223,-1,-1,-1,-1,224,225,226,227,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,
-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,228,
229,-1,-1,-1,-1,-1,-1,-1,-1,230,231,-1,-1,-1,-1,-1,-1,-1,-1,232,233,-1,-1,-1,-1,-1,-1,-1,-1,234,235,-1,-1,-1,-1,-1,-1,-1,-1,236,237,-1,-1,-1,-1,-1,-1,-1,-1,238,
239,-1,-1,-1,-1,-1,-1,-1,-1,240,241,-1,-1,-1,-1,-1,-1,-1,-1,242,243,-1,-1,-1,-1,-1,-1,-1,-1,244,245,-1,-1,-1,-1,-1,-1,-1,-1,246,247,-1,-1,-1,-1,-1,-1,-1,-1,248,
249,-1,-1,-1,-1,-1,-1,-1,-1,250,251,-1,-1,-1,-1,-1,-1,-1,-1,252,253,-1,-1,-1,-1,-1,-1,-1,-1,254,255,256,257,-1,-1,-1,-1,-1,-1,258,259,260,261,-1,-1,-1,-1,-1,-1,262,
263,-1,-1,-1,-1,-1,-1,-1,-1,264,265,266,267,-1,-1,-1,-1,-1,-1,268,269,270,271,-1,-1,-1,-1,-1,-1,272,273,-1,-1,-1,-1,-1,-1,-1,-1,274,275,-1,-1,-1,-1,-1,-1,-1,-1,276,
277,-1,-1,-1,-1,-1,-1,-1,-1,278,279,-1,-1,-1,-1,-1,-1,-1,-1,280,281,282,283,284,285,-1,-1,-1,-1,286,287,288,289,290,291,-1,-1,-1,-1,292,293,-1,-1,-1,-1,-1,-1,-1,-1,294,
295,296,297,298,299,-1,-1,-1,-1,300,301,302,303,304,305,-1,-1,-1,-1,306,307,-1,-1,-1,-1,-1,-1,-1,-1,308,309,-1,-1,-1,-1,-1,-1,-1,-1,310,311,-1,-1,-1,-1,-1,-1,-1,-1,312,
313,-1,-1,-1,-1,-1,-1,-1,-1,314,315,316,317,318,319,-1,-1,-1,-1,320,321,322,323,324,325,-1,-1,-1,-1,326,327,-1,-1,-1,-1,-1,-1,-1,-1,328,329,330,331,332,333,-1,-1,-1,-1,334,
335,336,337,338,339,340,341,-1,-1,342,343,-1,-1,-1,-1,-1,-1,-1,-1,344,345,-1,-1,-1,-1,-1,-1,-1,-1,346,347,-1,-1,-1,-1,-1,-1,-1,-1,348,349,-1,-1,-1,-1,-1,-1,-1,-1,350,
351,352,353,354,355,-1,-1,-1,-1,356,357,358,359,360,361,-1,-1,-1,-1,362,363,-1,-1,-1,-1,-1,-1,-1,-1,364,365,366,367,368,369,-1,-1,-1,-1,370,371,372,373,374,375,376,377,378,379};
 
__device__ int redindexcpp(int j1, int j2, int h, int j1p, int j2p, int hp, int part) {
   return indexdictcpp[part+2*hp+10*j2p+30*j1p+90*h+450*j2+1350*j1];
 }
 
__device__ double nw_comp_matrix[2][2][380];

__device__ double nw_comp(int year_opt,int trig_opt,int j1,int j2,int h,int j1p,int j2p,int hp,int part) {
   return nw_comp_matrix[year_opt][trig_opt][redindexcpp(j1,j2,h,j1p,j2p,hp,part)];
 }


// ##########################################
// Splines
 
__device__ double knots_wide[6] = {0.0,0.9,1.4,2.1,3.1,12.0}; 
__device__ double knots_narrow[6] = {0.0,0.9,1.4,2.0,3.1,12.0};
 
__device__ double a_2011_L0TIS_wide[5][4] = {{0.0,-0.00112621527248,1.58569721831,-0.819679115309},
{-0.695240812888,2.31634316102,-0.989268755348,0.134011986046},
{-0.968319206849,2.90151114808,-1.40724588896,0.233530351192},
{1.49093519128,-0.611709420676,0.265716286636,-0.0320192004899},
{0.480295052314,0.366329423485,-0.0497801147062,0.00190514374049}}; 
__device__ double a_2012_L0TIS_wide[5][4] = {{0.0,-0.00108246180814,2.40916212056,-1.38444396395},
{-1.71192960187,5.70534954441,-3.93131788635,0.963881964535},
{1.5097616167,-1.19827449537,0.999842142073,-0.210203756517},
{-1.60039623414,3.24480814869,-1.11591149796,0.125630154599},
{2.3184526579,-0.547626262968,0.107454441287,-0.00591457005072}}; 
__device__ double a_2011_L0noTIS_wide[5][4] = {{0.0,-0.00098639638413,2.21305126523,-1.24975680956},
{-1.38905016398,4.62918081687,-2.93157897172,0.655661796718},
{0.301283465128,1.00703732593,-0.344333621046,0.0396509989394},
{0.280998193193,1.03601628584,-0.358133125764,0.0418413965137},
{1.65203424402,-0.290792795608,0.0698698037346,-0.00418042386257}}; 
__device__ double a_2012_L0noTIS_wide[5][4] = {{0.0,-0.00143959572436,4.23974268964,-2.51791074478},
{-3.18832348534,10.6263053554,-7.56886281162,1.85564684828},
{2.64645621312,-1.87679399844,1.36192244113,-0.270730592851},
{-1.04557535896,3.39753681882,-1.14966366233,0.127933868015},
{2.9370024549,-0.456570742976,0.0935968414771,-0.00575005712493}}; 
__device__ double a_2011_L0TIS_narrow[5][4] = {{0.0,-0.00105007049752,2.10287696037,-1.21206867385},
{-1.42394206761,4.74542348821,-3.17098254931,0.741212626035},
{0.646630522127,0.308482224484,-0.00173878950165,-0.0133692215384},
{0.350645013584,0.752460487299,-0.223727920909,0.0236289670295},
{1.06879626479,0.0574754054844,0.000460815159922,-0.000477348676871}}; 
__device__ double a_2012_L0TIS_narrow[5][4] = {{0.0,-0.0217677276861,1.92932115124,-1.02522029051},
{-0.924217061529,3.05895581074,-1.49370500257,0.242567173866},
{-1.04708252048,3.32223893706,-1.68176437851,0.287343215756},
{1.38760726219,-0.329795736933,0.144252958489,-0.0169930070766},
{0.873682232338,0.167551066145,-0.0161814941165,0.000258009332581}}; 
__device__ double a_2011_L0noTIS_narrow[5][4] = {{0.0,-0.00141362744144,7.02846926232,-4.03198237274},
{-4.36857014682,14.5604868619,-9.15142017033,1.96056926898},
{-0.84323191785,7.00619065702,-3.75549430966,0.67582501644},
{5.39200898979,-2.34667070444,0.920936371063,-0.103580097015},
{2.42800757735,0.521717759207,-0.00435023011253,-0.00408691409238}}; 
__device__ double a_2012_L0noTIS_narrow[5][4] = {{0.0,-0.0135116503737,3.32589922088,-2.04848218086},
{-2.87136660569,9.55771036858,-7.3087919113,1.89029231253},
{3.9544829436,-5.06911009418,3.13893699067,-0.597262187936},
{-1.99801374513,3.85963493892,-1.32543552587,0.146799898155},
{2.48668454014,-0.480395659729,0.0745743446588,-0.00373879760085}};

__device__ double spline_knot(int wide_window,int i) {
   if (wide_window == 0) {return knots_narrow[i];}
   else {return knots_wide[i];}
 }

__device__ double spline_coef(int year_opt,int trig_opt,int wide_window,int ibin,int deg) {
   if (year_opt == 0) {
      if (trig_opt == 0) {
         if (wide_window == 0) {return a_2011_L0TIS_narrow[ibin][deg];}
         else {return a_2011_L0TIS_wide[ibin][deg];}
      }
      else {
         if (wide_window == 0) {return a_2011_L0noTIS_narrow[ibin][deg];}
         else {return a_2011_L0noTIS_wide[ibin][deg];}
      }
   }
   else {
      if (trig_opt == 0) {
         if (wide_window == 0) {return a_2012_L0TIS_narrow[ibin][deg];}
         else {return a_2012_L0TIS_wide[ibin][deg];}
      }
      else {
         if (wide_window == 0) {return a_2012_L0noTIS_narrow[ibin][deg];}
         else {return a_2012_L0noTIS_wide[ibin][deg];}
      }
   }
 }


// ##########################################
// Buffer variables

__device__ const int max_N_events = 100000;

__device__ int wide_window;
__device__ int year_opt[4];
__device__ int trig_opt[4];
__device__ double alt_fit;
__device__ double option;
__device__ double inftres;
__device__ double acctype;
__device__ double A_j1;
__device__ double A_j2;
__device__ double A_h;
__device__ double A_j1p;
__device__ double A_j2p;
__device__ double A_hp;
__device__ double qcomp;
__device__ int decision_SSK[4][max_N_events];
__device__ int decision_OS[4][max_N_events];
__device__ double etamistag_SSK[4][max_N_events];
__device__ double etamistag_OS[4][max_N_events];
__device__ double m1[4][max_N_events];
__device__ double m2[4][max_N_events];
__device__ double cos1[4][max_N_events];
__device__ double cos2[4][max_N_events];
__device__ double phi[4][max_N_events];
__device__ double t[4][max_N_events];
__device__ double t_err[4][max_N_events];
__device__ double max_fun[max_N_events];
__device__ double fun_ran[max_N_events];
__device__ double dec_accepted[max_N_events];
__device__ double reA00;
__device__ double reA01;
__device__ double reA10;
__device__ double reA02;
__device__ double reA20;
__device__ double reA110;
__device__ double reA11par;
__device__ double reA11perp;
__device__ double reA120;
__device__ double reA12par;
__device__ double reA12perp;
__device__ double reA210;
__device__ double reA21par;
__device__ double reA21perp;
__device__ double reA220;
__device__ double reA22par;
__device__ double reA22perp;
__device__ double reA22par2;
__device__ double reA22perp2;
__device__ double DCP_SS;
__device__ double DCP_SV;
__device__ double DCP_VS;
__device__ double DCP_ST;
__device__ double DCP_TS;
__device__ double DCP;
__device__ double DCP_VT;
__device__ double DCP_TV;
__device__ double DCP_TT;
__device__ double imA00;
__device__ double imA01;
__device__ double imA10;
__device__ double imA02;
__device__ double imA20;
__device__ double imA110;
__device__ double imA11par;
__device__ double imA11perp;
__device__ double imA120;
__device__ double imA12par;
__device__ double imA12perp;
__device__ double imA210;
__device__ double imA21par;
__device__ double imA21perp;
__device__ double imA220;
__device__ double imA22par;
__device__ double imA22perp;
__device__ double imA22par2;
__device__ double imA22perp2;
__device__ double phis;
__device__ double dphi_SS;
__device__ double dphi_SV;
__device__ double dphi_VS;
__device__ double dphi_ST;
__device__ double dphi_TS;
__device__ double dphi_VT;
__device__ double dphi_TV;
__device__ double dphi_TT;
__device__ double delta_m_freq;
__device__ double gamma_Bs_freq;
__device__ double delta_gamma_freq;
__device__ double p0metac_tag_SSK;
__device__ double p0metac_tag_OS;
__device__ double Dp0half_tag_SSK;
__device__ double Dp0half_tag_OS;
__device__ double p1_tag_SSK;
__device__ double p1_tag_OS;
__device__ double Dp1half_tag_SSK;
__device__ double Dp1half_tag_OS;
__device__ double etac_tag_SSK;
__device__ double etac_tag_OS;
__device__ double deltatmean_tres_11;
__device__ double p0_tres_11;
__device__ double p1_tres_11;
__device__ double deltatmean_tres_12;
__device__ double p0_tres_12;
__device__ double p1_tres_12;
__device__ double mv;
__device__ double ms;
__device__ double mt;
__device__ double gv;
__device__ double gs;
__device__ double gt;
__device__ double c1_mass_swave;
__device__ double c2_mass_swave;
__device__ double c3_mass_swave;
__device__ double c4_mass_swave;
__device__ double c5_mass_swave;
__device__ double c6_mass_swave;
__device__ double c7_mass_swave;
__device__ double c8_mass_swave;
__device__ double c9_mass_swave;
__device__ double res_mass;
__device__ double tag_eff_SSK;
__device__ double mu1_SSK;
__device__ double sigma1_SSK;
__device__ double c_SSK;
__device__ double mu2_SSK;
__device__ double sigma2_SSK;
__device__ double tag_eff_OS;
__device__ double mu1_OS;
__device__ double sigma1_OS;
__device__ double c_OS;
__device__ double mu2_OS;
__device__ double sigma2_OS;
__device__ double gamma1_dt;
__device__ double beta1_dt;
__device__ double c_dt;
__device__ double gamma2_dt;
__device__ double beta2_dt;
__device__ double pw_mass_altmodel;
__device__ double f_1410_rel2_892;
__device__ double delta_1410_rel2_892;
__device__ double f_1680_rel2_892;
__device__ double delta_1680_rel2_892;
__device__ double Im00;
__device__ double Im01;
__device__ double Im10;
__device__ double Im02;
__device__ double Im20;
__device__ double Im11;
__device__ double Im12;
__device__ double Im21;
__device__ double Im22;
__device__ double Ih1Re;
__device__ double Ih2Re;
__device__ double Ih3Re;
__device__ double Ih4Re;
__device__ double Ih5Re;
__device__ double Ih6Re;
__device__ double Ih7Re;
__device__ double Ih8Re;
__device__ double Ih9Re;
__device__ double Ih10Re;
__device__ double Ih11Re;
__device__ double Ih12Re;
__device__ double Ih13Re;
__device__ double Ih14Re;
__device__ double Ih15Re;
__device__ double Ih16Re;
__device__ double Ih17Re;
__device__ double Ih18Re;
__device__ double Ih19Re;
__device__ double Ih20Re;
__device__ double Ih21Re;
__device__ double Ih22Re;
__device__ double Ih23Re;
__device__ double Ih24Re;
__device__ double Ih25Re;
__device__ double Ih26Re;
__device__ double Ih27Re;
__device__ double Ih28Re;
__device__ double Ih29Re;
__device__ double Ih30Re;
__device__ double Ih1Im;
__device__ double Ih2Im;
__device__ double Ih3Im;
__device__ double Ih4Im;
__device__ double Ih5Im;
__device__ double Ih6Im;
__device__ double Ih7Im;
__device__ double Ih8Im;
__device__ double Ih9Im;
__device__ double Ih10Im;
__device__ double Ih11Im;
__device__ double Ih12Im;
__device__ double Ih13Im;
__device__ double Ih14Im;
__device__ double Ih15Im;
__device__ double Ih16Im;
__device__ double Ih17Im;
__device__ double Ih18Im;
__device__ double Ih19Im;
__device__ double Ih20Im;
__device__ double Ih21Im;
__device__ double If1;
__device__ double If2;
__device__ double If3;
__device__ double If4;
__device__ double If5;
__device__ double If6;
__device__ double If7;
__device__ double If8;
__device__ double If9;
__device__ double If10;
__device__ double If11;
__device__ double If12;
__device__ double If13;
__device__ double If14;
__device__ double If15;
__device__ double If16;
__device__ double If17;
__device__ double If18;

__device__ double reAj1j2h_temp[3][3][5];
__device__ double imAj1j2h_temp[3][3][5];
__device__ double reAbarj1j2h_temp[3][3][5];
__device__ double imAbarj1j2h_temp[3][3][5];

__device__ pycuda::complex<double> Aj1j2h_temp(int j1, int j2, int h) {

   return pycuda::complex<double>(reAj1j2h_temp[j1][j2][h],imAj1j2h_temp[j1][j2][h]);

 }

__device__ pycuda::complex<double> Abarj1j2h_temp(int j1, int j2, int h) {

   return pycuda::complex<double>(reAbarj1j2h_temp[j1][j2][h],imAbarj1j2h_temp[j1][j2][h]);

 }

__device__ double T_cosh_temp[4][max_N_events];
__device__ double T_sinh_temp[4][max_N_events];
__device__ double T_cos_temp[4][max_N_events];
__device__ double T_sin_temp[4][max_N_events];
__device__ double IT_cosh_temp_deltat[4][max_N_events];
__device__ double IT_sinh_temp_deltat[4][max_N_events];
__device__ double IT_cos_temp_deltat[4][max_N_events];
__device__ double IT_sin_temp_deltat[4][max_N_events];
__device__ double zeta_temp[4][max_N_events];
__device__ double DCP_tzero_temp[4][max_N_events];
__device__ double fi_cos1_temp[18][4][max_N_events];
__device__ double fi_cos2_temp[18][4][max_N_events];
__device__ double gi_temp[15][4][max_N_events];
__device__ double reMj1j2_temp[3][3][4][max_N_events];
__device__ double imMj1j2_temp[3][3][4][max_N_events];
__device__ double phasespace_temp[4][max_N_events];
__device__ double reIhj1j2j1pj2pdict[3][3][3][3];
__device__ double imIhj1j2j1pj2pdict[3][3][3][3];

__device__ pycuda::complex<double> Mj1j2_temp(int j1, int j2, int icat, int iev) {

   return pycuda::complex<double>(reMj1j2_temp[j1][j2][icat][iev],imMj1j2_temp[j1][j2][icat][iev]);

 }

__device__ pycuda::complex<double> Ihj1j2j1pj2p(int j1, int j2, int j1p, int j2p) {

   return pycuda::complex<double>(reIhj1j2j1pj2pdict[j1][j2][j1p][j2p],imIhj1j2j1pj2pdict[j1][j2][j1p][j2p]);

 }

// ##########################################
// Toy MC generation variables

__device__ double knots_gen_wide[6] = {0.0,0.9,1.3,1.9,3.0,12.0};
__device__ double knots_gen_narrow[6] = {0.0,0.9,1.3,1.9,3.0,12.0};

__device__ double a_gen_wide[5][4] = {{0.0,-0.00138436998913,2.5481847953,-1.45909728079},
{-1.6653800648,5.54988251268,-3.61988951878,0.82537468739},
{-0.289336418837,2.37439717584,-1.17720849044,0.199046218586},
{0.993185871959,0.349361979846,-0.111400492548,0.0120623593064},
{1.32606052325,0.0164873285591,-0.000442275452223,-0.000266331481965}};

__device__ double a_gen_narrow[5][4] = {{0.0,0.00101382530285,4.89487359849,-2.83048035352},
{-3.54249846114,11.8093420291,-8.22549107238,2.02891396902},
{1.06333885612,1.18048668157,-0.0494484973637,-0.0675072040589},
{-0.421082535913,3.52430993215,-1.28303968188,0.148912301997},
{3.78015377185,-0.67692637561,0.117372420705,-0.006689042735}};

__device__ double k1_gen(int wide_window) {
   if (wide_window) {return -0.40631262195;}
   else {return -0.505556252411;}
 }

__device__ double k2_gen(int wide_window) {
   if (wide_window) {return -0.39861379722;}
   else {return -0.404368705592;}
 }

__device__ double k3_gen(int wide_window) {
   if (wide_window) {return -0.0363987194893;}
   else {return -0.0483750503137;}
 }

__device__ double k4_gen(int wide_window) {
   if (wide_window) {return -0.0644151228873;}
   else {return -0.0175772310185;}
 }

__device__ double k5_gen(int wide_window) {
   if (wide_window) {return 0.0270906873059;}
   else {return 0.0389936024545;}
 }

__device__ double p1_gen(int wide_window) {
   if (wide_window) {return -0.000100573256821;}
   else {return 4.35273527839e-05;}
 }

__device__ double knot_gen(int wide_window,int i) {
   if (wide_window == 0) {return knots_gen_narrow[i];}
   else {return knots_gen_wide[i];}
 }

__device__ double coef_gen(int wide_window,int ibin,int deg) {
   if (wide_window == 0) {return a_gen_narrow[ibin][deg];}
   else {return a_gen_wide[ibin][deg];}
 }

__device__ double accGenTime(double tau) { 

   int tau_bin;
   if (tau < knot_gen(wide_window,1)) {tau_bin = 0;}
   else if ((tau >= knot_gen(wide_window,1)) and (tau < knot_gen(wide_window,2))) {tau_bin = 1;}
   else if ((tau >= knot_gen(wide_window,2)) and (tau < knot_gen(wide_window,3))) {tau_bin = 2;}
   else if ((tau >= knot_gen(wide_window,3)) and (tau < knot_gen(wide_window,4))) {tau_bin = 3;}
   else {tau_bin = 4;}

   return coef_gen(wide_window,tau_bin,0)+tau*coef_gen(wide_window,tau_bin,1)+tau*tau*coef_gen(wide_window,tau_bin,2)+tau*tau*tau*coef_gen(wide_window,tau_bin,3);

 }

__device__ double accGenAng(double x) { 

   return 1.+k1_gen(wide_window)*x+k2_gen(wide_window)*(2.*x*x-1.)+k3_gen(wide_window)*(4.*x*x*x-3.*x)+k4_gen(wide_window)*(8.*x*x*x*x-8.*x*x+1.)+k5_gen(wide_window)*(16.*x*x*x*x*x-20.*x*x*x+5.*x);

 }

__device__ double accGenMass(double m) { 
   
   return 1. + p1_gen(wide_window)*m;

 }

__device__ double accGen(double tau, double ma, double mb, double cos1var, double cos2var, double phivar) {
   return accGenTime(tau)*accGenMass(ma)*accGenMass(mb)*accGenAng(cos1var)*accGenAng(cos2var);
 }


// ##########################################
// Physical terms

__device__ double reAj1j2h(int j1, int j2, int h) {

   switch(j1) {
   case 0 :
      switch(j2) {
      case 0 : return reA00;
      case 1 : return reA01;
      case 2 : return reA02;
      }
   case 1 :
      switch(j2) {
      case 0 : return reA10;
      case 1 :
         switch(h) {
         case 0 : return reA110;
         case 1 : return reA11par;
         case 2 : return reA11perp;
         }
      case 2 :
         switch(h) {
         case 0 : return reA120;
         case 1 : return reA12par;
         case 2 : return reA12perp;      
         }
      }
   case 2 :
      switch(j2) {
      case 0 : return reA20;
      case 1 :
         switch(h) {
         case 0 : return reA210;
         case 1 : return reA21par;
         case 2 : return reA21perp;
         }
      case 2 :
         switch(h) {
         case 0 : return reA220;
         case 1 : return reA22par;
         case 2 : return reA22perp;
         case 3 : return reA22par2;
         case 4 : return reA22perp2;
         }
      }
   }
   return 0.;

 }

__device__ double imAj1j2h(int j1, int j2, int h) {

   switch(j1) {
   case 0 :
      switch(j2) {
      case 0 : return imA00;
      case 1 : return imA01;
      case 2 : return imA02;
      }
   case 1 :
      switch(j2) {
      case 0 : return imA10;
      case 1 :
         switch(h) {
         case 0 : return imA110;
         case 1 : return imA11par;
         case 2 : return imA11perp;
         }
      case 2 :
         switch(h) {
         case 0 : return imA120;
         case 1 : return imA12par;
         case 2 : return imA12perp;      
         }
      }
   case 2 :
      switch(j2) {
      case 0 : return imA20;
      case 1 :
         switch(h) {
         case 0 : return imA210;
         case 1 : return imA21par;
         case 2 : return imA21perp;
         }
      case 2 :
         switch(h) {
         case 0 : return imA220;
         case 1 : return imA22par;
         case 2 : return imA22perp;
         case 3 : return imA22par2;
         case 4 : return imA22perp2;
         }
      }
   }
   return 0.;

 }

__device__ double DCPj1j2(int j1, int j2) {

   switch(j1) {
   case 0 :
      switch(j2) {
      case 0 : return DCP;//+DCP_SS;
      case 1 : return DCP;//+DCP_SV;
      case 2 : return DCP;//+DCP_TT;//DCP_ST;
      }
   case 1 :
      switch(j2) {
      case 0 : return DCP;//+DCP_VS;
      case 1 : return DCP;
      case 2 : return DCP;//+DCP_TT;//DCP_VT;
      }
   case 2 :
      switch(j2) {
      case 0 : return DCP;//+DCP_TT;//DCP_TS;
      case 1 : return DCP;//+DCP_TT;//DCP_TV;
      case 2 : return DCP;//+DCP_TT;
      }
   }
   return 0;

 }

__device__ double dphij1j2(int j1, int j2) {

   switch(j1) {
   case 0 :
      switch(j2) {
      case 0 : return dphi_SS;
      case 1 : return dphi_SV;
      case 2 : return dphi_ST;
      }
   case 1 :
      switch(j2) {
      case 0 : return dphi_VS;
      case 1 : return 0.;
      case 2 : return dphi_VT;
      }
   case 2 :
      switch(j2) {
      case 0 : return dphi_TS;
      case 1 : return dphi_TV;
      case 2 : return dphi_TT;
      }
   }
   return 0;

 }

__device__ double etah(int h) {

   if ((h == 2) or (h == 4)) {return -1.;}
   else {return 1.;}

 }

__device__ double etaj1j2h(int j1, int j2, int h) {

   return pow(-1.,j1+j2)*etah(h);

 }

__device__ pycuda::complex<double> Aj1j2h(int j1, int j2, int h) {

   pycuda::complex<double> I(0.,1.);
   return pycuda::complex<double>(reAj1j2h(j1,j2,h),imAj1j2h(j1,j2,h))*pycuda::complex<double>(sqrt(1.+DCPj1j2(j1,j2)))*exp(I*0.5*(phis+dphij1j2(j1,j2)));

 }

__device__ pycuda::complex<double> Abarj1j2h(int j1, int j2, int h) {

   pycuda::complex<double> I(0.,1.);
   return etaj1j2h(j2,j1,h)*pycuda::complex<double>(reAj1j2h(j2,j1,h),imAj1j2h(j2,j1,h))*pycuda::complex<double>(sqrt(1.-DCPj1j2(j2,j1)))*exp(-I*0.5*(phis+dphij1j2(j2,j1)));

 }

__device__ pycuda::complex<double> M_Average(int j1, int j2, int h, int j1p, int j2p, int hp) {

   return Aj1j2h_temp(j1,j2,h)*pycuda::conj(Aj1j2h_temp(j1p,j2p,hp))+Abarj1j2h_temp(j1,j2,h)*pycuda::conj(Abarj1j2h_temp(j1p,j2p,hp));

 }

__device__ pycuda::complex<double> M_DeltaGamma(int j1, int j2, int h, int j1p, int j2p, int hp) {

   return Aj1j2h_temp(j1,j2,h)*pycuda::conj(Abarj1j2h_temp(j1p,j2p,hp))+Abarj1j2h_temp(j1,j2,h)*pycuda::conj(Aj1j2h_temp(j1p,j2p,hp));

 }

__device__ pycuda::complex<double> M_DirCP(int j1, int j2, int h, int j1p, int j2p, int hp) {

   return Aj1j2h_temp(j1,j2,h)*pycuda::conj(Aj1j2h_temp(j1p,j2p,hp))-Abarj1j2h_temp(j1,j2,h)*pycuda::conj(Abarj1j2h_temp(j1p,j2p,hp));

 }

__device__ pycuda::complex<double> M_MixCP(int j1, int j2, int h, int j1p, int j2p, int hp) {

   return pycuda::complex<double>(0.,-1.)*(Aj1j2h_temp(j1,j2,h)*pycuda::conj(Abarj1j2h_temp(j1p,j2p,hp))-Abarj1j2h_temp(j1,j2,h)*pycuda::conj(Aj1j2h_temp(j1p,j2p,hp)));

 }


// ##########################################
// Flavour tagging terms

__device__ double omega_SSK(double eta) {

   return (p0metac_tag_SSK+etac_tag_SSK+Dp0half_tag_SSK)+(p1_tag_SSK+Dp1half_tag_SSK)*(eta-etac_tag_SSK);

 }

__device__ double omegabar_SSK(double eta) {

   return (p0metac_tag_SSK+etac_tag_SSK-Dp0half_tag_SSK)+(p1_tag_SSK-Dp1half_tag_SSK)*(eta-etac_tag_SSK);

 }

__device__ double omega_OS(double eta) {

   return (p0metac_tag_OS+etac_tag_OS+Dp0half_tag_OS)+(p1_tag_OS+Dp1half_tag_OS)*(eta-etac_tag_OS);

 }

__device__ double omegabar_OS(double eta) {

   return (p0metac_tag_OS+etac_tag_OS-Dp0half_tag_OS)+(p1_tag_OS-Dp1half_tag_OS)*(eta-etac_tag_OS);

 }

__device__ double P_Bs(int q1, int q2, double eta1, double eta2) {

   return (1.+0.5*q1*(1.-q1-2.*omega_SSK(eta1)))*(1.+0.5*q2*(1.-q2-2.*omega_OS(eta2)));

 }

__device__ double P_Bsbar(int q1, int q2, double eta1, double eta2) {

   return (1.-0.5*q1*(1.+q1-2.*omegabar_SSK(eta1)))*(1.-0.5*q2*(1.+q2-2.*omegabar_OS(eta2)));

 }

__device__ double zeta(int q1, int q2, double eta1, double eta2) {

   return 0.5*((1.+DCP_prod)*P_Bs(q1,q2,eta1,eta2)+(1.-DCP_prod)*P_Bsbar(q1,q2,eta1,eta2));

 }

__device__ double DCP_tzero(int q1, int q2, double eta1, double eta2) {

   return 0.5/zeta(q1,q2,eta1,eta2)*((1.+DCP_prod)*P_Bs(q1,q2,eta1,eta2)-(1.-DCP_prod)*P_Bsbar(q1,q2,eta1,eta2));

 }


// ##########################################
// Time dependent terms

#define errf_const 1.12837916709551
#define xLim 5.33
#define yLim 4.29

__device__ pycuda::complex<double> faddeeva(pycuda::complex<double> z) {

   double in_real = pycuda::real(z);
   double in_imag = pycuda::imag(z);
   int n, nc, nu;
   double h, q, Saux, Sx, Sy, Tn, Tx, Ty, Wx, Wy, xh, xl, x, yh, y;
   double Rx [33];
   double Ry [33];

   x = fabs(in_real);
   y = fabs(in_imag);

   if (y < yLim && x < xLim) {
      q = (1.0 - y / yLim) * sqrt(1.0 - (x / xLim) * (x / xLim));
      h  = 1.0 / (3.2 * q);
      nc = 7 + int(23.0 * q);
      xl = pow(h, double(1 - nc));
      xh = y + 0.5 / h;
      yh = x;
      nu = 10 + int(21.0 * q);
      Rx[nu] = 0.;
      Ry[nu] = 0.;
      for (n = nu; n > 0; n--){
         Tx = xh + n * Rx[n];
         Ty = yh - n * Ry[n];
         Tn = Tx*Tx + Ty*Ty;
         Rx[n-1] = 0.5 * Tx / Tn;
         Ry[n-1] = 0.5 * Ty / Tn;
         }
      Sx = 0.;
      Sy = 0.;
      for (n = nc; n>0; n--){
         Saux = Sx + xl;
         Sx = Rx[n-1] * Saux - Ry[n-1] * Sy;
         Sy = Rx[n-1] * Sy + Ry[n-1] * Saux;
         xl = h * xl;
      };
      Wx = errf_const * Sx;
      Wy = errf_const * Sy;
   }
   else {
      xh = y;
      yh = x;
      Rx[0] = 0.;
      Ry[0] = 0.;
      for (n = 9; n>0; n--){
         Tx = xh + n * Rx[0];
         Ty = yh - n * Ry[0];
         Tn = Tx * Tx + Ty * Ty;
         Rx[0] = 0.5 * Tx / Tn;
         Ry[0] = 0.5 * Ty / Tn;
      };
      Wx = errf_const * Rx[0];
      Wy = errf_const * Ry[0];
   }

   if (y == 0.) {
      Wx = exp(-x * x);
   }
   if (in_imag < 0.) {
      Wx =   2.0 * exp(y * y - x * x) * cos(2.0 * x * y) - Wx;
      Wy = - 2.0 * exp(y * y - x * x) * sin(2.0 * x * y) - Wy;
      if (in_real > 0.) {
         Wy = -Wy;
      }
   }
   else if (in_real < 0.) {
      Wy = -Wy;
   }

   return pycuda::complex<double>(Wx,Wy);

}

__device__ pycuda::complex<double> conv_exp(double x, pycuda::complex<double> z) {

   double re = pycuda::real(z)-x;

   if (re>-5.0) {return 0.5*faddeeva(pycuda::complex<double>(-pycuda::real(z),re))*exp(-x*x);}

   else {
      pycuda::complex<double> mi(0,-1);
      pycuda::complex<double> zp  = mi*(z-x);
      pycuda::complex<double> zsq = zp*zp;
      pycuda::complex<double> v = -zsq -x*x;
      pycuda::complex<double> iz(pycuda::real(z)+x,pycuda::real(z)-x);
      return 0.5*exp(v)*(exp(zsq)/(iz*sqrt(pi)) + 1.)*2. ;
   }

 }

__device__ pycuda::complex<double> Kn(pycuda::complex<double> z, int n) {

   if (n == 0) {return 1./(2.*z);}
   else if (n == 1) {return 1./(2.*z*z);}
   else if (n == 2) {return 1./z*(1.+1./(z*z));}
   else if (n == 3) {return 3./(z*z)*(1.+1./(z*z));}
   return pycuda::complex<double>(0.,0.);

 }

__device__ pycuda::complex<double> Mn_x(double x, pycuda::complex<double> z, int n) {

   if (n == 0) {return pycuda::complex<double>(erf(x),0.)-2.*conv_exp(x,z);}
   else if (n == 1) {return 2.*(-pycuda::complex<double>(sqrt(1./pi)*exp(-x*x),0.)-2.*x*conv_exp(x,z));}
   else if (n == 2) {return 2.*(-2.*x*exp(-x*x)*pycuda::complex<double>(sqrt(1./pi),0.)-(2.*x*x-1.)*2.*conv_exp(x,z));}
   else if (n == 3) {return 4.*(-(2.*x*x-1.)*exp(-x*x)*pycuda::complex<double>(sqrt(1./pi),0.)-x*(2.*x*x-3.)*2.*conv_exp(x,z));}
   return pycuda::complex<double>(0.,0.);

 }

__device__ pycuda::complex<double> Mn(double x_1, double x_2, pycuda::complex<double> z, int n) {

   return Mn_x(x_2,z,n)-Mn_x(x_1,z,n);

 }

__device__ pycuda::complex<double> Tj1j2hj1pj2php(int j1, int j2, int h, int j1p, int j2p, int hp, int icat, int iev) {

   return zeta_temp[icat][iev]*((T_cosh_temp[icat][iev]*M_Average(j1,j2,h,j1p,j2p,hp)-T_sinh_temp[icat][iev]*M_DeltaGamma(j1,j2,h,j1p,j2p,hp))+DCP_tzero_temp[icat][iev]*(T_cos_temp[icat][iev]*M_DirCP(j1,j2,h,j1p,j2p,hp)+T_sin_temp[icat][iev]*M_MixCP(j1,j2,h,j1p,j2p,hp)));

 }

__device__ pycuda::complex<double> ITj1j2hj1pj2php_deltat(int j1, int j2, int h, int j1p, int j2p, int hp, int icat, int iev) {

   return (IT_cosh_temp_deltat[icat][iev]*M_Average(j1,j2,h,j1p,j2p,hp)-IT_sinh_temp_deltat[icat][iev]*M_DeltaGamma(j1,j2,h,j1p,j2p,hp))+DCP_prod*(IT_cos_temp_deltat[icat][iev]*M_DirCP(j1,j2,h,j1p,j2p,hp)+IT_sin_temp_deltat[icat][iev]*M_MixCP(j1,j2,h,j1p,j2p,hp));

 }


// ##########################################
// Angular terms

__device__ double fi(double x, int i) { 

   switch(i) {
      case 1 : return 1.;
      case 2 : return x;
      case 3 : return sqrt(1.-x*x);
      case 4 : return 3.*x*x-1.;
      case 5 : return x*sqrt(1.-x*x);
      case 6 : return x*x;
      case 7 : return x*(3.*x*x-1.);
      case 8 : return x*x*sqrt(1.-x*x);
      case 9 : return 1.-x*x;
      case 10 : return (3.*x*x-1.)*sqrt(1.-x*x);
      case 11 : return x*(1.-x*x);
      case 12 : return (3.*x*x-1.)*(3.*x*x-1.);
      case 13 : return x*(3.*x*x-1.)*sqrt(1.-x*x);
      case 14 : return x*x*(1.-x*x);
      case 15 : return (1.-x*x)*sqrt(1.-x*x);
      case 16 : return (3.*x*x-1.)*(1.-x*x);
      case 17 : return x*(1.-x*x)*sqrt(1.-x*x);
      case 18 : return (1.-x*x)*(1.-x*x);
   }
   return 0.;

 }

__device__ double gi(double x, int i) { 

   switch(i) {
      case 1 : return 1.;
      case 2 : return cos(x);
      case 3 : return sin(x);
      case 4 : return cos(x)*cos(x);
      case 5 : return sin(x)*cos(x);
      case 6 : return sin(x)*sin(x);
      case 7 : return cos(2.*x);
      case 8 : return sin(2.*x);
      case 9 : return cos(x)*cos(2.*x);
      case 10 : return cos(x)*sin(2.*x);
      case 11 : return sin(x)*cos(2.*x);
      case 12 : return sin(x)*sin(2.*x);
      case 13 : return cos(2.*x)*cos(2.*x);
      case 14 : return sin(2.*x)*cos(2.*x);
      case 15 : return sin(2.*x)*sin(2.*x);
   }
   return 0.;

 }

__device__ double fjjphhp_cos1(int j, int jp, int h, int hp, int icat, int iev) { 

   return fi_cos1_temp[(int) fjjphhpindexdict[j][jp][h][hp]-1][icat][iev];

 }

__device__ double fjjphhp_cos2(int j, int jp, int h, int hp, int icat, int iev) { 

   return fi_cos2_temp[(int) fjjphhpindexdict[j][jp][h][hp]-1][icat][iev];

 }

__device__ double ghhp_phi(int h, int hp, int icat, int iev) { 

   return gi_temp[(int) ghhpindexdict[h][hp]-1][icat][iev];

 }

__device__ pycuda::complex<double> Nj1j2hj1pj2php(int j1, int j2, int h, int j1p, int j2p, int hp) {

   return Nj1j2hdict(j1,j2,h)*pycuda::conj(Nj1j2hdict(j1p,j2p,hp));
  
 }


// ##########################################
// Mass dependent terms

__device__ double get_q(double M, double ma, double mb) {

   double M2 = M*M;
   double m12 = ma*ma;
   double m22 = mb*mb;
   double q2 = .25*( M2*M2 - 2*M2*(m12+m22) +(m12*m12+m22*m22)-2*m12*m22) /M2;
   if (q2<0) {return 0.;}
   return sqrt(q2);

 }

__device__ double Blatt_Weisskopf2(double q, double q0, int L) {

   if (L<1.) {return 1.;}  
   double d = 1.6e-03;
   double z = q*d*q*d;
   double z0 = q0*d*q0*d;
   if (L==1) {return (1+z0)/(1+z);}
   else if (L==2) {return ((z0-3)*(z0-3) + 9*z0) / ((z-3)*(z-3) + 9*z);}
   else if (L==3) {return (z0*(z0-15)*(z0-15) + 9*(z0-5)) / (z*(z-15)*(z-15) + 9*(z-5));}
   return ( pow(z0*z0 -45*z0+105,2) +25*z0*(2*z0-21)*(2*z0-21)) /(pow(z*z -45*z+105,2) +25*z*(2*z-21)*(2*z-21));

 }

__device__ double FL_j1j2(int j1, int j2, double ma, double mb) {

   double p = get_q(MBs,ma,mb);
   double q_1 = get_q(ma,MPion,MKaon);
   double q_2 = get_q(mb,MPion,MKaon);

   double m0 = 895.81;
   double p0 = get_q(MBs,m0,m0);
   double q0 = get_q(m0,MPion,MKaon);

   //int L;
   //if (j1*j2>0) {L = abs(j1-j2)+1;}
   //else {L = abs(j1-j2);}
   int L = abs(j1-j2);
   double FL_Bs = pow(p/p0,L)*sqrt(Blatt_Weisskopf2(p,p0,L));

   double FL_Kpi1 = pow((q_1/q0),j1)*sqrt(Blatt_Weisskopf2(q_1,q0,j1));

   double FL_Kpi2 = pow((q_2/q0),j2)*sqrt(Blatt_Weisskopf2(q_2,q0,j2));

   return FL_Bs*FL_Kpi1*FL_Kpi2;

 }

__device__ pycuda::complex<double> Resonance(double m, double m0, double g0, int J) {

   double q = get_q(m,MPion,MKaon);
   double q0 = get_q(m0,MPion,MKaon);

   double gamma = g0*pow(q/q0,2*J+1)*(m0/m)*Blatt_Weisskopf2(q,q0,J);

   pycuda::complex<double> num(m0*g0,0.);
   pycuda::complex<double> denom(m0*m0-m*m,-m0*gamma);
   pycuda::complex<double> BW = num/denom;

   pycuda::complex<double> I(0.,1.);
   if (J == 1) {return BW*exp(-I*1.5707963267948966);}
   else if (J == 2) {return BW*exp(-I*0.006008360479292941);}
   return BW;

 }

__device__ double omega_Stheo(double m) {

   double m_GeV = m/1000.;
   double svar_GeV = m_GeV*m_GeV;
   double Delta_Kpi = MKaon/1000.*MKaon/1000.-MPion/1000.*MPion/1000.;
   double y_s = pow((svar_GeV-Delta_Kpi)/(svar_GeV+Delta_Kpi),2);
   double y_s0 = pow((s0_Stheo-Delta_Kpi)/(s0_Stheo+Delta_Kpi),2);
   return (sqrt(y_s)-alpha_Stheo*sqrt(y_s0-y_s))/(sqrt(y_s)+alpha_Stheo*sqrt(y_s0-y_s));

 }

__device__ double cotdelta_Stheo(double m) {

   double m_GeV = m/1000.;
   double svar_GeV = m_GeV*m_GeV;
   double q_Kpi_GeV = get_q(m,MKaon,MPion)/1000.;
   double omega_func = omega_Stheo(m);
   return m_GeV/(2.*q_Kpi_GeV*(svar_GeV-sAdler_Stheo))*(B0_Stheo+B1_Stheo*omega_func);

 }

__device__ double get_p1_Stheo(double q) {

   return 1.+a_Stheo*q/1000.*q/1000.+b_Stheo*q/1000.*q/1000.*q/1000.*q/1000.;

 }

__device__ double get_p2_Stheo(double q) {

   return 1.+c_Stheo*q/1000.*q/1000.;

 }

__device__ double Theta_Keta(double m) {

   if (m>=(MKaon+MEta)) {return 1.;}
   return 0.;

 }

__device__ pycuda::complex<double> Prop_Stheo(double m) { 

   pycuda::complex<double> I(0.,1.);
   double m_GeV = m/1000.;
   double svar_GeV = m_GeV*m_GeV;
   double q_Kpi_GeV = get_q(m,MKaon,MPion)/1000.;
   double q_Keta_GeV = get_q(m,MKaon,MEta)/1000.;

   pycuda::complex<double> T;

   if (m<(MKaon+MEta)) {
      T = 1./(cotdelta_Stheo(m)-I);
   }

   else {
      pycuda::complex<double> S0b = exp(I*2.*q_Keta_GeV*(phi0_Stheo+phi1_Stheo*q_Keta_GeV*q_Keta_GeV));
      double q_Kpi_r1_GeV = get_q(sqrtsr1_Stheo*1000.,MKaon,MPion)/1000.;
      double q_Keta_r1_GeV = get_q(sqrtsr1_Stheo*1000.,MKaon,MEta)/1000.;
      double q_Kpi_r2_GeV = get_q(sqrtsr2_Stheo*1000.,MKaon,MPion)/1000.;
      double q_Keta_r2_GeV = get_q(sqrtsr2_Stheo*1000.,MKaon,MEta)/1000.;
      double q_Kpi_hat_GeV = get_q(MKaon+MEta,MKaon,MPion)/1000.;
      double beta_Stheo = 1./cotdelta_Stheo(MKaon+MEta);
      double P1_s = (sqrtsr1_Stheo*sqrtsr1_Stheo-svar_GeV)*beta_Stheo+e1_Stheo*G1_Stheo*(get_p1_Stheo(q_Kpi_GeV*1000.)*(q_Kpi_GeV-q_Kpi_hat_GeV))/(get_p1_Stheo(q_Kpi_r1_GeV*1000.)*(q_Kpi_r1_GeV-q_Kpi_hat_GeV));
      double Q1_s = (1.-e1_Stheo)*G1_Stheo*get_p1_Stheo(q_Kpi_GeV*1000.)/get_p1_Stheo(q_Kpi_r1_GeV*1000.)*q_Keta_GeV/q_Keta_r1_GeV*Theta_Keta(m);
      double P2_s = e2_Stheo*G2_Stheo*(get_p2_Stheo(q_Kpi_GeV*1000.)*(q_Kpi_GeV-q_Kpi_hat_GeV))/(get_p2_Stheo(q_Kpi_r2_GeV*1000.)*(q_Kpi_r2_GeV-q_Kpi_hat_GeV));
      double Q2_s = (1.-e2_Stheo)*G2_Stheo*get_p2_Stheo(q_Kpi_GeV*1000.)/get_p2_Stheo(q_Kpi_r2_GeV*1000.)*q_Keta_GeV/q_Keta_r2_GeV*Theta_Keta(m);
      pycuda::complex<double> S1r = (sqrtsr1_Stheo*sqrtsr1_Stheo-svar_GeV+I*(P1_s-Q1_s))/(sqrtsr1_Stheo*sqrtsr1_Stheo-svar_GeV-I*(P1_s+Q1_s));
      pycuda::complex<double> S2r = (sqrtsr2_Stheo*sqrtsr2_Stheo-svar_GeV+I*(P2_s-Q2_s))/(sqrtsr2_Stheo*sqrtsr2_Stheo-svar_GeV-I*(P2_s+Q2_s));
      T = (S0b*S1r*S2r-1.)/(2.*I);
   }

   double xm = (m-1175.)/425.;
   double modulus = 1.+c1_mass_swave*xm+c2_mass_swave*(2.*xm*xm-1.)+c3_mass_swave*(4.*xm*xm*xm-3.*xm)+c4_mass_swave*(8.*xm*xm*xm*xm-8.*xm*xm+1.);

   return pycuda::complex<double>(modulus)*exp(I*(pycuda::arg(T)-0.7095863518296103));

 }

__device__ pycuda::complex<double> Prop_ModInd(double m) { 

   double xm = (m-1175.)/425.;
   double re_T = 1.+c1_mass_swave*xm+c2_mass_swave*(2.*xm*xm-1.)+c3_mass_swave*(4.*xm*xm*xm-3.*xm)+c4_mass_swave*(8.*xm*xm*xm*xm-8.*xm*xm+1.);
   double im_T = c5_mass_swave+c6_mass_swave*xm+c7_mass_swave*(2.*xm*xm-1.)+c8_mass_swave*(4.*xm*xm*xm-3.*xm)+c9_mass_swave*(8.*xm*xm*xm*xm-8.*xm*xm+1.);

   pycuda::complex<double> T(re_T,im_T);

   return T;

 }

__device__ pycuda::complex<double> Prop_S_Palano(double m) {

   pycuda::complex<double> i(0.,1.);
   double m_GeV = m/1000.;
   double svar_GeV = m_GeV*m_GeV;
   double q_Kpi_GeV = get_q(m,MKaon,MPion)/1000.;
   double q_Keta_GeV = get_q(m,MKaon,MEta)/1000.;

   double rho_1 = 2.*q_Kpi_GeV/m_GeV;
   double rho_2 = 2.*q_Keta_GeV/m_GeV;

   double sbot_GeV = 0.36;
   double stop_GeV = 5.832;
   double X = (2.*svar_GeV-(stop_GeV+sbot_GeV))/(stop_GeV-sbot_GeV);

   double K11 = (svar_GeV-s_A_palano)/s_Kpi_palano*(g_1_a_palano*g_1_a_palano/(svar_GeV-s_a_palano)+g_1_b_palano*g_1_b_palano/(svar_GeV-s_b_palano)+C_11_0_palano+C_11_1_palano*X+C_11_2_palano*X*X+C_11_3_palano*X*X*X);
   double K12 = (svar_GeV-s_A_palano)/s_Kpi_palano*(g_1_a_palano*g_2_a_palano/(svar_GeV-s_a_palano)+g_1_b_palano*g_2_b_palano/(svar_GeV-s_b_palano)+C_12_0_palano+C_12_1_palano*X+C_12_2_palano*X*X+C_12_3_palano*X*X*X);
   double K22 = (svar_GeV-s_A_palano)/s_Kpi_palano*(g_2_a_palano*g_2_a_palano/(svar_GeV-s_a_palano)+g_2_b_palano*g_2_b_palano/(svar_GeV-s_b_palano)+C_22_0_palano+C_22_1_palano*X+C_22_2_palano*X*X+C_22_3_palano*X*X*X);

   double detK = K11*K22-K12*K12;
   pycuda::complex<double> Delta(1.-rho_1*rho_2*detK,-rho_1*K11-rho_2*K22);

   pycuda::complex<double> T11_hat = s_Kpi_palano/(svar_GeV-s_A_palano)*(K11-rho_2*detK)/Delta;
   pycuda::complex<double> T12_hat = s_Kpi_palano/(svar_GeV-s_A_palano)*K12/Delta;

   double xm = X;//(m-1175.)/425.;
   double alpha_1_s = 1.+c1_mass_swave*xm+c2_mass_swave*(2.*xm*xm-1.)+c3_mass_swave*(4.*xm*xm*xm-3.*xm)+c4_mass_swave*(8.*xm*xm*xm*xm-8.*xm*xm+1.);
   double alpha_2_s = c5_mass_swave+c6_mass_swave*xm+c7_mass_swave*(2.*xm*xm-1.)+c8_mass_swave*(4.*xm*xm*xm-3.*xm)+c9_mass_swave*(8.*xm*xm*xm*xm-8.*xm*xm+1.);

   pycuda::complex<double> T = alpha_1_s*T11_hat+alpha_2_s*T12_hat;

   return T*exp(i*3.06573);

 }

__device__ pycuda::complex<double> Prop_Lass(double m) { 

   pycuda::complex<double> i(0,1);

   double a_lass_ = 1./c1_mass_swave;
   double r_lass_ = c2_mass_swave;
   double m0_ = c3_mass_swave;
   double g0_ = c4_mass_swave;

   double q = get_q(m,MPion,MKaon);
   double q0 = get_q(m0_,MPion,MKaon);

   double cotg_deltaB = 1./(a_lass_*q)+0.5*r_lass_*q;
   double deltaB = atan(1./cotg_deltaB);
   pycuda::complex<double> expo = exp(i*2.*deltaB);

   double gamma = g0_*(q/q0)*(m0_/m);
   double cotg_deltaR = (m0_*m0_-m*m)/(m0_*gamma);

   pycuda::complex<double> T = 1./(cotg_deltaB-i)+expo/(cotg_deltaR-i);

   return T;

 }

__device__ pycuda::complex<double> Mji(double m, int ji) {

   pycuda::complex<double> i(0,1);
   pycuda::complex<double> T;

   if (ji == 0)
	{
	T = Prop_Stheo(m)*exp(-i*pycuda::arg(Prop_Stheo(mv)));
	}

   else if (ji == 1)
	{
	//T = Resonance(m,mv,gv,1)*exp(-i*pycuda::arg(Resonance(mv,mv,gv,1)));
	T = (Resonance(m,mv,gv,1)+pycuda::complex<double>(c5_mass_swave,c6_mass_swave)*Resonance(m,MKst_1_1410,GKst_1_1410,1)+pycuda::complex<double>(c7_mass_swave,c8_mass_swave)*Resonance(m,MKst_1_1680,GKst_1_1680,1))*exp(-i*pycuda::arg(Resonance(mv,mv,gv,1)+pycuda::complex<double>(c5_mass_swave,c6_mass_swave)*Resonance(mv,MKst_1_1410,GKst_1_1410,1)+pycuda::complex<double>(c7_mass_swave,c8_mass_swave)*Resonance(mv,MKst_1_1680,GKst_1_1680,1)));
	}

   else if (ji == 2)
	{
	T = Resonance(m,mt,gt,2)*exp(-i*pycuda::arg(Resonance(mv,mt,gt,2)));
	}
   
   return T;

 }
 
__device__ pycuda::complex<double> Mj1j2(double ma, double mb, int j1, int j2) { 

   double scale_factor = 1.;

   if ((j1 == 0) and (j2 == 0)) {return Mji(ma,0)*Mji(mb,0)*FL_j1j2(0,0,ma,mb)*(scale_factor/sqrt(Im00));}
   else if ((j1 == 0) and (j2 == 1)) {return Mji(ma,0)*Mji(mb,1)*FL_j1j2(0,1,ma,mb)*(scale_factor/sqrt(Im01));}
   else if ((j1 == 1) and (j2 == 0)) {return Mji(ma,1)*Mji(mb,0)*FL_j1j2(1,0,ma,mb)*(scale_factor/sqrt(Im10));}
   else if ((j1 == 0) and (j2 == 2)) {return Mji(ma,0)*Mji(mb,2)*FL_j1j2(0,2,ma,mb)*(scale_factor/sqrt(Im02));}
   else if ((j1 == 2) and (j2 == 0)) {return Mji(ma,2)*Mji(mb,0)*FL_j1j2(2,0,ma,mb)*(scale_factor/sqrt(Im20));}
   else if ((j1 == 1) and (j2 == 1)) {return Mji(ma,1)*Mji(mb,1)*FL_j1j2(1,1,ma,mb)*(scale_factor/sqrt(Im11));}
   else if ((j1 == 1) and (j2 == 2)) {return Mji(ma,1)*Mji(mb,2)*FL_j1j2(1,2,ma,mb)*(scale_factor/sqrt(Im12));}
   else if ((j1 == 2) and (j2 == 1)) {return Mji(ma,2)*Mji(mb,1)*FL_j1j2(2,1,ma,mb)*(scale_factor/sqrt(Im21));}
   else if ((j1 == 2) and (j2 == 2)) {return Mji(ma,2)*Mji(mb,2)*FL_j1j2(2,2,ma,mb)*(scale_factor/sqrt(Im22));}
   return pycuda::complex<double>(0.,0.);

 }

__device__ pycuda::complex<double> Mj1j2_unnorm(double ma, double mb, int j1, int j2) { 

   if ((j1 == 0) and (j2 == 0)) {return Mji(ma,0)*Mji(mb,0)*FL_j1j2(0,0,ma,mb);}
   else if ((j1 == 0) and (j2 == 1)) {return Mji(ma,0)*Mji(mb,1)*FL_j1j2(0,1,ma,mb);}
   else if ((j1 == 1) and (j2 == 0)) {return Mji(ma,1)*Mji(mb,0)*FL_j1j2(1,0,ma,mb);}
   else if ((j1 == 0) and (j2 == 2)) {return Mji(ma,0)*Mji(mb,2)*FL_j1j2(0,2,ma,mb);}
   else if ((j1 == 2) and (j2 == 0)) {return Mji(ma,2)*Mji(mb,0)*FL_j1j2(2,0,ma,mb);}
   else if ((j1 == 1) and (j2 == 1)) {return Mji(ma,1)*Mji(mb,1)*FL_j1j2(1,1,ma,mb);}
   else if ((j1 == 1) and (j2 == 2)) {return Mji(ma,1)*Mji(mb,2)*FL_j1j2(1,2,ma,mb);}
   else if ((j1 == 2) and (j2 == 1)) {return Mji(ma,2)*Mji(mb,1)*FL_j1j2(2,1,ma,mb);}
   else if ((j1 == 2) and (j2 == 2)) {return Mji(ma,2)*Mji(mb,2)*FL_j1j2(2,2,ma,mb);}
   return pycuda::complex<double>(0.,0.);

 }

__device__ double phasespace(double ma, double mb) {
 
   double Q1 = get_q(ma,MKaon,MPion);
   double Q2 = get_q(mb,MKaon,MPion);
   double QB = get_q(MBs,ma,mb);
   double phsp = Q1*Q2*QB;

   return phsp;

 }

__device__ pycuda::complex<double> hj1j2j1pj2p(int j1, int j2, int j1p, int j2p, int icat, int iev) { 

   return Mj1j2_temp(j1,j2,icat,iev)*pycuda::conj(Mj1j2_temp(j1p,j2p,icat,iev))*phasespace_temp[icat][iev];

 }


// ##########################################
// PDF elements

__device__ double comp_num_fit(int j1, int j2, int h, int j1p, int j2p, int hp, int icat, int iev) {

   return pycuda::real(Tj1j2hj1pj2php(j1,j2,h,j1p,j2p,hp,icat,iev)*Nj1j2hj1pj2php(j1,j2,h,j1p,j2p,hp)*hj1j2j1pj2p(j1,j2,j1p,j2p,icat,iev))*ghhp_phi(h,hp,icat,iev)*fjjphhp_cos1(j1,j1p,h,hp,icat,iev)*fjjphhp_cos2(j2,j2p,h,hp,icat,iev);

 }

__device__ double comp_den_fit(int j1, int j2, int h, int j1p, int j2p, int hp, int icat, int iev) { 
   return pycuda::real(ITj1j2hj1pj2php_deltat(j1,j2,h,j1p,j2p,hp,icat,iev)*pycuda::complex<double>(nw_comp(year_opt[icat],trig_opt[icat],j1,j2,h,j1p,j2p,hp,0),nw_comp(year_opt[icat],trig_opt[icat],j1,j2,h,j1p,j2p,hp,1)));
 }

__device__ double num_fit(int icat, int iev) {

   return comp_num_fit(0,0,0,0,0,0,icat,iev)+comp_num_fit(0,1,0,0,1,0,icat,iev)+comp_num_fit(0,2,0,0,2,0,icat,iev)+comp_num_fit(1,0,0,1,0,0,icat,iev)+comp_num_fit(1,1,0,1,1,0,icat,iev)+comp_num_fit(1,1,1,1,1,1,icat,iev)
+comp_num_fit(1,1,2,1,1,2,icat,iev)+comp_num_fit(1,2,0,1,2,0,icat,iev)+comp_num_fit(1,2,1,1,2,1,icat,iev)+comp_num_fit(1,2,2,1,2,2,icat,iev)+comp_num_fit(2,0,0,2,0,0,icat,iev)+comp_num_fit(2,1,0,2,1,0,icat,iev)+comp_num_fit(2,1,1,2,1,1,icat,iev)
+comp_num_fit(2,1,2,2,1,2,icat,iev)+comp_num_fit(2,2,0,2,2,0,icat,iev)+comp_num_fit(2,2,1,2,2,1,icat,iev)+comp_num_fit(2,2,2,2,2,2,icat,iev)+comp_num_fit(2,2,3,2,2,3,icat,iev)+comp_num_fit(2,2,4,2,2,4,icat,iev)+2.*comp_num_fit(0,1,0,0,0,0,icat,iev)
+2.*comp_num_fit(0,1,0,1,0,0,icat,iev)+2.*comp_num_fit(0,1,0,2,0,0,icat,iev)+2.*comp_num_fit(0,2,0,0,0,0,icat,iev)+2.*comp_num_fit(0,2,0,0,1,0,icat,iev)+2.*comp_num_fit(0,2,0,1,0,0,icat,iev)+2.*comp_num_fit(0,2,0,1,1,0,icat,iev)
+2.*comp_num_fit(0,2,0,2,0,0,icat,iev)+2.*comp_num_fit(0,2,0,2,1,0,icat,iev)+2.*comp_num_fit(1,0,0,0,0,0,icat,iev)+2.*comp_num_fit(1,1,0,0,0,0,icat,iev)+2.*comp_num_fit(1,1,0,0,1,0,icat,iev)+2.*comp_num_fit(1,1,0,1,0,0,icat,iev)
+2.*comp_num_fit(1,1,0,2,0,0,icat,iev)+2.*comp_num_fit(1,1,1,0,0,0,icat,iev)+2.*comp_num_fit(1,1,1,0,1,0,icat,iev)+2.*comp_num_fit(1,1,1,0,2,0,icat,iev)+2.*comp_num_fit(1,1,1,1,0,0,icat,iev)+2.*comp_num_fit(1,1,1,1,1,0,icat,iev)
+2.*comp_num_fit(1,1,1,1,2,0,icat,iev)+2.*comp_num_fit(1,1,1,2,0,0,icat,iev)+2.*comp_num_fit(1,1,1,2,1,0,icat,iev)+2.*comp_num_fit(1,1,1,2,2,0,icat,iev)+2.*comp_num_fit(1,1,2,0,0,0,icat,iev)+2.*comp_num_fit(1,1,2,0,1,0,icat,iev)
+2.*comp_num_fit(1,1,2,0,2,0,icat,iev)+2.*comp_num_fit(1,1,2,1,0,0,icat,iev)+2.*comp_num_fit(1,1,2,1,1,0,icat,iev)+2.*comp_num_fit(1,1,2,1,1,1,icat,iev)+2.*comp_num_fit(1,1,2,1,2,0,icat,iev)+2.*comp_num_fit(1,1,2,1,2,1,icat,iev)
+2.*comp_num_fit(1,1,2,2,0,0,icat,iev)+2.*comp_num_fit(1,1,2,2,1,0,icat,iev)+2.*comp_num_fit(1,1,2,2,1,1,icat,iev)+2.*comp_num_fit(1,1,2,2,2,0,icat,iev)+2.*comp_num_fit(1,1,2,2,2,1,icat,iev)+2.*comp_num_fit(1,2,0,0,0,0,icat,iev)
+2.*comp_num_fit(1,2,0,0,1,0,icat,iev)+2.*comp_num_fit(1,2,0,0,2,0,icat,iev)+2.*comp_num_fit(1,2,0,1,0,0,icat,iev)+2.*comp_num_fit(1,2,0,1,1,0,icat,iev)+2.*comp_num_fit(1,2,0,2,0,0,icat,iev)+2.*comp_num_fit(1,2,0,2,1,0,icat,iev)
+2.*comp_num_fit(1,2,1,0,0,0,icat,iev)+2.*comp_num_fit(1,2,1,0,1,0,icat,iev)+2.*comp_num_fit(1,2,1,0,2,0,icat,iev)+2.*comp_num_fit(1,2,1,1,0,0,icat,iev)+2.*comp_num_fit(1,2,1,1,1,0,icat,iev)+2.*comp_num_fit(1,2,1,1,1,1,icat,iev)
+2.*comp_num_fit(1,2,1,1,2,0,icat,iev)+2.*comp_num_fit(1,2,1,2,0,0,icat,iev)+2.*comp_num_fit(1,2,1,2,1,0,icat,iev)+2.*comp_num_fit(1,2,1,2,1,1,icat,iev)+2.*comp_num_fit(1,2,1,2,2,0,icat,iev)+2.*comp_num_fit(1,2,2,0,0,0,icat,iev)
+2.*comp_num_fit(1,2,2,0,1,0,icat,iev)+2.*comp_num_fit(1,2,2,0,2,0,icat,iev)+2.*comp_num_fit(1,2,2,1,0,0,icat,iev)+2.*comp_num_fit(1,2,2,1,1,0,icat,iev)+2.*comp_num_fit(1,2,2,1,1,1,icat,iev)+2.*comp_num_fit(1,2,2,1,1,2,icat,iev)
+2.*comp_num_fit(1,2,2,1,2,0,icat,iev)+2.*comp_num_fit(1,2,2,1,2,1,icat,iev)+2.*comp_num_fit(1,2,2,2,0,0,icat,iev)+2.*comp_num_fit(1,2,2,2,1,0,icat,iev)+2.*comp_num_fit(1,2,2,2,1,1,icat,iev)+2.*comp_num_fit(1,2,2,2,1,2,icat,iev)
+2.*comp_num_fit(1,2,2,2,2,0,icat,iev)+2.*comp_num_fit(1,2,2,2,2,1,icat,iev)+2.*comp_num_fit(2,0,0,0,0,0,icat,iev)+2.*comp_num_fit(2,0,0,1,0,0,icat,iev)+2.*comp_num_fit(2,1,0,0,0,0,icat,iev)+2.*comp_num_fit(2,1,0,0,1,0,icat,iev)
+2.*comp_num_fit(2,1,0,1,0,0,icat,iev)+2.*comp_num_fit(2,1,0,1,1,0,icat,iev)+2.*comp_num_fit(2,1,0,2,0,0,icat,iev)+2.*comp_num_fit(2,1,1,0,0,0,icat,iev)+2.*comp_num_fit(2,1,1,0,1,0,icat,iev)+2.*comp_num_fit(2,1,1,0,2,0,icat,iev)
+2.*comp_num_fit(2,1,1,1,0,0,icat,iev)+2.*comp_num_fit(2,1,1,1,1,0,icat,iev)+2.*comp_num_fit(2,1,1,1,1,1,icat,iev)+2.*comp_num_fit(2,1,1,1,2,0,icat,iev)+2.*comp_num_fit(2,1,1,2,0,0,icat,iev)+2.*comp_num_fit(2,1,1,2,1,0,icat,iev)
+2.*comp_num_fit(2,1,1,2,2,0,icat,iev)+2.*comp_num_fit(2,1,2,0,0,0,icat,iev)+2.*comp_num_fit(2,1,2,0,1,0,icat,iev)+2.*comp_num_fit(2,1,2,0,2,0,icat,iev)+2.*comp_num_fit(2,1,2,1,0,0,icat,iev)+2.*comp_num_fit(2,1,2,1,1,0,icat,iev)
+2.*comp_num_fit(2,1,2,1,1,1,icat,iev)+2.*comp_num_fit(2,1,2,1,1,2,icat,iev)+2.*comp_num_fit(2,1,2,1,2,0,icat,iev)+2.*comp_num_fit(2,1,2,1,2,1,icat,iev)+2.*comp_num_fit(2,1,2,2,0,0,icat,iev)+2.*comp_num_fit(2,1,2,2,1,0,icat,iev)
+2.*comp_num_fit(2,1,2,2,1,1,icat,iev)+2.*comp_num_fit(2,1,2,2,2,0,icat,iev)+2.*comp_num_fit(2,1,2,2,2,1,icat,iev)+2.*comp_num_fit(2,2,0,0,0,0,icat,iev)+2.*comp_num_fit(2,2,0,0,1,0,icat,iev)+2.*comp_num_fit(2,2,0,0,2,0,icat,iev)
+2.*comp_num_fit(2,2,0,1,0,0,icat,iev)+2.*comp_num_fit(2,2,0,1,1,0,icat,iev)+2.*comp_num_fit(2,2,0,1,2,0,icat,iev)+2.*comp_num_fit(2,2,0,2,0,0,icat,iev)+2.*comp_num_fit(2,2,0,2,1,0,icat,iev)+2.*comp_num_fit(2,2,1,0,0,0,icat,iev)
+2.*comp_num_fit(2,2,1,0,1,0,icat,iev)+2.*comp_num_fit(2,2,1,0,2,0,icat,iev)+2.*comp_num_fit(2,2,1,1,0,0,icat,iev)+2.*comp_num_fit(2,2,1,1,1,0,icat,iev)+2.*comp_num_fit(2,2,1,1,1,1,icat,iev)+2.*comp_num_fit(2,2,1,1,2,0,icat,iev)
+2.*comp_num_fit(2,2,1,1,2,1,icat,iev)+2.*comp_num_fit(2,2,1,2,0,0,icat,iev)+2.*comp_num_fit(2,2,1,2,1,0,icat,iev)+2.*comp_num_fit(2,2,1,2,1,1,icat,iev)+2.*comp_num_fit(2,2,1,2,2,0,icat,iev)+2.*comp_num_fit(2,2,2,0,0,0,icat,iev)
+2.*comp_num_fit(2,2,2,0,1,0,icat,iev)+2.*comp_num_fit(2,2,2,0,2,0,icat,iev)+2.*comp_num_fit(2,2,2,1,0,0,icat,iev)+2.*comp_num_fit(2,2,2,1,1,0,icat,iev)+2.*comp_num_fit(2,2,2,1,1,1,icat,iev)+2.*comp_num_fit(2,2,2,1,1,2,icat,iev)
+2.*comp_num_fit(2,2,2,1,2,0,icat,iev)+2.*comp_num_fit(2,2,2,1,2,1,icat,iev)+2.*comp_num_fit(2,2,2,1,2,2,icat,iev)+2.*comp_num_fit(2,2,2,2,0,0,icat,iev)+2.*comp_num_fit(2,2,2,2,1,0,icat,iev)+2.*comp_num_fit(2,2,2,2,1,1,icat,iev)
+2.*comp_num_fit(2,2,2,2,1,2,icat,iev)+2.*comp_num_fit(2,2,2,2,2,0,icat,iev)+2.*comp_num_fit(2,2,2,2,2,1,icat,iev)+2.*comp_num_fit(2,2,3,0,0,0,icat,iev)+2.*comp_num_fit(2,2,3,0,1,0,icat,iev)+2.*comp_num_fit(2,2,3,0,2,0,icat,iev)
+2.*comp_num_fit(2,2,3,1,0,0,icat,iev)+2.*comp_num_fit(2,2,3,1,1,0,icat,iev)+2.*comp_num_fit(2,2,3,1,1,1,icat,iev)+2.*comp_num_fit(2,2,3,1,1,2,icat,iev)+2.*comp_num_fit(2,2,3,1,2,0,icat,iev)+2.*comp_num_fit(2,2,3,1,2,1,icat,iev)
+2.*comp_num_fit(2,2,3,1,2,2,icat,iev)+2.*comp_num_fit(2,2,3,2,0,0,icat,iev)+2.*comp_num_fit(2,2,3,2,1,0,icat,iev)+2.*comp_num_fit(2,2,3,2,1,1,icat,iev)+2.*comp_num_fit(2,2,3,2,1,2,icat,iev)+2.*comp_num_fit(2,2,3,2,2,0,icat,iev)
+2.*comp_num_fit(2,2,3,2,2,1,icat,iev)+2.*comp_num_fit(2,2,3,2,2,2,icat,iev)+2.*comp_num_fit(2,2,4,0,0,0,icat,iev)+2.*comp_num_fit(2,2,4,0,1,0,icat,iev)+2.*comp_num_fit(2,2,4,0,2,0,icat,iev)+2.*comp_num_fit(2,2,4,1,0,0,icat,iev)
+2.*comp_num_fit(2,2,4,1,1,0,icat,iev)+2.*comp_num_fit(2,2,4,1,1,1,icat,iev)+2.*comp_num_fit(2,2,4,1,1,2,icat,iev)+2.*comp_num_fit(2,2,4,1,2,0,icat,iev)+2.*comp_num_fit(2,2,4,1,2,1,icat,iev)+2.*comp_num_fit(2,2,4,1,2,2,icat,iev)
+2.*comp_num_fit(2,2,4,2,0,0,icat,iev)+2.*comp_num_fit(2,2,4,2,1,0,icat,iev)+2.*comp_num_fit(2,2,4,2,1,1,icat,iev)+2.*comp_num_fit(2,2,4,2,1,2,icat,iev)+2.*comp_num_fit(2,2,4,2,2,0,icat,iev)+2.*comp_num_fit(2,2,4,2,2,1,icat,iev)
+2.*comp_num_fit(2,2,4,2,2,2,icat,iev)+2.*comp_num_fit(2,2,4,2,2,3,icat,iev);

 }

__device__ double den_fit(int icat, int iev) {

   return comp_den_fit(0,0,0,0,0,0,icat,iev)+comp_den_fit(0,1,0,0,1,0,icat,iev)+comp_den_fit(0,2,0,0,2,0,icat,iev)+comp_den_fit(1,0,0,1,0,0,icat,iev)+comp_den_fit(1,1,0,1,1,0,icat,iev)+comp_den_fit(1,1,1,1,1,1,icat,iev)
+comp_den_fit(1,1,2,1,1,2,icat,iev)+comp_den_fit(1,2,0,1,2,0,icat,iev)+comp_den_fit(1,2,1,1,2,1,icat,iev)+comp_den_fit(1,2,2,1,2,2,icat,iev)+comp_den_fit(2,0,0,2,0,0,icat,iev)+comp_den_fit(2,1,0,2,1,0,icat,iev)+comp_den_fit(2,1,1,2,1,1,icat,iev)
+comp_den_fit(2,1,2,2,1,2,icat,iev)+comp_den_fit(2,2,0,2,2,0,icat,iev)+comp_den_fit(2,2,1,2,2,1,icat,iev)+comp_den_fit(2,2,2,2,2,2,icat,iev)+comp_den_fit(2,2,3,2,2,3,icat,iev)+comp_den_fit(2,2,4,2,2,4,icat,iev)+2.*comp_den_fit(0,1,0,0,0,0,icat,iev)
+2.*comp_den_fit(0,1,0,1,0,0,icat,iev)+2.*comp_den_fit(0,1,0,2,0,0,icat,iev)+2.*comp_den_fit(0,2,0,0,0,0,icat,iev)+2.*comp_den_fit(0,2,0,0,1,0,icat,iev)+2.*comp_den_fit(0,2,0,1,0,0,icat,iev)+2.*comp_den_fit(0,2,0,1,1,0,icat,iev)
+2.*comp_den_fit(0,2,0,2,0,0,icat,iev)+2.*comp_den_fit(0,2,0,2,1,0,icat,iev)+2.*comp_den_fit(1,0,0,0,0,0,icat,iev)+2.*comp_den_fit(1,1,0,0,0,0,icat,iev)+2.*comp_den_fit(1,1,0,0,1,0,icat,iev)+2.*comp_den_fit(1,1,0,1,0,0,icat,iev)
+2.*comp_den_fit(1,1,0,2,0,0,icat,iev)+2.*comp_den_fit(1,1,1,0,0,0,icat,iev)+2.*comp_den_fit(1,1,1,0,1,0,icat,iev)+2.*comp_den_fit(1,1,1,0,2,0,icat,iev)+2.*comp_den_fit(1,1,1,1,0,0,icat,iev)+2.*comp_den_fit(1,1,1,1,1,0,icat,iev)
+2.*comp_den_fit(1,1,1,1,2,0,icat,iev)+2.*comp_den_fit(1,1,1,2,0,0,icat,iev)+2.*comp_den_fit(1,1,1,2,1,0,icat,iev)+2.*comp_den_fit(1,1,1,2,2,0,icat,iev)+2.*comp_den_fit(1,1,2,0,0,0,icat,iev)+2.*comp_den_fit(1,1,2,0,1,0,icat,iev)
+2.*comp_den_fit(1,1,2,0,2,0,icat,iev)+2.*comp_den_fit(1,1,2,1,0,0,icat,iev)+2.*comp_den_fit(1,1,2,1,1,0,icat,iev)+2.*comp_den_fit(1,1,2,1,1,1,icat,iev)+2.*comp_den_fit(1,1,2,1,2,0,icat,iev)+2.*comp_den_fit(1,1,2,1,2,1,icat,iev)
+2.*comp_den_fit(1,1,2,2,0,0,icat,iev)+2.*comp_den_fit(1,1,2,2,1,0,icat,iev)+2.*comp_den_fit(1,1,2,2,1,1,icat,iev)+2.*comp_den_fit(1,1,2,2,2,0,icat,iev)+2.*comp_den_fit(1,1,2,2,2,1,icat,iev)+2.*comp_den_fit(1,2,0,0,0,0,icat,iev)
+2.*comp_den_fit(1,2,0,0,1,0,icat,iev)+2.*comp_den_fit(1,2,0,0,2,0,icat,iev)+2.*comp_den_fit(1,2,0,1,0,0,icat,iev)+2.*comp_den_fit(1,2,0,1,1,0,icat,iev)+2.*comp_den_fit(1,2,0,2,0,0,icat,iev)+2.*comp_den_fit(1,2,0,2,1,0,icat,iev)
+2.*comp_den_fit(1,2,1,0,0,0,icat,iev)+2.*comp_den_fit(1,2,1,0,1,0,icat,iev)+2.*comp_den_fit(1,2,1,0,2,0,icat,iev)+2.*comp_den_fit(1,2,1,1,0,0,icat,iev)+2.*comp_den_fit(1,2,1,1,1,0,icat,iev)+2.*comp_den_fit(1,2,1,1,1,1,icat,iev)
+2.*comp_den_fit(1,2,1,1,2,0,icat,iev)+2.*comp_den_fit(1,2,1,2,0,0,icat,iev)+2.*comp_den_fit(1,2,1,2,1,0,icat,iev)+2.*comp_den_fit(1,2,1,2,1,1,icat,iev)+2.*comp_den_fit(1,2,1,2,2,0,icat,iev)+2.*comp_den_fit(1,2,2,0,0,0,icat,iev)
+2.*comp_den_fit(1,2,2,0,1,0,icat,iev)+2.*comp_den_fit(1,2,2,0,2,0,icat,iev)+2.*comp_den_fit(1,2,2,1,0,0,icat,iev)+2.*comp_den_fit(1,2,2,1,1,0,icat,iev)+2.*comp_den_fit(1,2,2,1,1,1,icat,iev)+2.*comp_den_fit(1,2,2,1,1,2,icat,iev)
+2.*comp_den_fit(1,2,2,1,2,0,icat,iev)+2.*comp_den_fit(1,2,2,1,2,1,icat,iev)+2.*comp_den_fit(1,2,2,2,0,0,icat,iev)+2.*comp_den_fit(1,2,2,2,1,0,icat,iev)+2.*comp_den_fit(1,2,2,2,1,1,icat,iev)+2.*comp_den_fit(1,2,2,2,1,2,icat,iev)
+2.*comp_den_fit(1,2,2,2,2,0,icat,iev)+2.*comp_den_fit(1,2,2,2,2,1,icat,iev)+2.*comp_den_fit(2,0,0,0,0,0,icat,iev)+2.*comp_den_fit(2,0,0,1,0,0,icat,iev)+2.*comp_den_fit(2,1,0,0,0,0,icat,iev)+2.*comp_den_fit(2,1,0,0,1,0,icat,iev)
+2.*comp_den_fit(2,1,0,1,0,0,icat,iev)+2.*comp_den_fit(2,1,0,1,1,0,icat,iev)+2.*comp_den_fit(2,1,0,2,0,0,icat,iev)+2.*comp_den_fit(2,1,1,0,0,0,icat,iev)+2.*comp_den_fit(2,1,1,0,1,0,icat,iev)+2.*comp_den_fit(2,1,1,0,2,0,icat,iev)
+2.*comp_den_fit(2,1,1,1,0,0,icat,iev)+2.*comp_den_fit(2,1,1,1,1,0,icat,iev)+2.*comp_den_fit(2,1,1,1,1,1,icat,iev)+2.*comp_den_fit(2,1,1,1,2,0,icat,iev)+2.*comp_den_fit(2,1,1,2,0,0,icat,iev)+2.*comp_den_fit(2,1,1,2,1,0,icat,iev)
+2.*comp_den_fit(2,1,1,2,2,0,icat,iev)+2.*comp_den_fit(2,1,2,0,0,0,icat,iev)+2.*comp_den_fit(2,1,2,0,1,0,icat,iev)+2.*comp_den_fit(2,1,2,0,2,0,icat,iev)+2.*comp_den_fit(2,1,2,1,0,0,icat,iev)+2.*comp_den_fit(2,1,2,1,1,0,icat,iev)
+2.*comp_den_fit(2,1,2,1,1,1,icat,iev)+2.*comp_den_fit(2,1,2,1,1,2,icat,iev)+2.*comp_den_fit(2,1,2,1,2,0,icat,iev)+2.*comp_den_fit(2,1,2,1,2,1,icat,iev)+2.*comp_den_fit(2,1,2,2,0,0,icat,iev)+2.*comp_den_fit(2,1,2,2,1,0,icat,iev)
+2.*comp_den_fit(2,1,2,2,1,1,icat,iev)+2.*comp_den_fit(2,1,2,2,2,0,icat,iev)+2.*comp_den_fit(2,1,2,2,2,1,icat,iev)+2.*comp_den_fit(2,2,0,0,0,0,icat,iev)+2.*comp_den_fit(2,2,0,0,1,0,icat,iev)+2.*comp_den_fit(2,2,0,0,2,0,icat,iev)
+2.*comp_den_fit(2,2,0,1,0,0,icat,iev)+2.*comp_den_fit(2,2,0,1,1,0,icat,iev)+2.*comp_den_fit(2,2,0,1,2,0,icat,iev)+2.*comp_den_fit(2,2,0,2,0,0,icat,iev)+2.*comp_den_fit(2,2,0,2,1,0,icat,iev)+2.*comp_den_fit(2,2,1,0,0,0,icat,iev)
+2.*comp_den_fit(2,2,1,0,1,0,icat,iev)+2.*comp_den_fit(2,2,1,0,2,0,icat,iev)+2.*comp_den_fit(2,2,1,1,0,0,icat,iev)+2.*comp_den_fit(2,2,1,1,1,0,icat,iev)+2.*comp_den_fit(2,2,1,1,1,1,icat,iev)+2.*comp_den_fit(2,2,1,1,2,0,icat,iev)
+2.*comp_den_fit(2,2,1,1,2,1,icat,iev)+2.*comp_den_fit(2,2,1,2,0,0,icat,iev)+2.*comp_den_fit(2,2,1,2,1,0,icat,iev)+2.*comp_den_fit(2,2,1,2,1,1,icat,iev)+2.*comp_den_fit(2,2,1,2,2,0,icat,iev)+2.*comp_den_fit(2,2,2,0,0,0,icat,iev)
+2.*comp_den_fit(2,2,2,0,1,0,icat,iev)+2.*comp_den_fit(2,2,2,0,2,0,icat,iev)+2.*comp_den_fit(2,2,2,1,0,0,icat,iev)+2.*comp_den_fit(2,2,2,1,1,0,icat,iev)+2.*comp_den_fit(2,2,2,1,1,1,icat,iev)+2.*comp_den_fit(2,2,2,1,1,2,icat,iev)
+2.*comp_den_fit(2,2,2,1,2,0,icat,iev)+2.*comp_den_fit(2,2,2,1,2,1,icat,iev)+2.*comp_den_fit(2,2,2,1,2,2,icat,iev)+2.*comp_den_fit(2,2,2,2,0,0,icat,iev)+2.*comp_den_fit(2,2,2,2,1,0,icat,iev)+2.*comp_den_fit(2,2,2,2,1,1,icat,iev)
+2.*comp_den_fit(2,2,2,2,1,2,icat,iev)+2.*comp_den_fit(2,2,2,2,2,0,icat,iev)+2.*comp_den_fit(2,2,2,2,2,1,icat,iev)+2.*comp_den_fit(2,2,3,0,0,0,icat,iev)+2.*comp_den_fit(2,2,3,0,1,0,icat,iev)+2.*comp_den_fit(2,2,3,0,2,0,icat,iev)
+2.*comp_den_fit(2,2,3,1,0,0,icat,iev)+2.*comp_den_fit(2,2,3,1,1,0,icat,iev)+2.*comp_den_fit(2,2,3,1,1,1,icat,iev)+2.*comp_den_fit(2,2,3,1,1,2,icat,iev)+2.*comp_den_fit(2,2,3,1,2,0,icat,iev)+2.*comp_den_fit(2,2,3,1,2,1,icat,iev)
+2.*comp_den_fit(2,2,3,1,2,2,icat,iev)+2.*comp_den_fit(2,2,3,2,0,0,icat,iev)+2.*comp_den_fit(2,2,3,2,1,0,icat,iev)+2.*comp_den_fit(2,2,3,2,1,1,icat,iev)+2.*comp_den_fit(2,2,3,2,1,2,icat,iev)+2.*comp_den_fit(2,2,3,2,2,0,icat,iev)
+2.*comp_den_fit(2,2,3,2,2,1,icat,iev)+2.*comp_den_fit(2,2,3,2,2,2,icat,iev)+2.*comp_den_fit(2,2,4,0,0,0,icat,iev)+2.*comp_den_fit(2,2,4,0,1,0,icat,iev)+2.*comp_den_fit(2,2,4,0,2,0,icat,iev)+2.*comp_den_fit(2,2,4,1,0,0,icat,iev)
+2.*comp_den_fit(2,2,4,1,1,0,icat,iev)+2.*comp_den_fit(2,2,4,1,1,1,icat,iev)+2.*comp_den_fit(2,2,4,1,1,2,icat,iev)+2.*comp_den_fit(2,2,4,1,2,0,icat,iev)+2.*comp_den_fit(2,2,4,1,2,1,icat,iev)+2.*comp_den_fit(2,2,4,1,2,2,icat,iev)
+2.*comp_den_fit(2,2,4,2,0,0,icat,iev)+2.*comp_den_fit(2,2,4,2,1,0,icat,iev)+2.*comp_den_fit(2,2,4,2,1,1,icat,iev)+2.*comp_den_fit(2,2,4,2,1,2,icat,iev)+2.*comp_den_fit(2,2,4,2,2,0,icat,iev)+2.*comp_den_fit(2,2,4,2,2,1,icat,iev)
+2.*comp_den_fit(2,2,4,2,2,2,icat,iev)+2.*comp_den_fit(2,2,4,2,2,3,icat,iev);

 }


// ##########################################
// PDF evaluator

__device__ void set_buffer_options(double *options, int icat) {

   wide_window = (int) options[0];
   year_opt[icat] = (int) options[1];
   trig_opt[icat] = (int) options[2];
   alt_fit = options[3];
   option = options[4];
   inftres = options[5];
   acctype = options[6];
   A_j1 = options[7];
   A_j2 = options[8];
   A_h = options[9];
   A_j1p = options[10];
   A_j2p = options[11];
   A_hp = options[12];
   qcomp = options[13];

 }

__device__ void set_buffer_amplitudes(double *re_amps, double *dirCP_asyms, double *im_amps, double *weak_phases, double *mixing_params, double *calib_params) {

   reA00 = re_amps[0];
   reA01 = re_amps[1];
   reA10 = re_amps[2];
   reA02 = re_amps[3];
   reA20 = re_amps[4];
   reA110 = re_amps[5];
   reA11par = re_amps[6];
   reA11perp = re_amps[7];
   reA120 = re_amps[8];
   reA12par = re_amps[9];
   reA12perp = re_amps[10];
   reA210 = re_amps[11];
   reA21par = re_amps[12];
   reA21perp = re_amps[13];
   reA220 = re_amps[14];
   reA22par = re_amps[15];
   reA22perp = re_amps[16];
   reA22par2 = re_amps[17];
   reA22perp2 = re_amps[18];

   DCP_SS = dirCP_asyms[0];
   DCP_SV = dirCP_asyms[1];
   DCP_VS = dirCP_asyms[2];
   DCP_ST = dirCP_asyms[3];
   DCP_TS = dirCP_asyms[4];
   DCP = dirCP_asyms[5];
   DCP_VT = dirCP_asyms[6];
   DCP_TV = dirCP_asyms[7];
   DCP_TT = dirCP_asyms[8];

   imA00 = im_amps[0];
   imA01 = im_amps[1];
   imA10 = im_amps[2];
   imA02 = im_amps[3];
   imA20 = im_amps[4];
   imA110 = im_amps[5];
   imA11par = im_amps[6];
   imA11perp = im_amps[7];
   imA120 = im_amps[8];
   imA12par = im_amps[9];
   imA12perp = im_amps[10];
   imA210 = im_amps[11];
   imA21par = im_amps[12];
   imA21perp = im_amps[13];
   imA220 = im_amps[14];
   imA22par = im_amps[15];
   imA22perp = im_amps[16];
   imA22par2 = im_amps[17];
   imA22perp2 = im_amps[18];

   phis = weak_phases[0];
   dphi_SS = weak_phases[1];
   dphi_SV = weak_phases[2];
   dphi_VS = weak_phases[3];
   dphi_ST = weak_phases[4];
   dphi_TS = weak_phases[5];
   dphi_VT = weak_phases[6];
   dphi_TV = weak_phases[7];
   dphi_TT = weak_phases[8];

   delta_m_freq = mixing_params[0];
   gamma_Bs_freq = mixing_params[1];
   delta_gamma_freq = mixing_params[2];

   p0metac_tag_SSK = calib_params[0];
   p0metac_tag_OS = calib_params[1];
   Dp0half_tag_SSK = calib_params[2];
   Dp0half_tag_OS = calib_params[3];
   p1_tag_SSK = calib_params[4];
   p1_tag_OS = calib_params[5];
   Dp1half_tag_SSK = calib_params[6];
   Dp1half_tag_OS = calib_params[7];
   etac_tag_SSK = calib_params[8];
   etac_tag_OS = calib_params[9];
   deltatmean_tres_11 = calib_params[10];
   p0_tres_11 = calib_params[11];
   p1_tres_11 = calib_params[12];
   deltatmean_tres_12 = calib_params[13];
   p0_tres_12 = calib_params[14];
   p1_tres_12 = calib_params[15];
   mv = calib_params[16];
   ms = calib_params[17];
   mt = calib_params[18];
   gv = calib_params[19];
   gs = calib_params[20];
   gt = calib_params[21];
   c1_mass_swave = calib_params[22];
   c2_mass_swave = calib_params[23];
   c3_mass_swave = calib_params[24];
   c4_mass_swave = calib_params[25];
   c5_mass_swave = calib_params[26];
   c6_mass_swave = calib_params[27];
   c7_mass_swave = calib_params[28];
   c8_mass_swave = calib_params[29];
   c9_mass_swave = calib_params[30];
   res_mass = calib_params[31];

   if (wide_window == 0) {DCP_prod = -0.0101;}
   else {DCP_prod = -0.0072;}

   reAj1j2h_temp[0][0][0] = pycuda::real(Aj1j2h(0,0,0));
   reAj1j2h_temp[0][1][0] = pycuda::real(Aj1j2h(0,1,0));
   reAj1j2h_temp[1][0][0] = pycuda::real(Aj1j2h(1,0,0));
   reAj1j2h_temp[0][2][0] = pycuda::real(Aj1j2h(0,2,0));
   reAj1j2h_temp[2][0][0] = pycuda::real(Aj1j2h(2,0,0));
   reAj1j2h_temp[1][1][0] = pycuda::real(Aj1j2h(1,1,0));
   reAj1j2h_temp[1][1][1] = pycuda::real(Aj1j2h(1,1,1));
   reAj1j2h_temp[1][1][2] = pycuda::real(Aj1j2h(1,1,2));
   reAj1j2h_temp[1][2][0] = pycuda::real(Aj1j2h(1,2,0));
   reAj1j2h_temp[1][2][1] = pycuda::real(Aj1j2h(1,2,1));
   reAj1j2h_temp[1][2][2] = pycuda::real(Aj1j2h(1,2,2));
   reAj1j2h_temp[2][1][0] = pycuda::real(Aj1j2h(2,1,0));
   reAj1j2h_temp[2][1][1] = pycuda::real(Aj1j2h(2,1,1));
   reAj1j2h_temp[2][1][2] = pycuda::real(Aj1j2h(2,1,2));
   reAj1j2h_temp[2][2][0] = pycuda::real(Aj1j2h(2,2,0));
   reAj1j2h_temp[2][2][1] = pycuda::real(Aj1j2h(2,2,1));
   reAj1j2h_temp[2][2][2] = pycuda::real(Aj1j2h(2,2,2));
   reAj1j2h_temp[2][2][3] = pycuda::real(Aj1j2h(2,2,3));
   reAj1j2h_temp[2][2][4] = pycuda::real(Aj1j2h(2,2,4));
   imAj1j2h_temp[0][0][0] = pycuda::imag(Aj1j2h(0,0,0));
   imAj1j2h_temp[0][1][0] = pycuda::imag(Aj1j2h(0,1,0));
   imAj1j2h_temp[1][0][0] = pycuda::imag(Aj1j2h(1,0,0));
   imAj1j2h_temp[0][2][0] = pycuda::imag(Aj1j2h(0,2,0));
   imAj1j2h_temp[2][0][0] = pycuda::imag(Aj1j2h(2,0,0));
   imAj1j2h_temp[1][1][0] = pycuda::imag(Aj1j2h(1,1,0));
   imAj1j2h_temp[1][1][1] = pycuda::imag(Aj1j2h(1,1,1));
   imAj1j2h_temp[1][1][2] = pycuda::imag(Aj1j2h(1,1,2));
   imAj1j2h_temp[1][2][0] = pycuda::imag(Aj1j2h(1,2,0));
   imAj1j2h_temp[1][2][1] = pycuda::imag(Aj1j2h(1,2,1));
   imAj1j2h_temp[1][2][2] = pycuda::imag(Aj1j2h(1,2,2));
   imAj1j2h_temp[2][1][0] = pycuda::imag(Aj1j2h(2,1,0));
   imAj1j2h_temp[2][1][1] = pycuda::imag(Aj1j2h(2,1,1));
   imAj1j2h_temp[2][1][2] = pycuda::imag(Aj1j2h(2,1,2));
   imAj1j2h_temp[2][2][0] = pycuda::imag(Aj1j2h(2,2,0));
   imAj1j2h_temp[2][2][1] = pycuda::imag(Aj1j2h(2,2,1));
   imAj1j2h_temp[2][2][2] = pycuda::imag(Aj1j2h(2,2,2));
   imAj1j2h_temp[2][2][3] = pycuda::imag(Aj1j2h(2,2,3));
   imAj1j2h_temp[2][2][4] = pycuda::imag(Aj1j2h(2,2,4));
   reAbarj1j2h_temp[0][0][0] = pycuda::real(Abarj1j2h(0,0,0));
   reAbarj1j2h_temp[0][1][0] = pycuda::real(Abarj1j2h(0,1,0));
   reAbarj1j2h_temp[1][0][0] = pycuda::real(Abarj1j2h(1,0,0));
   reAbarj1j2h_temp[0][2][0] = pycuda::real(Abarj1j2h(0,2,0));
   reAbarj1j2h_temp[2][0][0] = pycuda::real(Abarj1j2h(2,0,0));
   reAbarj1j2h_temp[1][1][0] = pycuda::real(Abarj1j2h(1,1,0));
   reAbarj1j2h_temp[1][1][1] = pycuda::real(Abarj1j2h(1,1,1));
   reAbarj1j2h_temp[1][1][2] = pycuda::real(Abarj1j2h(1,1,2));
   reAbarj1j2h_temp[1][2][0] = pycuda::real(Abarj1j2h(1,2,0));
   reAbarj1j2h_temp[1][2][1] = pycuda::real(Abarj1j2h(1,2,1));
   reAbarj1j2h_temp[1][2][2] = pycuda::real(Abarj1j2h(1,2,2));
   reAbarj1j2h_temp[2][1][0] = pycuda::real(Abarj1j2h(2,1,0));
   reAbarj1j2h_temp[2][1][1] = pycuda::real(Abarj1j2h(2,1,1));
   reAbarj1j2h_temp[2][1][2] = pycuda::real(Abarj1j2h(2,1,2));
   reAbarj1j2h_temp[2][2][0] = pycuda::real(Abarj1j2h(2,2,0));
   reAbarj1j2h_temp[2][2][1] = pycuda::real(Abarj1j2h(2,2,1));
   reAbarj1j2h_temp[2][2][2] = pycuda::real(Abarj1j2h(2,2,2));
   reAbarj1j2h_temp[2][2][3] = pycuda::real(Abarj1j2h(2,2,3));
   reAbarj1j2h_temp[2][2][4] = pycuda::real(Abarj1j2h(2,2,4));
   imAbarj1j2h_temp[0][0][0] = pycuda::imag(Abarj1j2h(0,0,0));
   imAbarj1j2h_temp[0][1][0] = pycuda::imag(Abarj1j2h(0,1,0));
   imAbarj1j2h_temp[1][0][0] = pycuda::imag(Abarj1j2h(1,0,0));
   imAbarj1j2h_temp[0][2][0] = pycuda::imag(Abarj1j2h(0,2,0));
   imAbarj1j2h_temp[2][0][0] = pycuda::imag(Abarj1j2h(2,0,0));
   imAbarj1j2h_temp[1][1][0] = pycuda::imag(Abarj1j2h(1,1,0));
   imAbarj1j2h_temp[1][1][1] = pycuda::imag(Abarj1j2h(1,1,1));
   imAbarj1j2h_temp[1][1][2] = pycuda::imag(Abarj1j2h(1,1,2));
   imAbarj1j2h_temp[1][2][0] = pycuda::imag(Abarj1j2h(1,2,0));
   imAbarj1j2h_temp[1][2][1] = pycuda::imag(Abarj1j2h(1,2,1));
   imAbarj1j2h_temp[1][2][2] = pycuda::imag(Abarj1j2h(1,2,2));
   imAbarj1j2h_temp[2][1][0] = pycuda::imag(Abarj1j2h(2,1,0));
   imAbarj1j2h_temp[2][1][1] = pycuda::imag(Abarj1j2h(2,1,1));
   imAbarj1j2h_temp[2][1][2] = pycuda::imag(Abarj1j2h(2,1,2));
   imAbarj1j2h_temp[2][2][0] = pycuda::imag(Abarj1j2h(2,2,0));
   imAbarj1j2h_temp[2][2][1] = pycuda::imag(Abarj1j2h(2,2,1));
   imAbarj1j2h_temp[2][2][2] = pycuda::imag(Abarj1j2h(2,2,2));
   imAbarj1j2h_temp[2][2][3] = pycuda::imag(Abarj1j2h(2,2,3));
   imAbarj1j2h_temp[2][2][4] = pycuda::imag(Abarj1j2h(2,2,4));

 }

__device__ void set_buffer_differential_terms(double *mass_integrals, int icat, int iev) {

   Im00 = mass_integrals[0];
   Im01 = mass_integrals[1];
   Im10 = mass_integrals[2];
   Im02 = mass_integrals[3];
   Im20 = mass_integrals[4];
   Im11 = mass_integrals[5];
   Im12 = mass_integrals[6];
   Im21 = mass_integrals[7];
   Im22 = mass_integrals[8];

   double f1,f2,s1,s2,x1,x2;

   if (acctype == 3) {
      f1 = 1.;
      f2 = 0.;
      s1 = p0_tres_12+p1_tres_12*(t_err[icat][iev]-deltatmean_tres_12);
      s2 = 1.;
      x1 = t[icat][iev]/(sqrt(2.)*s1);
      x2 = t[icat][iev]/(sqrt(2.)*s2);
   }
   else {
      f1 = 1.;
      f2 = 0.;
      if (year_opt == 0) {s1 = p0_tres_11+p1_tres_11*(t_err[icat][iev]-deltatmean_tres_11);}
      else {s1 = p0_tres_12+p1_tres_12*(t_err[icat][iev]-deltatmean_tres_12);}
      s2 = 1.;
      x1 = t[icat][iev]/(sqrt(2.)*s1);
      x2 = t[icat][iev]/(sqrt(2.)*s2);
   }

   pycuda::complex<double> z1_hyper_plus = s1/sqrt(2.)*pycuda::complex<double>(gamma_Bs_freq-0.5*delta_gamma_freq,0.);
   pycuda::complex<double> z2_hyper_plus = s2/sqrt(2.)*pycuda::complex<double>(gamma_Bs_freq-0.5*delta_gamma_freq,0.);
   pycuda::complex<double> z1_hyper_minus = s1/sqrt(2.)*pycuda::complex<double>(gamma_Bs_freq+0.5*delta_gamma_freq,0.);
   pycuda::complex<double> z2_hyper_minus = s2/sqrt(2.)*pycuda::complex<double>(gamma_Bs_freq+0.5*delta_gamma_freq,0.);
   pycuda::complex<double> z1_trigo = s1/sqrt(2.)*pycuda::complex<double>(gamma_Bs_freq,-delta_m_freq);
   pycuda::complex<double> z2_trigo = s2/sqrt(2.)*pycuda::complex<double>(gamma_Bs_freq,-delta_m_freq);

   double conv_exp_hyper_plus = pycuda::real(f1*conv_exp(x1,z1_hyper_plus)+f2*conv_exp(x2,z2_hyper_plus));
   double conv_exp_hyper_minus = pycuda::real(f1*conv_exp(x1,z1_hyper_minus)+f2*conv_exp(x2,z2_hyper_minus));
   pycuda::complex<double> conv_exp_trigo = f1*conv_exp(x1,z1_trigo)+f2*conv_exp(x2,z2_trigo);

   T_cosh_temp[icat][iev] = 0.5*(conv_exp_hyper_plus + conv_exp_hyper_minus);
   T_sinh_temp[icat][iev] = 0.5*(conv_exp_hyper_plus - conv_exp_hyper_minus);
   T_cos_temp[icat][iev] = pycuda::real(conv_exp_trigo);
   T_sin_temp[icat][iev] = pycuda::imag(conv_exp_trigo);

   zeta_temp[icat][iev] = zeta(decision_SSK[icat][iev],decision_OS[icat][iev],etamistag_SSK[icat][iev],etamistag_OS[icat][iev]);
   DCP_tzero_temp[icat][iev] = DCP_tzero(decision_SSK[icat][iev],decision_OS[icat][iev],etamistag_SSK[icat][iev],etamistag_OS[icat][iev]);

   for (int i=0; i<18; ++i) {fi_cos1_temp[i][icat][iev] = fi(cos1[icat][iev],i+1);}
   for (int i=0; i<18; ++i) {fi_cos2_temp[i][icat][iev] = fi(cos2[icat][iev],i+1);}
   for (int i=0; i<15; ++i) {gi_temp[i][icat][iev] = gi(phi[icat][iev],i+1);}

   for (int j1=0; j1<3; ++j1) {
      for (int j2=0; j2<3; ++j2) {
         pycuda::complex<double> M_temp = Mj1j2(m1[icat][iev],m2[icat][iev],j1,j2);
         reMj1j2_temp[j1][j2][icat][iev] = pycuda::real(M_temp);
         imMj1j2_temp[j1][j2][icat][iev] = pycuda::imag(M_temp);
      }
   }

   phasespace_temp[icat][iev] = phasespace(m1[icat][iev],m2[icat][iev]);

 }

__device__ double Factorial(int n) {

   if(n <= 0) return 1.;
   double x = 1;
   int b = 0;
   do {
      b++;
      x *= b;
   } while(b!=n);
   return x;

 }

__device__ void set_buffer_integral_terms(int icat, int iev) {

   double s1_deltat;

   if (acctype == 3) {
      s1_deltat = p0_tres_12+p1_tres_12*(t_err[icat][iev]-deltatmean_tres_12);
      for (int i=0; i<6; ++i) {spl_knot_vector[year_opt[icat]][trig_opt[icat]][i] = knot_gen(wide_window,i);}
   }

   else {
      if (year_opt == 0) {s1_deltat = p0_tres_11+p1_tres_11*(t_err[icat][iev]-deltatmean_tres_11);}
      else {s1_deltat = p0_tres_12+p1_tres_12*(t_err[icat][iev]-deltatmean_tres_12);}
      for (int i=0; i<6; ++i) {spl_knot_vector[year_opt[icat]][trig_opt[icat]][i] = spline_knot(wide_window,i);}
   }

   pycuda::complex<double> z1_hyper_plus_deltat = s1_deltat/sqrt(2.)*pycuda::complex<double>(gamma_Bs_freq-0.5*delta_gamma_freq,0.);
   pycuda::complex<double> z1_hyper_minus_deltat = s1_deltat/sqrt(2.)*pycuda::complex<double>(gamma_Bs_freq+0.5*delta_gamma_freq,0.);
   pycuda::complex<double> z1_trigo_deltat = s1_deltat/sqrt(2.)*pycuda::complex<double>(gamma_Bs_freq,-delta_m_freq);

   double spl_knot_x1_vector_deltat[6];
   double spl_coef_array_deltat[5][4][4][4];

   if (acctype == 1 or acctype == 2) {
      spl_knot_x1_vector_deltat[0] = spl_knot_vector[year_opt[icat]][trig_opt[icat]][0]/(sqrt(2.)*s1_deltat);
      spl_knot_x1_vector_deltat[1] = spl_knot_vector[year_opt[icat]][trig_opt[icat]][1]/(sqrt(2.)*s1_deltat);
      spl_knot_x1_vector_deltat[2] = spl_knot_vector[year_opt[icat]][trig_opt[icat]][2]/(sqrt(2.)*s1_deltat);
      spl_knot_x1_vector_deltat[3] = spl_knot_vector[year_opt[icat]][trig_opt[icat]][3]/(sqrt(2.)*s1_deltat);
      spl_knot_x1_vector_deltat[4] = spl_knot_vector[year_opt[icat]][trig_opt[icat]][4]/(sqrt(2.)*s1_deltat);
      spl_knot_x1_vector_deltat[5] = spl_knot_vector[year_opt[icat]][trig_opt[icat]][5]/(sqrt(2.)*s1_deltat);
      for (int ibin=0; ibin<5; ++ibin) {
         for (int k=0; k<4; ++k) {
            for (int i=0; i<(k+1); ++i) {
               for (int j=0; j<(i+1); ++j) {
                  spl_coef_array_deltat[ibin][k][i][j] = spline_coef(year_opt[icat],trig_opt[icat],wide_window,ibin,k)*Factorial(k)/Factorial(k-i)/Factorial(i-j)/Factorial(j)*pow(s1_deltat/sqrt(2.),i+1)*pow(0.,k-i);
               }
            }
         }
      }
   }
   else if (acctype == 3) {
      spl_knot_x1_vector_deltat[0] = spl_knot_vector[year_opt[icat]][trig_opt[icat]][0]/(sqrt(2.)*s1_deltat);
      spl_knot_x1_vector_deltat[1] = spl_knot_vector[year_opt[icat]][trig_opt[icat]][1]/(sqrt(2.)*s1_deltat);
      spl_knot_x1_vector_deltat[2] = spl_knot_vector[year_opt[icat]][trig_opt[icat]][2]/(sqrt(2.)*s1_deltat);
      spl_knot_x1_vector_deltat[3] = spl_knot_vector[year_opt[icat]][trig_opt[icat]][3]/(sqrt(2.)*s1_deltat);
      spl_knot_x1_vector_deltat[4] = spl_knot_vector[year_opt[icat]][trig_opt[icat]][4]/(sqrt(2.)*s1_deltat);
      spl_knot_x1_vector_deltat[5] = spl_knot_vector[year_opt[icat]][trig_opt[icat]][5]/(sqrt(2.)*s1_deltat);
      for (int ibin=0; ibin<5; ++ibin) {
         for (int k=0; k<4; ++k) {
            for (int i=0; i<(k+1); ++i) {
               for (int j=0; j<(i+1); ++j) {
                  spl_coef_array_deltat[ibin][k][i][j] = coef_gen(wide_window,ibin,k)*Factorial(k)/Factorial(k-i)/Factorial(i-j)/Factorial(j)*pow(s1_deltat/sqrt(2.),i+1)*pow(0.,k-i);
               }
            }
         }
      }
   }

   double integral_conv_exp_hyper_plus_deltat = 0;
   double integral_conv_exp_hyper_minus_deltat = 0;
   pycuda::complex<double> integral_conv_exp_trigo_deltat = pycuda::complex<double>(0.,0.);

   if (acctype == 0) {
      integral_conv_exp_hyper_plus_deltat += pycuda::real(s1_deltat/sqrt(2.)*Mn(0.,12./(sqrt(2.)*s1_deltat),z1_hyper_plus_deltat,0)*Kn(z1_hyper_plus_deltat,0));
      integral_conv_exp_hyper_minus_deltat += pycuda::real(s1_deltat/sqrt(2.)*Mn(0.,12./(sqrt(2.)*s1_deltat),z1_hyper_minus_deltat,0)*Kn(z1_hyper_minus_deltat,0));
      integral_conv_exp_trigo_deltat += s1_deltat/sqrt(2.)*Mn(0.,12./(sqrt(2.)*s1_deltat),z1_trigo_deltat,0)*Kn(z1_trigo_deltat,0);
   }
   else {
      for (int ibin=0; ibin<5; ++ibin) {
         for (int k=0; k<4; ++k) {
            for (int i=0; i<(k+1); ++i) {
               for (int j=0; j<(i+1); ++j) {
                  integral_conv_exp_hyper_plus_deltat += pycuda::real(spl_coef_array_deltat[ibin][k][i][j]*Mn(spl_knot_x1_vector_deltat[ibin],spl_knot_x1_vector_deltat[ibin+1],z1_hyper_plus_deltat,i-j)*Kn(z1_hyper_plus_deltat,j));
                  integral_conv_exp_hyper_minus_deltat += pycuda::real(spl_coef_array_deltat[ibin][k][i][j]*Mn(spl_knot_x1_vector_deltat[ibin],spl_knot_x1_vector_deltat[ibin+1],z1_hyper_minus_deltat,i-j)*Kn(z1_hyper_minus_deltat,j));
                  integral_conv_exp_trigo_deltat += spl_coef_array_deltat[ibin][k][i][j]*Mn(spl_knot_x1_vector_deltat[ibin],spl_knot_x1_vector_deltat[ibin+1],z1_trigo_deltat,i-j)*Kn(z1_trigo_deltat,j);
               }
            }
         }
      }
   }

   IT_cosh_temp_deltat[icat][iev] = 0.5*(integral_conv_exp_hyper_plus_deltat + integral_conv_exp_hyper_minus_deltat);
   IT_sinh_temp_deltat[icat][iev] = 0.5*(integral_conv_exp_hyper_plus_deltat - integral_conv_exp_hyper_minus_deltat);
   IT_cos_temp_deltat[icat][iev] = pycuda::real(integral_conv_exp_trigo_deltat);
   IT_sin_temp_deltat[icat][iev] = pycuda::imag(integral_conv_exp_trigo_deltat);

 }

__global__ void evaluate(double *data, double *out, double *check, double *options, double *re_amps, double *dirCP_asyms, double *im_amps, double *weak_phases, double *mixing_params, double *calib_params, double *mass_integrals, int Nevt) {

   int row = threadIdx.x + blockDim.x * blockIdx.x;
   if (row >= Nevt) { return;}

   int cat_index;
   if (options[1]==0) {
      if (options[2]==0) {cat_index = 0;}
      else {cat_index = 1;}
   }
   else {
      if (options[2]==0) {cat_index = 2;}
      else {cat_index = 3;}
   }

   set_buffer_options(options,cat_index);
   set_buffer_amplitudes(re_amps,dirCP_asyms,im_amps,weak_phases,mixing_params,calib_params);

   int i0 = row*12;
   decision_SSK[cat_index][row] = (int) data[0 + i0];
   decision_OS[cat_index][row] = (int) data[1 + i0];
   etamistag_SSK[cat_index][row] = data[2 + i0];
   etamistag_OS[cat_index][row] = data[3 + i0];
   m1[cat_index][row] = data[4 + i0];
   m2[cat_index][row] = data[5 + i0];
   cos1[cat_index][row] = data[6 + i0];
   cos2[cat_index][row] = data[7 + i0];
   phi[cat_index][row] = data[8 + i0];
   t[cat_index][row] = data[9 + i0];
   t_err[cat_index][row] = data[10 + i0];
   double ev_weight = data[11 + i0];

   double xm1 = (m1[cat_index][row]-1175.)/425.;
   double modulus1 = 1.+c1_mass_swave*xm1+c2_mass_swave*(2.*xm1*xm1-1.)+c3_mass_swave*(4.*xm1*xm1*xm1-3.*xm1)+c4_mass_swave*(8.*xm1*xm1*xm1*xm1-8.*xm1*xm1+1.);
   double xm2 = (m2[cat_index][row]-1175.)/425.;
   double modulus2 = 1.+c1_mass_swave*xm2+c2_mass_swave*(2.*xm2*xm2-1.)+c3_mass_swave*(4.*xm2*xm2*xm2-3.*xm2)+c4_mass_swave*(8.*xm2*xm2*xm2*xm2-8.*xm2*xm2+1.);

   if (modulus1<0 or modulus2<0) {out[row] = -10000000000;}

   else {

      set_buffer_differential_terms(mass_integrals,cat_index,row);
      set_buffer_integral_terms(cat_index,row);

      double num_fit_temp = num_fit(cat_index,row);
      double den_fit_temp = den_fit(cat_index,row);

      if (num_fit_temp/den_fit_temp<=0) {out[row] = -10000000000;}
      else {out[row] = ev_weight*(log(num_fit_temp/den_fit_temp));}

   }

   /*set_buffer_differential_terms(mass_integrals,cat_index,row);
   set_buffer_integral_terms(cat_index,row);

   double num_fit_temp = num_fit(cat_index,row);
   double den_fit_temp = den_fit(cat_index,row);

   if (num_fit_temp/den_fit_temp<=0) {out[row] = -10000000000;}
   else {out[row] = ev_weight*(log(num_fit_temp/den_fit_temp));}*/

 }


// ##########################################
// Event generator

__device__ double Ifi(int i) {

   if (i == 1) {return If1;}
   else if (i == 2) {return If2;}
   else if (i == 3) {return If3;}
   else if (i == 4) {return If4;}
   else if (i == 5) {return If5;}
   else if (i == 6) {return If6;}
   else if (i == 7) {return If7;}
   else if (i == 8) {return If8;}
   else if (i == 9) {return If9;}
   else if (i == 10) {return If10;}
   else if (i == 11) {return If11;}
   else if (i == 12) {return If12;}
   else if (i == 13) {return If13;}
   else if (i == 14) {return If14;}
   else if (i == 15) {return If15;}
   else if (i == 16) {return If16;}
   else if (i == 17) {return If17;}
   else if (i == 18) {return If18;}

   return 0.;

 }

__device__ double Igi(int i) {

   if (i == 1) {return 2.*pi;}
   else if (i == 2) {return 0.;}
   else if (i == 3) {return 0.;}
   else if (i == 4) {return pi;}
   else if (i == 5) {return 0;}
   else if (i == 6) {return pi;}
   else if (i == 7) {return 0.;}
   else if (i == 8) {return 0.;}
   else if (i == 9) {return 0.;}
   else if (i == 10) {return 0.;}
   else if (i == 11) {return 0.;}
   else if (i == 12) {return 0.;}
   else if (i == 13) {return pi;}
   else if (i == 14) {return 0.;}
   else if (i == 15) {return pi;}

   return 0.;

 }

__device__ double Ifjjphhp(int j, int jp, int h, int hp) {

   return Ifi(fjjphhpindexdict[j][jp][h][hp]);

 }

__device__ double Ighhp(int h, int hp) {

   return Igi(ghhpindexdict[h][hp]);

 }

__device__ double Gaus(double x, double mean, double sigma, bool norm = 0) {
   if (sigma == 0) return 1.e30;
   double arg = (x-mean)/sigma;
   double res = exp(-0.5*arg*arg);
   if (!norm) return res;
   return res/(2.50662827463100024*sigma);
 }

__device__ double ran_gamma(hiprandState localState, double a, double b) {

   if (a < 1){
      double u = hiprand_uniform_double(&localState);
      return ran_gamma (localState, 1.0 + a, b) * pow (u, 1.0 / a);
   }

   {
      double x, v, u;
      double d = a - 1.0 / 3.0;
      double c = (1.0 / 3.0) / sqrt (d);

      while (1){
         do{
            x = hiprand_normal_double(&localState);
            v = 1.0 + c * x;
         }
         while (v <= 0);
         v = v * v * v;
         u = hiprand_uniform_double(&localState);
         if (u < 1 - 0.0331 * x * x * x * x)
            break;
         if (log (u) < 0.5 * x * x + d * (1 - v + log (v)))
            break;
      }
      return b * d * v;
   }

 }

__device__ double ran_P_deltat(hiprandState localState) {

   double result;
   while (1) {
      result = ran_gamma(localState,gamma1_dt,beta1_dt);
      if (result>=0. and result<=0.1) {break;}
   }
   return result;

 }

__device__ double P_eta_SSK(double eta) {
   if (eta < 0.5) {return c_SSK*Gaus(eta,mu1_SSK,sigma1_SSK)+(1.-c_SSK)*Gaus(eta,mu2_SSK,sigma2_SSK);}
   else {return 0.;}
 }

__device__ double P_eta_OS(double eta) {
   if (eta < 0.5) {return c_OS*Gaus(eta,mu1_OS,sigma1_OS)+(1.-c_OS)*Gaus(eta,mu2_OS,sigma2_OS);}
   else {return 0.;}
 }

__global__ void get_max_P_eta_SSK(double *out) {

   double funmax = 0;
   double etavar;
   double funvar;
   for (int i=0; i<10000; i++) {
      etavar = i/20000.;
      funvar = P_eta_SSK(etavar);
      if (funvar > funmax) {funmax = funvar;}
   }
   out[0] = 1.1*funmax;
   return;

 }

__global__ void get_max_P_eta_OS(double *out) {

   double funmax = 0;
   double etavar;
   double funvar;
   for (int i=0; i<10000; i++) {
      etavar = i/20000.;
      funvar = P_eta_OS(etavar);
      if (funvar > funmax) {funmax = funvar;}
   }
   out[0] = 1.1*funmax;
   return;

 }

__global__ void set_generator(double *options, double *re_amps, double *dirCP_asyms, double *im_amps, double *weak_phases, double *mixing_params, double *calib_params, double *cond_distr_params, double *mass_integrals, double *ang_integrals) {

   set_buffer_options(options,0);
   year_opt[0] = 0;
   trig_opt[0] = 0;
   set_buffer_amplitudes(re_amps,dirCP_asyms,im_amps,weak_phases,mixing_params,calib_params);

   tag_eff_SSK = cond_distr_params[0];
   mu1_SSK = cond_distr_params[1];
   sigma1_SSK = cond_distr_params[2];
   c_SSK = cond_distr_params[3];
   mu2_SSK = cond_distr_params[4];
   sigma2_SSK = cond_distr_params[5];
   tag_eff_OS = cond_distr_params[6];
   mu1_OS = cond_distr_params[7];
   sigma1_OS = cond_distr_params[8];
   c_OS = cond_distr_params[9];
   mu2_OS = cond_distr_params[10];
   sigma2_OS = cond_distr_params[11];
   gamma1_dt = cond_distr_params[12];
   beta1_dt = cond_distr_params[13];
   c_dt = cond_distr_params[14];
   gamma2_dt = cond_distr_params[15];
   beta2_dt = cond_distr_params[16];

   Im00 = mass_integrals[0];
   Im01 = mass_integrals[1];
   Im10 = mass_integrals[2];
   Im02 = mass_integrals[3];
   Im20 = mass_integrals[4];
   Im11 = mass_integrals[5];
   Im12 = mass_integrals[6];
   Im21 = mass_integrals[7];
   Im22 = mass_integrals[8];

   Ih1Re = mass_integrals[9];
   Ih2Re = mass_integrals[10];
   Ih3Re = mass_integrals[11];
   Ih4Re = mass_integrals[12];
   Ih5Re = mass_integrals[13];
   Ih6Re = mass_integrals[14];
   Ih7Re = mass_integrals[15];
   Ih8Re = mass_integrals[16];
   Ih9Re = mass_integrals[17];
   Ih10Re = mass_integrals[18];
   Ih11Re = mass_integrals[19];
   Ih12Re = mass_integrals[20];
   Ih13Re = mass_integrals[21];
   Ih14Re = mass_integrals[22];
   Ih15Re = mass_integrals[23];
   Ih16Re = mass_integrals[24];
   Ih17Re = mass_integrals[25];
   Ih18Re = mass_integrals[26];
   Ih19Re = mass_integrals[27];
   Ih20Re = mass_integrals[28];
   Ih21Re = mass_integrals[29];
   Ih22Re = mass_integrals[30];
   Ih23Re = mass_integrals[31];
   Ih24Re = mass_integrals[32];
   Ih25Re = mass_integrals[33];
   Ih26Re = mass_integrals[34];
   Ih27Re = mass_integrals[35];
   Ih28Re = mass_integrals[36];
   Ih29Re = mass_integrals[37];
   Ih30Re = mass_integrals[38];

   Ih1Im = mass_integrals[39];
   Ih2Im = mass_integrals[40];
   Ih3Im = mass_integrals[41];
   Ih4Im = mass_integrals[42];
   Ih5Im = mass_integrals[43];
   Ih6Im = mass_integrals[44];
   Ih7Im = mass_integrals[45];
   Ih8Im = mass_integrals[46];
   Ih9Im = mass_integrals[47];
   Ih10Im = mass_integrals[48];
   Ih11Im = mass_integrals[49];
   Ih12Im = mass_integrals[50];
   Ih13Im = mass_integrals[51];
   Ih14Im = mass_integrals[52];
   Ih15Im = mass_integrals[53];
   Ih16Im = mass_integrals[54];
   Ih17Im = mass_integrals[55];
   Ih18Im = mass_integrals[56];
   Ih19Im = mass_integrals[57];
   Ih20Im = mass_integrals[58];
   Ih21Im = mass_integrals[59];

   If1 = ang_integrals[0];
   If2 = ang_integrals[1];
   If3 = ang_integrals[2];
   If4 = ang_integrals[3];
   If5 = ang_integrals[4];
   If6 = ang_integrals[5];
   If7 = ang_integrals[6];
   If8 = ang_integrals[7];
   If9 = ang_integrals[8];
   If10 = ang_integrals[9];
   If11 = ang_integrals[10];
   If12 = ang_integrals[11];
   If13 = ang_integrals[12];
   If14 = ang_integrals[13];
   If15 = ang_integrals[14];
   If16 = ang_integrals[15];
   If17 = ang_integrals[16];
   If18 = ang_integrals[17];

   reIhj1j2j1pj2pdict[0][0][0][0] = pycuda::real(pycuda::complex<double>(Ih22Re,0.));
   reIhj1j2j1pj2pdict[0][0][0][1] = pycuda::real(pycuda::complex<double>(Ih1Re,Ih1Im));
   reIhj1j2j1pj2pdict[0][0][0][2] = pycuda::real(pycuda::complex<double>(Ih2Re,Ih2Im));
   reIhj1j2j1pj2pdict[0][0][1][0] = pycuda::real(pycuda::complex<double>(Ih1Re,Ih1Im));
   reIhj1j2j1pj2pdict[0][0][1][1] = pycuda::real(pycuda::complex<double>(Ih3Re,Ih3Im));
   reIhj1j2j1pj2pdict[0][0][1][2] = pycuda::real(pycuda::complex<double>(Ih4Re,Ih4Im));
   reIhj1j2j1pj2pdict[0][0][2][0] = pycuda::real(pycuda::complex<double>(Ih2Re,Ih2Im));
   reIhj1j2j1pj2pdict[0][0][2][1] = pycuda::real(pycuda::complex<double>(Ih4Re,Ih4Im));
   reIhj1j2j1pj2pdict[0][0][2][2] = pycuda::real(pycuda::complex<double>(Ih5Re,Ih5Im));
   reIhj1j2j1pj2pdict[0][1][0][0] = pycuda::real(pycuda::complex<double>(Ih1Re,-Ih1Im));
   reIhj1j2j1pj2pdict[0][1][0][1] = pycuda::real(pycuda::complex<double>(Ih23Re,0.));
   reIhj1j2j1pj2pdict[0][1][0][2] = pycuda::real(pycuda::complex<double>(Ih6Re,Ih6Im));
   reIhj1j2j1pj2pdict[0][1][1][0] = pycuda::real(pycuda::complex<double>(Ih7Re,Ih7Im));
   reIhj1j2j1pj2pdict[0][1][1][1] = pycuda::real(pycuda::complex<double>(Ih8Re,Ih8Im));
   reIhj1j2j1pj2pdict[0][1][1][2] = pycuda::real(pycuda::complex<double>(Ih9Re,Ih9Im));
   reIhj1j2j1pj2pdict[0][1][2][0] = pycuda::real(pycuda::complex<double>(Ih10Re,Ih10Im));
   reIhj1j2j1pj2pdict[0][1][2][1] = pycuda::real(pycuda::complex<double>(Ih11Re,Ih11Im));
   reIhj1j2j1pj2pdict[0][1][2][2] = pycuda::real(pycuda::complex<double>(Ih12Re,Ih12Im));
   reIhj1j2j1pj2pdict[0][2][0][0] = pycuda::real(pycuda::complex<double>(Ih2Re,-Ih2Im));
   reIhj1j2j1pj2pdict[0][2][0][1] = pycuda::real(pycuda::complex<double>(Ih6Re,-Ih6Im));
   reIhj1j2j1pj2pdict[0][2][0][2] = pycuda::real(pycuda::complex<double>(Ih25Re,0.));
   reIhj1j2j1pj2pdict[0][2][1][0] = pycuda::real(pycuda::complex<double>(Ih10Re,-Ih10Im));
   reIhj1j2j1pj2pdict[0][2][1][1] = pycuda::real(pycuda::complex<double>(Ih13Re,Ih13Im));
   reIhj1j2j1pj2pdict[0][2][1][2] = pycuda::real(pycuda::complex<double>(Ih14Re,Ih14Im));
   reIhj1j2j1pj2pdict[0][2][2][0] = pycuda::real(pycuda::complex<double>(Ih15Re,Ih15Im));
   reIhj1j2j1pj2pdict[0][2][2][1] = pycuda::real(pycuda::complex<double>(Ih16Re,Ih16Im));
   reIhj1j2j1pj2pdict[0][2][2][2] = pycuda::real(pycuda::complex<double>(Ih17Re,Ih17Im));
   reIhj1j2j1pj2pdict[1][0][0][0] = pycuda::real(pycuda::complex<double>(Ih1Re,-Ih1Im));
   reIhj1j2j1pj2pdict[1][0][0][1] = pycuda::real(pycuda::complex<double>(Ih7Re,Ih7Im));
   reIhj1j2j1pj2pdict[1][0][0][2] = pycuda::real(pycuda::complex<double>(Ih10Re,Ih10Im));
   reIhj1j2j1pj2pdict[1][0][1][0] = pycuda::real(pycuda::complex<double>(Ih24Re,0.));
   reIhj1j2j1pj2pdict[1][0][1][1] = pycuda::real(pycuda::complex<double>(Ih8Re,Ih8Im));
   reIhj1j2j1pj2pdict[1][0][1][2] = pycuda::real(pycuda::complex<double>(Ih11Re,Ih11Im));
   reIhj1j2j1pj2pdict[1][0][2][0] = pycuda::real(pycuda::complex<double>(Ih6Re,Ih6Im));
   reIhj1j2j1pj2pdict[1][0][2][1] = pycuda::real(pycuda::complex<double>(Ih9Re,Ih9Im));
   reIhj1j2j1pj2pdict[1][0][2][2] = pycuda::real(pycuda::complex<double>(Ih12Re,Ih12Im));
   reIhj1j2j1pj2pdict[1][1][0][0] = pycuda::real(pycuda::complex<double>(Ih3Re,-Ih3Im));
   reIhj1j2j1pj2pdict[1][1][0][1] = pycuda::real(pycuda::complex<double>(Ih8Re,-Ih8Im));
   reIhj1j2j1pj2pdict[1][1][0][2] = pycuda::real(pycuda::complex<double>(Ih13Re,-Ih13Im));
   reIhj1j2j1pj2pdict[1][1][1][0] = pycuda::real(pycuda::complex<double>(Ih8Re,-Ih8Im));
   reIhj1j2j1pj2pdict[1][1][1][1] = pycuda::real(pycuda::complex<double>(Ih27Re,0.));
   reIhj1j2j1pj2pdict[1][1][1][2] = pycuda::real(pycuda::complex<double>(Ih18Re,Ih18Im));
   reIhj1j2j1pj2pdict[1][1][2][0] = pycuda::real(pycuda::complex<double>(Ih13Re,-Ih13Im));
   reIhj1j2j1pj2pdict[1][1][2][1] = pycuda::real(pycuda::complex<double>(Ih18Re,Ih18Im));
   reIhj1j2j1pj2pdict[1][1][2][2] = pycuda::real(pycuda::complex<double>(Ih19Re,Ih19Im));
   reIhj1j2j1pj2pdict[1][2][0][0] = pycuda::real(pycuda::complex<double>(Ih4Re,-Ih4Im));
   reIhj1j2j1pj2pdict[1][2][0][1] = pycuda::real(pycuda::complex<double>(Ih9Re,-Ih9Im));
   reIhj1j2j1pj2pdict[1][2][0][2] = pycuda::real(pycuda::complex<double>(Ih14Re,-Ih14Im));
   reIhj1j2j1pj2pdict[1][2][1][0] = pycuda::real(pycuda::complex<double>(Ih11Re,-Ih11Im));
   reIhj1j2j1pj2pdict[1][2][1][1] = pycuda::real(pycuda::complex<double>(Ih18Re,-Ih18Im));
   reIhj1j2j1pj2pdict[1][2][1][2] = pycuda::real(pycuda::complex<double>(Ih28Re,0.));
   reIhj1j2j1pj2pdict[1][2][2][0] = pycuda::real(pycuda::complex<double>(Ih16Re,-Ih16Im));
   reIhj1j2j1pj2pdict[1][2][2][1] = pycuda::real(pycuda::complex<double>(Ih20Re,Ih20Im));
   reIhj1j2j1pj2pdict[1][2][2][2] = pycuda::real(pycuda::complex<double>(Ih21Re,Ih21Im));
   reIhj1j2j1pj2pdict[2][0][0][0] = pycuda::real(pycuda::complex<double>(Ih2Re,-Ih2Im));
   reIhj1j2j1pj2pdict[2][0][0][1] = pycuda::real(pycuda::complex<double>(Ih10Re,-Ih10Im));
   reIhj1j2j1pj2pdict[2][0][0][2] = pycuda::real(pycuda::complex<double>(Ih15Re,Ih15Im));
   reIhj1j2j1pj2pdict[2][0][1][0] = pycuda::real(pycuda::complex<double>(Ih6Re,-Ih6Im));
   reIhj1j2j1pj2pdict[2][0][1][1] = pycuda::real(pycuda::complex<double>(Ih13Re,Ih13Im));
   reIhj1j2j1pj2pdict[2][0][1][2] = pycuda::real(pycuda::complex<double>(Ih16Re,Ih16Im));
   reIhj1j2j1pj2pdict[2][0][2][0] = pycuda::real(pycuda::complex<double>(Ih26Re,0.));
   reIhj1j2j1pj2pdict[2][0][2][1] = pycuda::real(pycuda::complex<double>(Ih14Re,Ih14Im));
   reIhj1j2j1pj2pdict[2][0][2][2] = pycuda::real(pycuda::complex<double>(Ih17Re,Ih17Im));
   reIhj1j2j1pj2pdict[2][1][0][0] = pycuda::real(pycuda::complex<double>(Ih4Re,-Ih4Im));
   reIhj1j2j1pj2pdict[2][1][0][1] = pycuda::real(pycuda::complex<double>(Ih11Re,-Ih11Im));
   reIhj1j2j1pj2pdict[2][1][0][2] = pycuda::real(pycuda::complex<double>(Ih16Re,-Ih16Im));
   reIhj1j2j1pj2pdict[2][1][1][0] = pycuda::real(pycuda::complex<double>(Ih9Re,-Ih9Im));
   reIhj1j2j1pj2pdict[2][1][1][1] = pycuda::real(pycuda::complex<double>(Ih18Re,-Ih18Im));
   reIhj1j2j1pj2pdict[2][1][1][2] = pycuda::real(pycuda::complex<double>(Ih20Re,Ih20Im));
   reIhj1j2j1pj2pdict[2][1][2][0] = pycuda::real(pycuda::complex<double>(Ih14Re,-Ih14Im));
   reIhj1j2j1pj2pdict[2][1][2][1] = pycuda::real(pycuda::complex<double>(Ih29Re,0.));
   reIhj1j2j1pj2pdict[2][1][2][2] = pycuda::real(pycuda::complex<double>(Ih21Re,Ih21Im));
   reIhj1j2j1pj2pdict[2][2][0][0] = pycuda::real(pycuda::complex<double>(Ih5Re,-Ih5Im));
   reIhj1j2j1pj2pdict[2][2][0][1] = pycuda::real(pycuda::complex<double>(Ih12Re,-Ih12Im));
   reIhj1j2j1pj2pdict[2][2][0][2] = pycuda::real(pycuda::complex<double>(Ih17Re,-Ih17Im));
   reIhj1j2j1pj2pdict[2][2][1][0] = pycuda::real(pycuda::complex<double>(Ih12Re,-Ih12Im));
   reIhj1j2j1pj2pdict[2][2][1][1] = pycuda::real(pycuda::complex<double>(Ih19Re,-Ih19Im));
   reIhj1j2j1pj2pdict[2][2][1][2] = pycuda::real(pycuda::complex<double>(Ih21Re,-Ih21Im));
   reIhj1j2j1pj2pdict[2][2][2][0] = pycuda::real(pycuda::complex<double>(Ih17Re,-Ih17Im));
   reIhj1j2j1pj2pdict[2][2][2][1] = pycuda::real(pycuda::complex<double>(Ih21Re,-Ih21Im));
   reIhj1j2j1pj2pdict[2][2][2][2] = pycuda::real(pycuda::complex<double>(Ih30Re,0.));
   imIhj1j2j1pj2pdict[0][0][0][0] = pycuda::imag(pycuda::complex<double>(Ih22Re,0.));
   imIhj1j2j1pj2pdict[0][0][0][1] = pycuda::imag(pycuda::complex<double>(Ih1Re,Ih1Im));
   imIhj1j2j1pj2pdict[0][0][0][2] = pycuda::imag(pycuda::complex<double>(Ih2Re,Ih2Im));
   imIhj1j2j1pj2pdict[0][0][1][0] = pycuda::imag(pycuda::complex<double>(Ih1Re,Ih1Im));
   imIhj1j2j1pj2pdict[0][0][1][1] = pycuda::imag(pycuda::complex<double>(Ih3Re,Ih3Im));
   imIhj1j2j1pj2pdict[0][0][1][2] = pycuda::imag(pycuda::complex<double>(Ih4Re,Ih4Im));
   imIhj1j2j1pj2pdict[0][0][2][0] = pycuda::imag(pycuda::complex<double>(Ih2Re,Ih2Im));
   imIhj1j2j1pj2pdict[0][0][2][1] = pycuda::imag(pycuda::complex<double>(Ih4Re,Ih4Im));
   imIhj1j2j1pj2pdict[0][0][2][2] = pycuda::imag(pycuda::complex<double>(Ih5Re,Ih5Im));
   imIhj1j2j1pj2pdict[0][1][0][0] = pycuda::imag(pycuda::complex<double>(Ih1Re,-Ih1Im));
   imIhj1j2j1pj2pdict[0][1][0][1] = pycuda::imag(pycuda::complex<double>(Ih23Re,0.));
   imIhj1j2j1pj2pdict[0][1][0][2] = pycuda::imag(pycuda::complex<double>(Ih6Re,Ih6Im));
   imIhj1j2j1pj2pdict[0][1][1][0] = pycuda::imag(pycuda::complex<double>(Ih7Re,Ih7Im));
   imIhj1j2j1pj2pdict[0][1][1][1] = pycuda::imag(pycuda::complex<double>(Ih8Re,Ih8Im));
   imIhj1j2j1pj2pdict[0][1][1][2] = pycuda::imag(pycuda::complex<double>(Ih9Re,Ih9Im));
   imIhj1j2j1pj2pdict[0][1][2][0] = pycuda::imag(pycuda::complex<double>(Ih10Re,Ih10Im));
   imIhj1j2j1pj2pdict[0][1][2][1] = pycuda::imag(pycuda::complex<double>(Ih11Re,Ih11Im));
   imIhj1j2j1pj2pdict[0][1][2][2] = pycuda::imag(pycuda::complex<double>(Ih12Re,Ih12Im));
   imIhj1j2j1pj2pdict[0][2][0][0] = pycuda::imag(pycuda::complex<double>(Ih2Re,-Ih2Im));
   imIhj1j2j1pj2pdict[0][2][0][1] = pycuda::imag(pycuda::complex<double>(Ih6Re,-Ih6Im));
   imIhj1j2j1pj2pdict[0][2][0][2] = pycuda::imag(pycuda::complex<double>(Ih25Re,0.));
   imIhj1j2j1pj2pdict[0][2][1][0] = pycuda::imag(pycuda::complex<double>(Ih10Re,-Ih10Im));
   imIhj1j2j1pj2pdict[0][2][1][1] = pycuda::imag(pycuda::complex<double>(Ih13Re,Ih13Im));
   imIhj1j2j1pj2pdict[0][2][1][2] = pycuda::imag(pycuda::complex<double>(Ih14Re,Ih14Im));
   imIhj1j2j1pj2pdict[0][2][2][0] = pycuda::imag(pycuda::complex<double>(Ih15Re,Ih15Im));
   imIhj1j2j1pj2pdict[0][2][2][1] = pycuda::imag(pycuda::complex<double>(Ih16Re,Ih16Im));
   imIhj1j2j1pj2pdict[0][2][2][2] = pycuda::imag(pycuda::complex<double>(Ih17Re,Ih17Im));
   imIhj1j2j1pj2pdict[1][0][0][0] = pycuda::imag(pycuda::complex<double>(Ih1Re,-Ih1Im));
   imIhj1j2j1pj2pdict[1][0][0][1] = pycuda::imag(pycuda::complex<double>(Ih7Re,Ih7Im));
   imIhj1j2j1pj2pdict[1][0][0][2] = pycuda::imag(pycuda::complex<double>(Ih10Re,Ih10Im));
   imIhj1j2j1pj2pdict[1][0][1][0] = pycuda::imag(pycuda::complex<double>(Ih24Re,0.));
   imIhj1j2j1pj2pdict[1][0][1][1] = pycuda::imag(pycuda::complex<double>(Ih8Re,Ih8Im));
   imIhj1j2j1pj2pdict[1][0][1][2] = pycuda::imag(pycuda::complex<double>(Ih11Re,Ih11Im));
   imIhj1j2j1pj2pdict[1][0][2][0] = pycuda::imag(pycuda::complex<double>(Ih6Re,Ih6Im));
   imIhj1j2j1pj2pdict[1][0][2][1] = pycuda::imag(pycuda::complex<double>(Ih9Re,Ih9Im));
   imIhj1j2j1pj2pdict[1][0][2][2] = pycuda::imag(pycuda::complex<double>(Ih12Re,Ih12Im));
   imIhj1j2j1pj2pdict[1][1][0][0] = pycuda::imag(pycuda::complex<double>(Ih3Re,-Ih3Im));
   imIhj1j2j1pj2pdict[1][1][0][1] = pycuda::imag(pycuda::complex<double>(Ih8Re,-Ih8Im));
   imIhj1j2j1pj2pdict[1][1][0][2] = pycuda::imag(pycuda::complex<double>(Ih13Re,-Ih13Im));
   imIhj1j2j1pj2pdict[1][1][1][0] = pycuda::imag(pycuda::complex<double>(Ih8Re,-Ih8Im));
   imIhj1j2j1pj2pdict[1][1][1][1] = pycuda::imag(pycuda::complex<double>(Ih27Re,0.));
   imIhj1j2j1pj2pdict[1][1][1][2] = pycuda::imag(pycuda::complex<double>(Ih18Re,Ih18Im));
   imIhj1j2j1pj2pdict[1][1][2][0] = pycuda::imag(pycuda::complex<double>(Ih13Re,-Ih13Im));
   imIhj1j2j1pj2pdict[1][1][2][1] = pycuda::imag(pycuda::complex<double>(Ih18Re,Ih18Im));
   imIhj1j2j1pj2pdict[1][1][2][2] = pycuda::imag(pycuda::complex<double>(Ih19Re,Ih19Im));
   imIhj1j2j1pj2pdict[1][2][0][0] = pycuda::imag(pycuda::complex<double>(Ih4Re,-Ih4Im));
   imIhj1j2j1pj2pdict[1][2][0][1] = pycuda::imag(pycuda::complex<double>(Ih9Re,-Ih9Im));
   imIhj1j2j1pj2pdict[1][2][0][2] = pycuda::imag(pycuda::complex<double>(Ih14Re,-Ih14Im));
   imIhj1j2j1pj2pdict[1][2][1][0] = pycuda::imag(pycuda::complex<double>(Ih11Re,-Ih11Im));
   imIhj1j2j1pj2pdict[1][2][1][1] = pycuda::imag(pycuda::complex<double>(Ih18Re,-Ih18Im));
   imIhj1j2j1pj2pdict[1][2][1][2] = pycuda::imag(pycuda::complex<double>(Ih28Re,0.));
   imIhj1j2j1pj2pdict[1][2][2][0] = pycuda::imag(pycuda::complex<double>(Ih16Re,-Ih16Im));
   imIhj1j2j1pj2pdict[1][2][2][1] = pycuda::imag(pycuda::complex<double>(Ih20Re,Ih20Im));
   imIhj1j2j1pj2pdict[1][2][2][2] = pycuda::imag(pycuda::complex<double>(Ih21Re,Ih21Im));
   imIhj1j2j1pj2pdict[2][0][0][0] = pycuda::imag(pycuda::complex<double>(Ih2Re,-Ih2Im));
   imIhj1j2j1pj2pdict[2][0][0][1] = pycuda::imag(pycuda::complex<double>(Ih10Re,-Ih10Im));
   imIhj1j2j1pj2pdict[2][0][0][2] = pycuda::imag(pycuda::complex<double>(Ih15Re,Ih15Im));
   imIhj1j2j1pj2pdict[2][0][1][0] = pycuda::imag(pycuda::complex<double>(Ih6Re,-Ih6Im));
   imIhj1j2j1pj2pdict[2][0][1][1] = pycuda::imag(pycuda::complex<double>(Ih13Re,Ih13Im));
   imIhj1j2j1pj2pdict[2][0][1][2] = pycuda::imag(pycuda::complex<double>(Ih16Re,Ih16Im));
   imIhj1j2j1pj2pdict[2][0][2][0] = pycuda::imag(pycuda::complex<double>(Ih26Re,0.));
   imIhj1j2j1pj2pdict[2][0][2][1] = pycuda::imag(pycuda::complex<double>(Ih14Re,Ih14Im));
   imIhj1j2j1pj2pdict[2][0][2][2] = pycuda::imag(pycuda::complex<double>(Ih17Re,Ih17Im));
   imIhj1j2j1pj2pdict[2][1][0][0] = pycuda::imag(pycuda::complex<double>(Ih4Re,-Ih4Im));
   imIhj1j2j1pj2pdict[2][1][0][1] = pycuda::imag(pycuda::complex<double>(Ih11Re,-Ih11Im));
   imIhj1j2j1pj2pdict[2][1][0][2] = pycuda::imag(pycuda::complex<double>(Ih16Re,-Ih16Im));
   imIhj1j2j1pj2pdict[2][1][1][0] = pycuda::imag(pycuda::complex<double>(Ih9Re,-Ih9Im));
   imIhj1j2j1pj2pdict[2][1][1][1] = pycuda::imag(pycuda::complex<double>(Ih18Re,-Ih18Im));
   imIhj1j2j1pj2pdict[2][1][1][2] = pycuda::imag(pycuda::complex<double>(Ih20Re,Ih20Im));
   imIhj1j2j1pj2pdict[2][1][2][0] = pycuda::imag(pycuda::complex<double>(Ih14Re,-Ih14Im));
   imIhj1j2j1pj2pdict[2][1][2][1] = pycuda::imag(pycuda::complex<double>(Ih29Re,0.));
   imIhj1j2j1pj2pdict[2][1][2][2] = pycuda::imag(pycuda::complex<double>(Ih21Re,Ih21Im));
   imIhj1j2j1pj2pdict[2][2][0][0] = pycuda::imag(pycuda::complex<double>(Ih5Re,-Ih5Im));
   imIhj1j2j1pj2pdict[2][2][0][1] = pycuda::imag(pycuda::complex<double>(Ih12Re,-Ih12Im));
   imIhj1j2j1pj2pdict[2][2][0][2] = pycuda::imag(pycuda::complex<double>(Ih17Re,-Ih17Im));
   imIhj1j2j1pj2pdict[2][2][1][0] = pycuda::imag(pycuda::complex<double>(Ih12Re,-Ih12Im));
   imIhj1j2j1pj2pdict[2][2][1][1] = pycuda::imag(pycuda::complex<double>(Ih19Re,-Ih19Im));
   imIhj1j2j1pj2pdict[2][2][1][2] = pycuda::imag(pycuda::complex<double>(Ih21Re,-Ih21Im));
   imIhj1j2j1pj2pdict[2][2][2][0] = pycuda::imag(pycuda::complex<double>(Ih17Re,-Ih17Im));
   imIhj1j2j1pj2pdict[2][2][2][1] = pycuda::imag(pycuda::complex<double>(Ih21Re,-Ih21Im));
   imIhj1j2j1pj2pdict[2][2][2][2] = pycuda::imag(pycuda::complex<double>(Ih30Re,0.));

 }

__device__ void set_buffer_differential_terms_gen(int iev) {

   double f1,f2,s1,s2,x1,x2;

   if (acctype == 3) {
      f1 = 1.;
      f2 = 0.;
      s1 = p0_tres_12+p1_tres_12*(t_err[0][iev]-deltatmean_tres_12);
      s2 = 1.;
      x1 = t[0][iev]/(sqrt(2.)*s1);
      x2 = t[0][iev]/(sqrt(2.)*s2);
   }
   else {
      f1 = 1.;
      f2 = 0.;
      if (year_opt == 0) {s1 = p0_tres_11+p1_tres_11*(t_err[0][iev]-deltatmean_tres_11);}
      else {s1 = p0_tres_12+p1_tres_12*(t_err[0][iev]-deltatmean_tres_12);}
      s2 = 1.;
      x1 = t[0][iev]/(sqrt(2.)*s1);
      x2 = t[0][iev]/(sqrt(2.)*s2);
   }

   pycuda::complex<double> z1_hyper_plus = s1/sqrt(2.)*pycuda::complex<double>(gamma_Bs_freq-0.5*delta_gamma_freq,0.);
   pycuda::complex<double> z2_hyper_plus = s2/sqrt(2.)*pycuda::complex<double>(gamma_Bs_freq-0.5*delta_gamma_freq,0.);
   pycuda::complex<double> z1_hyper_minus = s1/sqrt(2.)*pycuda::complex<double>(gamma_Bs_freq+0.5*delta_gamma_freq,0.);
   pycuda::complex<double> z2_hyper_minus = s2/sqrt(2.)*pycuda::complex<double>(gamma_Bs_freq+0.5*delta_gamma_freq,0.);
   pycuda::complex<double> z1_trigo = s1/sqrt(2.)*pycuda::complex<double>(gamma_Bs_freq,-delta_m_freq);
   pycuda::complex<double> z2_trigo = s2/sqrt(2.)*pycuda::complex<double>(gamma_Bs_freq,-delta_m_freq);

   double conv_exp_hyper_plus = pycuda::real(f1*conv_exp(x1,z1_hyper_plus)+f2*conv_exp(x2,z2_hyper_plus));
   double conv_exp_hyper_minus = pycuda::real(f1*conv_exp(x1,z1_hyper_minus)+f2*conv_exp(x2,z2_hyper_minus));
   pycuda::complex<double> conv_exp_trigo = f1*conv_exp(x1,z1_trigo)+f2*conv_exp(x2,z2_trigo);

   T_cosh_temp[0][iev] = 0.5*(conv_exp_hyper_plus + conv_exp_hyper_minus);
   T_sinh_temp[0][iev] = 0.5*(conv_exp_hyper_plus - conv_exp_hyper_minus);
   T_cos_temp[0][iev] = pycuda::real(conv_exp_trigo);
   T_sin_temp[0][iev] = pycuda::imag(conv_exp_trigo);

   zeta_temp[0][iev] = zeta(decision_SSK[0][iev],decision_OS[0][iev],etamistag_SSK[0][iev],etamistag_OS[0][iev]);
   DCP_tzero_temp[0][iev] = DCP_tzero(decision_SSK[0][iev],decision_OS[0][iev],etamistag_SSK[0][iev],etamistag_OS[0][iev]);

   for (int i=0; i<18; ++i) {fi_cos1_temp[i][0][iev] = fi(cos1[0][iev],i+1);}
   for (int i=0; i<18; ++i) {fi_cos2_temp[i][0][iev] = fi(cos2[0][iev],i+1);}
   for (int i=0; i<15; ++i) {gi_temp[i][0][iev] = gi(phi[0][iev],i+1);}

   for (int j1=0; j1<3; ++j1) {
      for (int j2=0; j2<3; ++j2) {
         pycuda::complex<double> M_temp = Mj1j2(m1[0][iev],m2[0][iev],j1,j2);
         reMj1j2_temp[j1][j2][0][iev] = pycuda::real(M_temp);
         imMj1j2_temp[j1][j2][0][iev] = pycuda::imag(M_temp);
      }
   }

   phasespace_temp[0][iev] = phasespace(m1[0][iev],m2[0][iev]);

 }

__device__ double comp_den_toy(int j1, int j2, int h, int j1p, int j2p, int hp, int iev) { 
   return pycuda::real(ITj1j2hj1pj2php_deltat(j1,j2,h,j1p,j2p,hp,0,iev)*Nj1j2hj1pj2php(j1,j2,h,j1p,j2p,hp)*Ihj1j2j1pj2p(j1,j2,j1p,j2p))*Ighhp(h,hp)*Ifjjphhp(j1,j1p,h,hp)*Ifjjphhp(j2,j2p,h,hp);
 }

__device__ double den_toy(int iev) {

   return comp_den_toy(0,0,0,0,0,0,iev)+comp_den_toy(0,1,0,0,1,0,iev)+comp_den_toy(0,2,0,0,2,0,iev)+comp_den_toy(1,0,0,1,0,0,iev)+comp_den_toy(1,1,0,1,1,0,iev)
+comp_den_toy(1,1,1,1,1,1,iev)+comp_den_toy(1,1,2,1,1,2,iev)+comp_den_toy(1,2,0,1,2,0,iev)+comp_den_toy(1,2,1,1,2,1,iev)+comp_den_toy(1,2,2,1,2,2,iev)
+comp_den_toy(2,0,0,2,0,0,iev)+comp_den_toy(2,1,0,2,1,0,iev)+comp_den_toy(2,1,1,2,1,1,iev)+comp_den_toy(2,1,2,2,1,2,iev)+comp_den_toy(2,2,0,2,2,0,iev)
+comp_den_toy(2,2,1,2,2,1,iev)+comp_den_toy(2,2,2,2,2,2,iev)+comp_den_toy(2,2,3,2,2,3,iev)+comp_den_toy(2,2,4,2,2,4,iev)+2.*comp_den_toy(0,1,0,0,0,0,iev)
+2.*comp_den_toy(0,1,0,1,0,0,iev)+2.*comp_den_toy(0,1,0,2,0,0,iev)+2.*comp_den_toy(0,2,0,0,0,0,iev)+2.*comp_den_toy(0,2,0,0,1,0,iev)+2.*comp_den_toy(0,2,0,1,0,0,iev)
+2.*comp_den_toy(0,2,0,1,1,0,iev)+2.*comp_den_toy(0,2,0,2,0,0,iev)+2.*comp_den_toy(0,2,0,2,1,0,iev)+2.*comp_den_toy(1,0,0,0,0,0,iev)+2.*comp_den_toy(1,1,0,0,0,0,iev)
+2.*comp_den_toy(1,1,0,0,1,0,iev)+2.*comp_den_toy(1,1,0,1,0,0,iev)+2.*comp_den_toy(1,1,0,2,0,0,iev)+2.*comp_den_toy(1,2,0,0,0,0,iev)+2.*comp_den_toy(1,2,0,0,1,0,iev)
+2.*comp_den_toy(1,2,0,0,2,0,iev)+2.*comp_den_toy(1,2,0,1,0,0,iev)+2.*comp_den_toy(1,2,0,1,1,0,iev)+2.*comp_den_toy(1,2,0,2,0,0,iev)+2.*comp_den_toy(1,2,0,2,1,0,iev)
+2.*comp_den_toy(1,2,1,1,1,1,iev)+2.*comp_den_toy(1,2,1,2,1,1,iev)+2.*comp_den_toy(1,2,2,1,1,2,iev)+2.*comp_den_toy(1,2,2,2,1,2,iev)+2.*comp_den_toy(2,0,0,0,0,0,iev)
+2.*comp_den_toy(2,0,0,1,0,0,iev)+2.*comp_den_toy(2,1,0,0,0,0,iev)+2.*comp_den_toy(2,1,0,0,1,0,iev)+2.*comp_den_toy(2,1,0,1,0,0,iev)+2.*comp_den_toy(2,1,0,1,1,0,iev)
+2.*comp_den_toy(2,1,0,2,0,0,iev)+2.*comp_den_toy(2,1,1,1,1,1,iev)+2.*comp_den_toy(2,1,2,1,1,2,iev)+2.*comp_den_toy(2,2,0,0,0,0,iev)+2.*comp_den_toy(2,2,0,0,1,0,iev)
+2.*comp_den_toy(2,2,0,0,2,0,iev)+2.*comp_den_toy(2,2,0,1,0,0,iev)+2.*comp_den_toy(2,2,0,1,1,0,iev)+2.*comp_den_toy(2,2,0,1,2,0,iev)+2.*comp_den_toy(2,2,0,2,0,0,iev)
+2.*comp_den_toy(2,2,0,2,1,0,iev)+2.*comp_den_toy(2,2,1,1,1,1,iev)+2.*comp_den_toy(2,2,1,1,2,1,iev)+2.*comp_den_toy(2,2,1,2,1,1,iev)+2.*comp_den_toy(2,2,2,1,1,2,iev)
+2.*comp_den_toy(2,2,2,1,2,2,iev)+2.*comp_den_toy(2,2,2,2,1,2,iev);

 }

__global__ void evaluate_CondPDF(double m1_ran, double m2_ran, double cos1_ran, double cos2_ran, double phi_ran, double t_ran, double t_err_ran, int q_SSK_ran, int q_OS_ran, double eta_SSK_ran, double eta_OS_ran, double *mixing_params, double *calib_params, double *out) {

   m1[0][0] = m1_ran;
   m2[0][0] = m2_ran;
   cos1[0][0] = cos1_ran;
   cos2[0][0] = cos2_ran;
   phi[0][0] = phi_ran;
   t[0][0] = t_ran;
   t_err[0][0] = t_err_ran;
   decision_SSK[0][0] = q_SSK_ran;
   decision_OS[0][0] = q_OS_ran;
   etamistag_SSK[0][0] = eta_SSK_ran;
   etamistag_OS[0][0] = eta_OS_ran;

   gamma_Bs_freq = mixing_params[1];
   delta_gamma_freq = mixing_params[2];
   p0metac_tag_SSK = calib_params[0];
   p0metac_tag_OS = calib_params[1];
   Dp0half_tag_SSK = calib_params[2];
   Dp0half_tag_OS = calib_params[3];
   p1_tag_SSK = calib_params[4];
   p1_tag_OS = calib_params[5];
   Dp1half_tag_SSK = calib_params[6];
   Dp1half_tag_OS = calib_params[7];
   p0_tres_12 = calib_params[14];
   p1_tres_12 = calib_params[15];

   set_buffer_differential_terms_gen(0);
   set_buffer_integral_terms(0,0);

   double num_temp = num_fit(0,0)*accGen(t_ran,m1_ran,m2_ran,cos1_ran,cos2_ran,phi_ran);
   double den_temp = den_toy(0);

   if (num_temp/den_temp<=0) {out[0] = -1.e20;}
   else {out[0] = log(num_temp/den_temp);}

 }

__global__ void generateEvent(double *gendata, double max_fun_eta_SSK, double max_fun_eta_OS, double max_fun_cond, int Nevt) {

   int row = threadIdx.x + blockDim.x * blockIdx.x;
   if (row >= Nevt) { return;}

   hiprandState state;
   hiprand_init((unsigned long long)clock(), row, 0, &state);

   // Determination of the per event decay time error.
   t_err[0][row] = ran_P_deltat(state);

   // Determination of the SSK mistag probability.
   double dec_SSK_tagged = hiprand_uniform(&state);
   if (dec_SSK_tagged < tag_eff_SSK) {
      double etamistag_SSK_ran;
      max_fun[row] = max_fun_eta_SSK;
      while (1) {
         etamistag_SSK_ran = 0.5*hiprand_uniform(&state);
         fun_ran[row] = P_eta_SSK(etamistag_SSK_ran);
         dec_accepted[row] = max_fun[row]*hiprand_uniform(&state);
         if (fun_ran[row] > dec_accepted[row]) {break;}
         }
      etamistag_SSK[0][row] = etamistag_SSK_ran;
      }
   else {
      etamistag_SSK[0][row] = 0.5;
      }

   // Determination of the OS mistag probability.
   double dec_OS_tagged = hiprand_uniform(&state);
   if (dec_OS_tagged < tag_eff_OS) {
      double etamistag_OS_ran;
      max_fun[row] = max_fun_eta_OS;
      while (1) {
         etamistag_OS_ran = 0.5*hiprand_uniform(&state);
         fun_ran[row] = P_eta_OS(etamistag_OS_ran);
         dec_accepted[row] = max_fun[row]*hiprand_uniform(&state);
         if (fun_ran[row] > dec_accepted[row]) {break;}
         }
      etamistag_OS[0][row] = etamistag_OS_ran;
      }
   else {
      etamistag_OS[0][row] = 0.5;
      }

   // Determination of the decay observables.
   max_fun[row] = max_fun_cond;
   while (1) {
      if (wide_window == 1) {
         m1[0][row] = 750.+hiprand_uniform(&state)*(1600.-750.);
         m2[0][row] = 750.+hiprand_uniform(&state)*(1600.-750.);
      }
      else {
         m1[0][row] = 750.+hiprand_uniform(&state)*(1050.-750.);
         m2[0][row] = 750.+hiprand_uniform(&state)*(1050.-750.);
      }
      cos1[0][row] = -1.+hiprand_uniform(&state)*2.;
      cos2[0][row] = -1.+hiprand_uniform(&state)*2.;
      phi[0][row] = hiprand_uniform(&state)*2.*pi;
      t[0][row] = hiprand_uniform(&state)*12.;
      if (etamistag_SSK[0][row] == 0.5) {decision_SSK[0][row] = 0;}
      else {
         double d_SSK = hiprand_uniform(&state);
         if (d_SSK <= 0.5) {decision_SSK[0][row] = -1;}
         else {decision_SSK[0][row] = 1;}
      }
      if (etamistag_OS[0][row] == 0.5) {decision_OS[0][row] = 0;}
      else {
         double d_OS = hiprand_uniform(&state);
         if (d_OS <= 0.5) {decision_OS[0][row] = -1;}
         else {decision_OS[0][row] = 1;}
      }
      set_buffer_differential_terms_gen(row);
      set_buffer_integral_terms(0,row);
      dec_accepted[row] = hiprand_uniform(&state);
      fun_ran[row] = num_fit(0,row)/den_toy(row)*accGen(t[0][row],m1[0][row],m2[0][row],cos1[0][row],cos2[0][row],phi[0][row])/max_fun[row];
      if (fun_ran[row] > dec_accepted[row]) {break;}

   }

   int i0 = row*12;
   gendata[0 + i0] = (double) decision_SSK[0][row];
   gendata[1 + i0] = (double) decision_OS[0][row];
   gendata[2 + i0] = etamistag_SSK[0][row];
   gendata[3 + i0] = etamistag_OS[0][row];
   gendata[4 + i0] = m1[0][row];
   gendata[5 + i0] = m2[0][row];
   gendata[6 + i0] = cos1[0][row];
   gendata[7 + i0] = cos2[0][row];
   gendata[8 + i0] = phi[0][row];
   gendata[9 + i0] = t[0][row];
   gendata[10 + i0] = t_err[0][row];
   gendata[11 + i0] = 1.;

   return;

 }

__global__ void evaluate_toy(double *data, double *out, double *re_amps, double *dirCP_asyms, double *im_amps, double *weak_phases, double *mixing_params, double *calib_params, int Nevt) {

   int row = threadIdx.x + blockDim.x * blockIdx.x;
   if (row >= Nevt) { return;}

   set_buffer_amplitudes(re_amps,dirCP_asyms,im_amps,weak_phases,mixing_params,calib_params);

   int i0 = row*12;
   decision_SSK[0][row] = (int) data[0 + i0];
   decision_OS[0][row] = (int) data[1 + i0];
   etamistag_SSK[0][row] = data[2 + i0];
   etamistag_OS[0][row] = data[3 + i0];
   m1[0][row] = data[4 + i0];
   m2[0][row] = data[5 + i0];
   cos1[0][row] = data[6 + i0];
   cos2[0][row] = data[7 + i0];
   phi[0][row] = data[8 + i0];
   t[0][row] = data[9 + i0];
   t_err[0][row] = data[10 + i0];

   set_buffer_differential_terms_gen(row);
   set_buffer_integral_terms(0,row);

   double num_fit_temp = num_fit(0,row);
   double den_fit_temp = den_toy(row);

   if (num_fit_temp/den_fit_temp<=0) {out[row] = -10000000000;}
   else {out[row] = log(num_fit_temp/den_fit_temp);}

 }

__global__ void set_mass_params(double *calib_params) {

   mv = calib_params[16];
   ms = calib_params[17];
   mt = calib_params[18];
   gv = calib_params[19];
   gs = calib_params[20];
   gt = calib_params[21];
   c1_mass_swave = calib_params[22];
   c2_mass_swave = calib_params[23];
   c3_mass_swave = calib_params[24];
   c4_mass_swave = calib_params[25];
   c5_mass_swave = calib_params[26];
   c6_mass_swave = calib_params[27];
   c7_mass_swave = calib_params[28];
   c8_mass_swave = calib_params[29];
   c9_mass_swave = calib_params[30];

 }

__global__ void find_max_mass_pdf(int mpdfid, int mintnpoints, double minthlimit, double *mpdfarray) {

   int mintindex = threadIdx.x + blockDim.x * blockIdx.x;
   if (mintindex >= mintnpoints*mintnpoints) { return;}

   int im1 = mintindex / mintnpoints;
   int im2 = mintindex % mintnpoints;

   double mintstep = (minthlimit-750.)/mintnpoints;
   double m1_ = 750.+im1*mintstep;
   double m2_ = 750.+im2*mintstep;

   if (mpdfid == 0) {mpdfarray[mintindex] =  pycuda::norm(Mj1j2_unnorm(m1_,m2_,0,0))*phasespace(m1_,m2_);}
   else if (mpdfid == 1) {mpdfarray[mintindex] =  pycuda::norm(Mj1j2_unnorm(m1_,m2_,0,1))*phasespace(m1_,m2_);}
   else if (mpdfid == 2) {mpdfarray[mintindex] =  pycuda::norm(Mj1j2_unnorm(m1_,m2_,1,0))*phasespace(m1_,m2_);}
   else if (mpdfid == 3) {mpdfarray[mintindex] =  pycuda::norm(Mj1j2_unnorm(m1_,m2_,0,2))*phasespace(m1_,m2_);}
   else if (mpdfid == 4) {mpdfarray[mintindex] =  pycuda::norm(Mj1j2_unnorm(m1_,m2_,2,0))*phasespace(m1_,m2_);}
   else if (mpdfid == 5) {mpdfarray[mintindex] =  pycuda::norm(Mj1j2_unnorm(m1_,m2_,1,1))*phasespace(m1_,m2_);}
   else if (mpdfid == 6) {mpdfarray[mintindex] =  pycuda::norm(Mj1j2_unnorm(m1_,m2_,1,2))*phasespace(m1_,m2_);}
   else if (mpdfid == 7) {mpdfarray[mintindex] =  pycuda::norm(Mj1j2_unnorm(m1_,m2_,2,1))*phasespace(m1_,m2_);}
   else if (mpdfid == 8) {mpdfarray[mintindex] =  pycuda::norm(Mj1j2_unnorm(m1_,m2_,2,2))*phasespace(m1_,m2_);}

 }

__global__ void compute_mass_integral(int mpdfid, int npoints, double minthlimit, double maxmpdf, int *mintarray) {

   int row = threadIdx.x + blockDim.x * blockIdx.x;
   if (row >= npoints) { return;}

   hiprandState state;
   hiprand_init((unsigned long long)clock(), row, 0, &state);

   double m1_ = 750.+hiprand_uniform(&state)*(minthlimit-750.);
   double m2_ = 750.+hiprand_uniform(&state)*(minthlimit-750.);
   double vertical_ = hiprand_uniform(&state)*maxmpdf;

   double mpdf_temp = 0;
   if (mpdfid == 0) {mpdf_temp =  pycuda::norm(Mj1j2_unnorm(m1_,m2_,0,0))*phasespace(m1_,m2_);}
   else if (mpdfid == 1) {mpdf_temp =  pycuda::norm(Mj1j2_unnorm(m1_,m2_,0,1))*phasespace(m1_,m2_);}
   else if (mpdfid == 2) {mpdf_temp =  pycuda::norm(Mj1j2_unnorm(m1_,m2_,1,0))*phasespace(m1_,m2_);}
   else if (mpdfid == 3) {mpdf_temp =  pycuda::norm(Mj1j2_unnorm(m1_,m2_,0,2))*phasespace(m1_,m2_);}
   else if (mpdfid == 4) {mpdf_temp =  pycuda::norm(Mj1j2_unnorm(m1_,m2_,2,0))*phasespace(m1_,m2_);}
   else if (mpdfid == 5) {mpdf_temp =  pycuda::norm(Mj1j2_unnorm(m1_,m2_,1,1))*phasespace(m1_,m2_);}
   else if (mpdfid == 6) {mpdf_temp =  pycuda::norm(Mj1j2_unnorm(m1_,m2_,1,2))*phasespace(m1_,m2_);}
   else if (mpdfid == 7) {mpdf_temp =  pycuda::norm(Mj1j2_unnorm(m1_,m2_,2,1))*phasespace(m1_,m2_);}
   else if (mpdfid == 8) {mpdf_temp =  pycuda::norm(Mj1j2_unnorm(m1_,m2_,2,2))*phasespace(m1_,m2_);}

   if (vertical_ <= mpdf_temp) {mintarray[row] = 1;}
   else {mintarray[row] = 0;}

 }

__global__ void compute_mint_array(int mpdfid, int mintnpoints, double minthlimit, double *mpdfarray) {

   int mintindex = threadIdx.x + blockDim.x * blockIdx.x;
   if (mintindex >= mintnpoints*mintnpoints) { return;}

   int im1 = mintindex / mintnpoints;
   int im2 = mintindex % mintnpoints;

   double mintstep = (minthlimit-750.)/mintnpoints;
   double m1_ = 750.+im1*mintstep;
   double m2_ = 750.+im2*mintstep;
   double m1next_ = 750.+(im1+1)*mintstep;
   double m2next_ = 750.+(im2+1)*mintstep;

   double point1 = 0.;
   double point2 = 0.;
   double point3 = 0.;
   double point4 = 0.;

   if (mpdfid == 0) {point1 =  pycuda::norm(Mj1j2_unnorm(m1_,m2_,0,0))*phasespace(m1_,m2_);}
   else if (mpdfid == 1) {point1 =  pycuda::norm(Mj1j2_unnorm(m1_,m2_,0,1))*phasespace(m1_,m2_);}
   else if (mpdfid == 2) {point1 =  pycuda::norm(Mj1j2_unnorm(m1_,m2_,1,0))*phasespace(m1_,m2_);}
   else if (mpdfid == 3) {point1 =  pycuda::norm(Mj1j2_unnorm(m1_,m2_,0,2))*phasespace(m1_,m2_);}
   else if (mpdfid == 4) {point1 =  pycuda::norm(Mj1j2_unnorm(m1_,m2_,2,0))*phasespace(m1_,m2_);}
   else if (mpdfid == 5) {point1 =  pycuda::norm(Mj1j2_unnorm(m1_,m2_,1,1))*phasespace(m1_,m2_);}
   else if (mpdfid == 6) {point1 =  pycuda::norm(Mj1j2_unnorm(m1_,m2_,1,2))*phasespace(m1_,m2_);}
   else if (mpdfid == 7) {point1 =  pycuda::norm(Mj1j2_unnorm(m1_,m2_,2,1))*phasespace(m1_,m2_);}
   else if (mpdfid == 8) {point1 =  pycuda::norm(Mj1j2_unnorm(m1_,m2_,2,2))*phasespace(m1_,m2_);}

   if (mpdfid == 0) {point2 =  pycuda::norm(Mj1j2_unnorm(m1next_,m2_,0,0))*phasespace(m1next_,m2_);}
   else if (mpdfid == 1) {point2 =  pycuda::norm(Mj1j2_unnorm(m1next_,m2_,0,1))*phasespace(m1next_,m2_);}
   else if (mpdfid == 2) {point2 =  pycuda::norm(Mj1j2_unnorm(m1next_,m2_,1,0))*phasespace(m1next_,m2_);}
   else if (mpdfid == 3) {point2 =  pycuda::norm(Mj1j2_unnorm(m1next_,m2_,0,2))*phasespace(m1next_,m2_);}
   else if (mpdfid == 4) {point2 =  pycuda::norm(Mj1j2_unnorm(m1next_,m2_,2,0))*phasespace(m1next_,m2_);}
   else if (mpdfid == 5) {point2 =  pycuda::norm(Mj1j2_unnorm(m1next_,m2_,1,1))*phasespace(m1next_,m2_);}
   else if (mpdfid == 6) {point2 =  pycuda::norm(Mj1j2_unnorm(m1next_,m2_,1,2))*phasespace(m1next_,m2_);}
   else if (mpdfid == 7) {point2 =  pycuda::norm(Mj1j2_unnorm(m1next_,m2_,2,1))*phasespace(m1next_,m2_);}
   else if (mpdfid == 8) {point2 =  pycuda::norm(Mj1j2_unnorm(m1next_,m2_,2,2))*phasespace(m1next_,m2_);}

   if (mpdfid == 0) {point3 =  pycuda::norm(Mj1j2_unnorm(m1_,m2next_,0,0))*phasespace(m1_,m2next_);}
   else if (mpdfid == 1) {point3 =  pycuda::norm(Mj1j2_unnorm(m1_,m2next_,0,1))*phasespace(m1_,m2next_);}
   else if (mpdfid == 2) {point3 =  pycuda::norm(Mj1j2_unnorm(m1_,m2next_,1,0))*phasespace(m1_,m2next_);}
   else if (mpdfid == 3) {point3 =  pycuda::norm(Mj1j2_unnorm(m1_,m2next_,0,2))*phasespace(m1_,m2next_);}
   else if (mpdfid == 4) {point3 =  pycuda::norm(Mj1j2_unnorm(m1_,m2next_,2,0))*phasespace(m1_,m2next_);}
   else if (mpdfid == 5) {point3 =  pycuda::norm(Mj1j2_unnorm(m1_,m2next_,1,1))*phasespace(m1_,m2next_);}
   else if (mpdfid == 6) {point3 =  pycuda::norm(Mj1j2_unnorm(m1_,m2next_,1,2))*phasespace(m1_,m2next_);}
   else if (mpdfid == 7) {point3 =  pycuda::norm(Mj1j2_unnorm(m1_,m2next_,2,1))*phasespace(m1_,m2next_);}
   else if (mpdfid == 8) {point3 =  pycuda::norm(Mj1j2_unnorm(m1_,m2next_,2,2))*phasespace(m1_,m2next_);}

   if (mpdfid == 0) {point4 =  pycuda::norm(Mj1j2_unnorm(m1next_,m2next_,0,0))*phasespace(m1next_,m2next_);}
   else if (mpdfid == 1) {point4 =  pycuda::norm(Mj1j2_unnorm(m1next_,m2next_,0,1))*phasespace(m1next_,m2next_);}
   else if (mpdfid == 2) {point4 =  pycuda::norm(Mj1j2_unnorm(m1next_,m2next_,1,0))*phasespace(m1next_,m2next_);}
   else if (mpdfid == 3) {point4 =  pycuda::norm(Mj1j2_unnorm(m1next_,m2next_,0,2))*phasespace(m1next_,m2next_);}
   else if (mpdfid == 4) {point4 =  pycuda::norm(Mj1j2_unnorm(m1next_,m2next_,2,0))*phasespace(m1next_,m2next_);}
   else if (mpdfid == 5) {point4 =  pycuda::norm(Mj1j2_unnorm(m1next_,m2next_,1,1))*phasespace(m1next_,m2next_);}
   else if (mpdfid == 6) {point4 =  pycuda::norm(Mj1j2_unnorm(m1next_,m2next_,1,2))*phasespace(m1next_,m2next_);}
   else if (mpdfid == 7) {point4 =  pycuda::norm(Mj1j2_unnorm(m1next_,m2next_,2,1))*phasespace(m1next_,m2next_);}
   else if (mpdfid == 8) {point4 =  pycuda::norm(Mj1j2_unnorm(m1next_,m2next_,2,2))*phasespace(m1next_,m2next_);}

   mpdfarray[mintindex] = 0.25*(point1+point2+point3+point4)*mintstep*mintstep;

 }

__global__ void set_mass_integrals(double *mass_integrals) {

   Im00 = mass_integrals[0];
   Im01 = mass_integrals[1];
   Im10 = mass_integrals[2];
   Im02 = mass_integrals[3];
   Im20 = mass_integrals[4];
   Im11 = mass_integrals[5];
   Im12 = mass_integrals[6];
   Im21 = mass_integrals[7];
   Im22 = mass_integrals[8];

 }

__global__ void compute_nw(double *MCdata, int j1, int j2, int h, int j1p, int j2p, int hp, int part, int NMCevts, double *evout) {

   int row = threadIdx.x + blockDim.x * blockIdx.x;
   if (row >= NMCevts) { return;}

   int i0 = row*6;
   double m1_ = MCdata[0 + i0];
   double m2_ = MCdata[1 + i0];
   double cos1_ = MCdata[2 + i0];
   double cos2_ = MCdata[3 + i0];
   double phi_ = MCdata[4 + i0];
   double weight_ = MCdata[5 + i0];

   pycuda::complex<double> ev_nw_temp = Nj1j2hj1pj2php(j1,j2,h,j1p,j2p,hp)*Mj1j2(m1_,m2_,j1,j2)*pycuda::conj(Mj1j2(m1_,m2_,j1p,j2p))*phasespace(m1_,m2_)*fi(cos1_,(int) fjjphhpindexdict[j1][j1p][h][hp])*fi(cos2_,(int) fjjphhpindexdict[j2][j2p][h][hp])*gi(phi_,(int) ghhpindexdict[h][hp]);

   if (part == 0) {evout[row] = 100.*weight_*pycuda::real(ev_nw_temp);}
   else {evout[row] = 100.*weight_*pycuda::imag(ev_nw_temp);}

 }

__global__ void set_nw_val(double nwval, int year_opt, int trig_opt, int inw) {

   nw_comp_matrix[year_opt][trig_opt][inw] = nwval;

 }

__global__ void compute_nwcov(double *masterevarray, int numofevts, double nwcovout[][336]) {

   int nwcovlinindex = threadIdx.x + blockDim.x * blockIdx.x;
   if (nwcovlinindex >= 336*336) { return;}

   int inw = nwcovlinindex / 336;
   int jnw = nwcovlinindex % 336;

   double sumi = 0;
   double sumj = 0;
   double sumij = 0;

   for( int kev = 0; kev < numofevts; kev++ ) {
      sumi += masterevarray[inw*numofevts+kev];
      sumj += masterevarray[jnw*numofevts+kev];
      sumij += masterevarray[inw*numofevts+kev]*masterevarray[jnw*numofevts+kev];
   }

   nwcovout[inw][jnw] = sumij-sumi*sumj/numofevts;

 }

__device__ double real_acc_mint(int imint, double ma, double mb) { 

   if (imint == 0) {return pycuda::real(Mj1j2(ma,mb,0,0)*pycuda::conj(Mj1j2(ma,mb,0,1)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 1) {return pycuda::real(Mj1j2(ma,mb,0,0)*pycuda::conj(Mj1j2(ma,mb,0,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 2) {return pycuda::real(Mj1j2(ma,mb,0,0)*pycuda::conj(Mj1j2(ma,mb,1,1)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 3) {return pycuda::real(Mj1j2(ma,mb,0,0)*pycuda::conj(Mj1j2(ma,mb,1,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 4) {return pycuda::real(Mj1j2(ma,mb,0,0)*pycuda::conj(Mj1j2(ma,mb,2,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 5) {return pycuda::real(Mj1j2(ma,mb,0,1)*pycuda::conj(Mj1j2(ma,mb,0,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 6) {return pycuda::real(Mj1j2(ma,mb,0,1)*pycuda::conj(Mj1j2(ma,mb,1,0)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 7) {return pycuda::real(Mj1j2(ma,mb,0,1)*pycuda::conj(Mj1j2(ma,mb,1,1)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 8) {return pycuda::real(Mj1j2(ma,mb,0,1)*pycuda::conj(Mj1j2(ma,mb,1,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 9) {return pycuda::real(Mj1j2(ma,mb,1,0)*pycuda::conj(Mj1j2(ma,mb,0,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 10) {return pycuda::real(Mj1j2(ma,mb,1,0)*pycuda::conj(Mj1j2(ma,mb,1,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 11) {return pycuda::real(Mj1j2(ma,mb,0,1)*pycuda::conj(Mj1j2(ma,mb,2,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 12) {return pycuda::real(Mj1j2(ma,mb,0,2)*pycuda::conj(Mj1j2(ma,mb,1,1)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 13) {return pycuda::real(Mj1j2(ma,mb,0,2)*pycuda::conj(Mj1j2(ma,mb,1,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 14) {return pycuda::real(Mj1j2(ma,mb,0,2)*pycuda::conj(Mj1j2(ma,mb,2,0)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 15) {return pycuda::real(Mj1j2(ma,mb,2,0)*pycuda::conj(Mj1j2(ma,mb,1,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 16) {return pycuda::real(Mj1j2(ma,mb,0,2)*pycuda::conj(Mj1j2(ma,mb,2,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 17) {return pycuda::real(Mj1j2(ma,mb,1,1)*pycuda::conj(Mj1j2(ma,mb,1,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 18) {return pycuda::real(Mj1j2(ma,mb,1,1)*pycuda::conj(Mj1j2(ma,mb,2,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 19) {return pycuda::real(Mj1j2(ma,mb,1,2)*pycuda::conj(Mj1j2(ma,mb,2,1)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 20) {return pycuda::real(Mj1j2(ma,mb,1,2)*pycuda::conj(Mj1j2(ma,mb,2,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 21) {return pycuda::real(Mj1j2(ma,mb,0,0)*pycuda::conj(Mj1j2(ma,mb,0,0)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 22) {return pycuda::real(Mj1j2(ma,mb,0,1)*pycuda::conj(Mj1j2(ma,mb,0,1)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 23) {return pycuda::real(Mj1j2(ma,mb,1,0)*pycuda::conj(Mj1j2(ma,mb,1,0)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 24) {return pycuda::real(Mj1j2(ma,mb,0,2)*pycuda::conj(Mj1j2(ma,mb,0,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 25) {return pycuda::real(Mj1j2(ma,mb,2,0)*pycuda::conj(Mj1j2(ma,mb,2,0)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 26) {return pycuda::real(Mj1j2(ma,mb,1,1)*pycuda::conj(Mj1j2(ma,mb,1,1)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 27) {return pycuda::real(Mj1j2(ma,mb,1,2)*pycuda::conj(Mj1j2(ma,mb,1,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 28) {return pycuda::real(Mj1j2(ma,mb,2,1)*pycuda::conj(Mj1j2(ma,mb,2,1)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 29) {return pycuda::real(Mj1j2(ma,mb,2,2)*pycuda::conj(Mj1j2(ma,mb,2,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   return 0.;

 }

__device__ double imag_acc_mint(int imint, double ma, double mb) { 

   if (imint == 0) {return pycuda::imag(Mj1j2(ma,mb,0,0)*pycuda::conj(Mj1j2(ma,mb,0,1)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 1) {return pycuda::imag(Mj1j2(ma,mb,0,0)*pycuda::conj(Mj1j2(ma,mb,0,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 2) {return pycuda::imag(Mj1j2(ma,mb,0,0)*pycuda::conj(Mj1j2(ma,mb,1,1)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 3) {return pycuda::imag(Mj1j2(ma,mb,0,0)*pycuda::conj(Mj1j2(ma,mb,1,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 4) {return pycuda::imag(Mj1j2(ma,mb,0,0)*pycuda::conj(Mj1j2(ma,mb,2,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 5) {return pycuda::imag(Mj1j2(ma,mb,0,1)*pycuda::conj(Mj1j2(ma,mb,0,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 6) {return pycuda::imag(Mj1j2(ma,mb,0,1)*pycuda::conj(Mj1j2(ma,mb,1,0)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 7) {return pycuda::imag(Mj1j2(ma,mb,0,1)*pycuda::conj(Mj1j2(ma,mb,1,1)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 8) {return pycuda::imag(Mj1j2(ma,mb,0,1)*pycuda::conj(Mj1j2(ma,mb,1,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 9) {return pycuda::imag(Mj1j2(ma,mb,1,0)*pycuda::conj(Mj1j2(ma,mb,0,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 10) {return pycuda::imag(Mj1j2(ma,mb,1,0)*pycuda::conj(Mj1j2(ma,mb,1,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 11) {return pycuda::imag(Mj1j2(ma,mb,0,1)*pycuda::conj(Mj1j2(ma,mb,2,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 12) {return pycuda::imag(Mj1j2(ma,mb,0,2)*pycuda::conj(Mj1j2(ma,mb,1,1)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 13) {return pycuda::imag(Mj1j2(ma,mb,0,2)*pycuda::conj(Mj1j2(ma,mb,1,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 14) {return pycuda::imag(Mj1j2(ma,mb,0,2)*pycuda::conj(Mj1j2(ma,mb,2,0)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 15) {return pycuda::imag(Mj1j2(ma,mb,2,0)*pycuda::conj(Mj1j2(ma,mb,1,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 16) {return pycuda::imag(Mj1j2(ma,mb,0,2)*pycuda::conj(Mj1j2(ma,mb,2,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 17) {return pycuda::imag(Mj1j2(ma,mb,1,1)*pycuda::conj(Mj1j2(ma,mb,1,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 18) {return pycuda::imag(Mj1j2(ma,mb,1,1)*pycuda::conj(Mj1j2(ma,mb,2,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 19) {return pycuda::imag(Mj1j2(ma,mb,1,2)*pycuda::conj(Mj1j2(ma,mb,2,1)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   else if (imint == 20) {return pycuda::imag(Mj1j2(ma,mb,1,2)*pycuda::conj(Mj1j2(ma,mb,2,2)))*phasespace(ma,mb)*accGenMass(ma)*accGenMass(mb);}
   return 0.;

 }

__global__ void compute_acc_mint_array(int imint, int part, int mintnpoints, double minthlimit, double *mpdfarray) {

   int mintindex = threadIdx.x + blockDim.x * blockIdx.x;
   if (mintindex >= mintnpoints*mintnpoints) { return;}

   int im1 = mintindex / mintnpoints;
   int im2 = mintindex % mintnpoints;

   double mintstep = (minthlimit-750.)/mintnpoints;
   double m1_ = 750.+im1*mintstep;
   double m2_ = 750.+im2*mintstep;
   double m1next_ = 750.+(im1+1)*mintstep;
   double m2next_ = 750.+(im2+1)*mintstep;

   double point1 = 0.;
   double point2 = 0.;
   double point3 = 0.;
   double point4 = 0.;

   if (part == 0) {
      point1 = real_acc_mint(imint,m1_,m2_);
      point2 = real_acc_mint(imint,m1next_,m2_);
      point3 = real_acc_mint(imint,m1_,m2next_);
      point4 = real_acc_mint(imint,m1next_,m2next_);
   }
   else {
      point1 = imag_acc_mint(imint,m1_,m2_);
      point2 = imag_acc_mint(imint,m1next_,m2_);
      point3 = imag_acc_mint(imint,m1_,m2next_);
      point4 = imag_acc_mint(imint,m1next_,m2next_);
   }

   mpdfarray[mintindex] = 0.25*(point1+point2+point3+point4)*mintstep*mintstep;

 }

__global__ void set_all_mass_integrals(double *mass_integrals) {

   Im00 = mass_integrals[0];
   Im01 = mass_integrals[1];
   Im10 = mass_integrals[2];
   Im02 = mass_integrals[3];
   Im20 = mass_integrals[4];
   Im11 = mass_integrals[5];
   Im12 = mass_integrals[6];
   Im21 = mass_integrals[7];
   Im22 = mass_integrals[8];

   Ih1Re = mass_integrals[9];
   Ih2Re = mass_integrals[10];
   Ih3Re = mass_integrals[11];
   Ih4Re = mass_integrals[12];
   Ih5Re = mass_integrals[13];
   Ih6Re = mass_integrals[14];
   Ih7Re = mass_integrals[15];
   Ih8Re = mass_integrals[16];
   Ih9Re = mass_integrals[17];
   Ih10Re = mass_integrals[18];
   Ih11Re = mass_integrals[19];
   Ih12Re = mass_integrals[20];
   Ih13Re = mass_integrals[21];
   Ih14Re = mass_integrals[22];
   Ih15Re = mass_integrals[23];
   Ih16Re = mass_integrals[24];
   Ih17Re = mass_integrals[25];
   Ih18Re = mass_integrals[26];
   Ih19Re = mass_integrals[27];
   Ih20Re = mass_integrals[28];
   Ih21Re = mass_integrals[29];
   Ih22Re = mass_integrals[30];
   Ih23Re = mass_integrals[31];
   Ih24Re = mass_integrals[32];
   Ih25Re = mass_integrals[33];
   Ih26Re = mass_integrals[34];
   Ih27Re = mass_integrals[35];
   Ih28Re = mass_integrals[36];
   Ih29Re = mass_integrals[37];
   Ih30Re = mass_integrals[38];

   Ih1Im = mass_integrals[39];
   Ih2Im = mass_integrals[40];
   Ih3Im = mass_integrals[41];
   Ih4Im = mass_integrals[42];
   Ih5Im = mass_integrals[43];
   Ih6Im = mass_integrals[44];
   Ih7Im = mass_integrals[45];
   Ih8Im = mass_integrals[46];
   Ih9Im = mass_integrals[47];
   Ih10Im = mass_integrals[48];
   Ih11Im = mass_integrals[49];
   Ih12Im = mass_integrals[50];
   Ih13Im = mass_integrals[51];
   Ih14Im = mass_integrals[52];
   Ih15Im = mass_integrals[53];
   Ih16Im = mass_integrals[54];
   Ih17Im = mass_integrals[55];
   Ih18Im = mass_integrals[56];
   Ih19Im = mass_integrals[57];
   Ih20Im = mass_integrals[58];
   Ih21Im = mass_integrals[59];

   reIhj1j2j1pj2pdict[0][0][0][0] = pycuda::real(pycuda::complex<double>(Ih22Re,0.));
   reIhj1j2j1pj2pdict[0][0][0][1] = pycuda::real(pycuda::complex<double>(Ih1Re,Ih1Im));
   reIhj1j2j1pj2pdict[0][0][0][2] = pycuda::real(pycuda::complex<double>(Ih2Re,Ih2Im));
   reIhj1j2j1pj2pdict[0][0][1][0] = pycuda::real(pycuda::complex<double>(Ih1Re,Ih1Im));
   reIhj1j2j1pj2pdict[0][0][1][1] = pycuda::real(pycuda::complex<double>(Ih3Re,Ih3Im));
   reIhj1j2j1pj2pdict[0][0][1][2] = pycuda::real(pycuda::complex<double>(Ih4Re,Ih4Im));
   reIhj1j2j1pj2pdict[0][0][2][0] = pycuda::real(pycuda::complex<double>(Ih2Re,Ih2Im));
   reIhj1j2j1pj2pdict[0][0][2][1] = pycuda::real(pycuda::complex<double>(Ih4Re,Ih4Im));
   reIhj1j2j1pj2pdict[0][0][2][2] = pycuda::real(pycuda::complex<double>(Ih5Re,Ih5Im));
   reIhj1j2j1pj2pdict[0][1][0][0] = pycuda::real(pycuda::complex<double>(Ih1Re,-Ih1Im));
   reIhj1j2j1pj2pdict[0][1][0][1] = pycuda::real(pycuda::complex<double>(Ih23Re,0.));
   reIhj1j2j1pj2pdict[0][1][0][2] = pycuda::real(pycuda::complex<double>(Ih6Re,Ih6Im));
   reIhj1j2j1pj2pdict[0][1][1][0] = pycuda::real(pycuda::complex<double>(Ih7Re,Ih7Im));
   reIhj1j2j1pj2pdict[0][1][1][1] = pycuda::real(pycuda::complex<double>(Ih8Re,Ih8Im));
   reIhj1j2j1pj2pdict[0][1][1][2] = pycuda::real(pycuda::complex<double>(Ih9Re,Ih9Im));
   reIhj1j2j1pj2pdict[0][1][2][0] = pycuda::real(pycuda::complex<double>(Ih10Re,Ih10Im));
   reIhj1j2j1pj2pdict[0][1][2][1] = pycuda::real(pycuda::complex<double>(Ih11Re,Ih11Im));
   reIhj1j2j1pj2pdict[0][1][2][2] = pycuda::real(pycuda::complex<double>(Ih12Re,Ih12Im));
   reIhj1j2j1pj2pdict[0][2][0][0] = pycuda::real(pycuda::complex<double>(Ih2Re,-Ih2Im));
   reIhj1j2j1pj2pdict[0][2][0][1] = pycuda::real(pycuda::complex<double>(Ih6Re,-Ih6Im));
   reIhj1j2j1pj2pdict[0][2][0][2] = pycuda::real(pycuda::complex<double>(Ih25Re,0.));
   reIhj1j2j1pj2pdict[0][2][1][0] = pycuda::real(pycuda::complex<double>(Ih10Re,-Ih10Im));
   reIhj1j2j1pj2pdict[0][2][1][1] = pycuda::real(pycuda::complex<double>(Ih13Re,Ih13Im));
   reIhj1j2j1pj2pdict[0][2][1][2] = pycuda::real(pycuda::complex<double>(Ih14Re,Ih14Im));
   reIhj1j2j1pj2pdict[0][2][2][0] = pycuda::real(pycuda::complex<double>(Ih15Re,Ih15Im));
   reIhj1j2j1pj2pdict[0][2][2][1] = pycuda::real(pycuda::complex<double>(Ih16Re,Ih16Im));
   reIhj1j2j1pj2pdict[0][2][2][2] = pycuda::real(pycuda::complex<double>(Ih17Re,Ih17Im));
   reIhj1j2j1pj2pdict[1][0][0][0] = pycuda::real(pycuda::complex<double>(Ih1Re,-Ih1Im));
   reIhj1j2j1pj2pdict[1][0][0][1] = pycuda::real(pycuda::complex<double>(Ih7Re,Ih7Im));
   reIhj1j2j1pj2pdict[1][0][0][2] = pycuda::real(pycuda::complex<double>(Ih10Re,Ih10Im));
   reIhj1j2j1pj2pdict[1][0][1][0] = pycuda::real(pycuda::complex<double>(Ih24Re,0.));
   reIhj1j2j1pj2pdict[1][0][1][1] = pycuda::real(pycuda::complex<double>(Ih8Re,Ih8Im));
   reIhj1j2j1pj2pdict[1][0][1][2] = pycuda::real(pycuda::complex<double>(Ih11Re,Ih11Im));
   reIhj1j2j1pj2pdict[1][0][2][0] = pycuda::real(pycuda::complex<double>(Ih6Re,Ih6Im));
   reIhj1j2j1pj2pdict[1][0][2][1] = pycuda::real(pycuda::complex<double>(Ih9Re,Ih9Im));
   reIhj1j2j1pj2pdict[1][0][2][2] = pycuda::real(pycuda::complex<double>(Ih12Re,Ih12Im));
   reIhj1j2j1pj2pdict[1][1][0][0] = pycuda::real(pycuda::complex<double>(Ih3Re,-Ih3Im));
   reIhj1j2j1pj2pdict[1][1][0][1] = pycuda::real(pycuda::complex<double>(Ih8Re,-Ih8Im));
   reIhj1j2j1pj2pdict[1][1][0][2] = pycuda::real(pycuda::complex<double>(Ih13Re,-Ih13Im));
   reIhj1j2j1pj2pdict[1][1][1][0] = pycuda::real(pycuda::complex<double>(Ih8Re,-Ih8Im));
   reIhj1j2j1pj2pdict[1][1][1][1] = pycuda::real(pycuda::complex<double>(Ih27Re,0.));
   reIhj1j2j1pj2pdict[1][1][1][2] = pycuda::real(pycuda::complex<double>(Ih18Re,Ih18Im));
   reIhj1j2j1pj2pdict[1][1][2][0] = pycuda::real(pycuda::complex<double>(Ih13Re,-Ih13Im));
   reIhj1j2j1pj2pdict[1][1][2][1] = pycuda::real(pycuda::complex<double>(Ih18Re,Ih18Im));
   reIhj1j2j1pj2pdict[1][1][2][2] = pycuda::real(pycuda::complex<double>(Ih19Re,Ih19Im));
   reIhj1j2j1pj2pdict[1][2][0][0] = pycuda::real(pycuda::complex<double>(Ih4Re,-Ih4Im));
   reIhj1j2j1pj2pdict[1][2][0][1] = pycuda::real(pycuda::complex<double>(Ih9Re,-Ih9Im));
   reIhj1j2j1pj2pdict[1][2][0][2] = pycuda::real(pycuda::complex<double>(Ih14Re,-Ih14Im));
   reIhj1j2j1pj2pdict[1][2][1][0] = pycuda::real(pycuda::complex<double>(Ih11Re,-Ih11Im));
   reIhj1j2j1pj2pdict[1][2][1][1] = pycuda::real(pycuda::complex<double>(Ih18Re,-Ih18Im));
   reIhj1j2j1pj2pdict[1][2][1][2] = pycuda::real(pycuda::complex<double>(Ih28Re,0.));
   reIhj1j2j1pj2pdict[1][2][2][0] = pycuda::real(pycuda::complex<double>(Ih16Re,-Ih16Im));
   reIhj1j2j1pj2pdict[1][2][2][1] = pycuda::real(pycuda::complex<double>(Ih20Re,Ih20Im));
   reIhj1j2j1pj2pdict[1][2][2][2] = pycuda::real(pycuda::complex<double>(Ih21Re,Ih21Im));
   reIhj1j2j1pj2pdict[2][0][0][0] = pycuda::real(pycuda::complex<double>(Ih2Re,-Ih2Im));
   reIhj1j2j1pj2pdict[2][0][0][1] = pycuda::real(pycuda::complex<double>(Ih10Re,-Ih10Im));
   reIhj1j2j1pj2pdict[2][0][0][2] = pycuda::real(pycuda::complex<double>(Ih15Re,Ih15Im));
   reIhj1j2j1pj2pdict[2][0][1][0] = pycuda::real(pycuda::complex<double>(Ih6Re,-Ih6Im));
   reIhj1j2j1pj2pdict[2][0][1][1] = pycuda::real(pycuda::complex<double>(Ih13Re,Ih13Im));
   reIhj1j2j1pj2pdict[2][0][1][2] = pycuda::real(pycuda::complex<double>(Ih16Re,Ih16Im));
   reIhj1j2j1pj2pdict[2][0][2][0] = pycuda::real(pycuda::complex<double>(Ih26Re,0.));
   reIhj1j2j1pj2pdict[2][0][2][1] = pycuda::real(pycuda::complex<double>(Ih14Re,Ih14Im));
   reIhj1j2j1pj2pdict[2][0][2][2] = pycuda::real(pycuda::complex<double>(Ih17Re,Ih17Im));
   reIhj1j2j1pj2pdict[2][1][0][0] = pycuda::real(pycuda::complex<double>(Ih4Re,-Ih4Im));
   reIhj1j2j1pj2pdict[2][1][0][1] = pycuda::real(pycuda::complex<double>(Ih11Re,-Ih11Im));
   reIhj1j2j1pj2pdict[2][1][0][2] = pycuda::real(pycuda::complex<double>(Ih16Re,-Ih16Im));
   reIhj1j2j1pj2pdict[2][1][1][0] = pycuda::real(pycuda::complex<double>(Ih9Re,-Ih9Im));
   reIhj1j2j1pj2pdict[2][1][1][1] = pycuda::real(pycuda::complex<double>(Ih18Re,-Ih18Im));
   reIhj1j2j1pj2pdict[2][1][1][2] = pycuda::real(pycuda::complex<double>(Ih20Re,Ih20Im));
   reIhj1j2j1pj2pdict[2][1][2][0] = pycuda::real(pycuda::complex<double>(Ih14Re,-Ih14Im));
   reIhj1j2j1pj2pdict[2][1][2][1] = pycuda::real(pycuda::complex<double>(Ih29Re,0.));
   reIhj1j2j1pj2pdict[2][1][2][2] = pycuda::real(pycuda::complex<double>(Ih21Re,Ih21Im));
   reIhj1j2j1pj2pdict[2][2][0][0] = pycuda::real(pycuda::complex<double>(Ih5Re,-Ih5Im));
   reIhj1j2j1pj2pdict[2][2][0][1] = pycuda::real(pycuda::complex<double>(Ih12Re,-Ih12Im));
   reIhj1j2j1pj2pdict[2][2][0][2] = pycuda::real(pycuda::complex<double>(Ih17Re,-Ih17Im));
   reIhj1j2j1pj2pdict[2][2][1][0] = pycuda::real(pycuda::complex<double>(Ih12Re,-Ih12Im));
   reIhj1j2j1pj2pdict[2][2][1][1] = pycuda::real(pycuda::complex<double>(Ih19Re,-Ih19Im));
   reIhj1j2j1pj2pdict[2][2][1][2] = pycuda::real(pycuda::complex<double>(Ih21Re,-Ih21Im));
   reIhj1j2j1pj2pdict[2][2][2][0] = pycuda::real(pycuda::complex<double>(Ih17Re,-Ih17Im));
   reIhj1j2j1pj2pdict[2][2][2][1] = pycuda::real(pycuda::complex<double>(Ih21Re,-Ih21Im));
   reIhj1j2j1pj2pdict[2][2][2][2] = pycuda::real(pycuda::complex<double>(Ih30Re,0.));
   imIhj1j2j1pj2pdict[0][0][0][0] = pycuda::imag(pycuda::complex<double>(Ih22Re,0.));
   imIhj1j2j1pj2pdict[0][0][0][1] = pycuda::imag(pycuda::complex<double>(Ih1Re,Ih1Im));
   imIhj1j2j1pj2pdict[0][0][0][2] = pycuda::imag(pycuda::complex<double>(Ih2Re,Ih2Im));
   imIhj1j2j1pj2pdict[0][0][1][0] = pycuda::imag(pycuda::complex<double>(Ih1Re,Ih1Im));
   imIhj1j2j1pj2pdict[0][0][1][1] = pycuda::imag(pycuda::complex<double>(Ih3Re,Ih3Im));
   imIhj1j2j1pj2pdict[0][0][1][2] = pycuda::imag(pycuda::complex<double>(Ih4Re,Ih4Im));
   imIhj1j2j1pj2pdict[0][0][2][0] = pycuda::imag(pycuda::complex<double>(Ih2Re,Ih2Im));
   imIhj1j2j1pj2pdict[0][0][2][1] = pycuda::imag(pycuda::complex<double>(Ih4Re,Ih4Im));
   imIhj1j2j1pj2pdict[0][0][2][2] = pycuda::imag(pycuda::complex<double>(Ih5Re,Ih5Im));
   imIhj1j2j1pj2pdict[0][1][0][0] = pycuda::imag(pycuda::complex<double>(Ih1Re,-Ih1Im));
   imIhj1j2j1pj2pdict[0][1][0][1] = pycuda::imag(pycuda::complex<double>(Ih23Re,0.));
   imIhj1j2j1pj2pdict[0][1][0][2] = pycuda::imag(pycuda::complex<double>(Ih6Re,Ih6Im));
   imIhj1j2j1pj2pdict[0][1][1][0] = pycuda::imag(pycuda::complex<double>(Ih7Re,Ih7Im));
   imIhj1j2j1pj2pdict[0][1][1][1] = pycuda::imag(pycuda::complex<double>(Ih8Re,Ih8Im));
   imIhj1j2j1pj2pdict[0][1][1][2] = pycuda::imag(pycuda::complex<double>(Ih9Re,Ih9Im));
   imIhj1j2j1pj2pdict[0][1][2][0] = pycuda::imag(pycuda::complex<double>(Ih10Re,Ih10Im));
   imIhj1j2j1pj2pdict[0][1][2][1] = pycuda::imag(pycuda::complex<double>(Ih11Re,Ih11Im));
   imIhj1j2j1pj2pdict[0][1][2][2] = pycuda::imag(pycuda::complex<double>(Ih12Re,Ih12Im));
   imIhj1j2j1pj2pdict[0][2][0][0] = pycuda::imag(pycuda::complex<double>(Ih2Re,-Ih2Im));
   imIhj1j2j1pj2pdict[0][2][0][1] = pycuda::imag(pycuda::complex<double>(Ih6Re,-Ih6Im));
   imIhj1j2j1pj2pdict[0][2][0][2] = pycuda::imag(pycuda::complex<double>(Ih25Re,0.));
   imIhj1j2j1pj2pdict[0][2][1][0] = pycuda::imag(pycuda::complex<double>(Ih10Re,-Ih10Im));
   imIhj1j2j1pj2pdict[0][2][1][1] = pycuda::imag(pycuda::complex<double>(Ih13Re,Ih13Im));
   imIhj1j2j1pj2pdict[0][2][1][2] = pycuda::imag(pycuda::complex<double>(Ih14Re,Ih14Im));
   imIhj1j2j1pj2pdict[0][2][2][0] = pycuda::imag(pycuda::complex<double>(Ih15Re,Ih15Im));
   imIhj1j2j1pj2pdict[0][2][2][1] = pycuda::imag(pycuda::complex<double>(Ih16Re,Ih16Im));
   imIhj1j2j1pj2pdict[0][2][2][2] = pycuda::imag(pycuda::complex<double>(Ih17Re,Ih17Im));
   imIhj1j2j1pj2pdict[1][0][0][0] = pycuda::imag(pycuda::complex<double>(Ih1Re,-Ih1Im));
   imIhj1j2j1pj2pdict[1][0][0][1] = pycuda::imag(pycuda::complex<double>(Ih7Re,Ih7Im));
   imIhj1j2j1pj2pdict[1][0][0][2] = pycuda::imag(pycuda::complex<double>(Ih10Re,Ih10Im));
   imIhj1j2j1pj2pdict[1][0][1][0] = pycuda::imag(pycuda::complex<double>(Ih24Re,0.));
   imIhj1j2j1pj2pdict[1][0][1][1] = pycuda::imag(pycuda::complex<double>(Ih8Re,Ih8Im));
   imIhj1j2j1pj2pdict[1][0][1][2] = pycuda::imag(pycuda::complex<double>(Ih11Re,Ih11Im));
   imIhj1j2j1pj2pdict[1][0][2][0] = pycuda::imag(pycuda::complex<double>(Ih6Re,Ih6Im));
   imIhj1j2j1pj2pdict[1][0][2][1] = pycuda::imag(pycuda::complex<double>(Ih9Re,Ih9Im));
   imIhj1j2j1pj2pdict[1][0][2][2] = pycuda::imag(pycuda::complex<double>(Ih12Re,Ih12Im));
   imIhj1j2j1pj2pdict[1][1][0][0] = pycuda::imag(pycuda::complex<double>(Ih3Re,-Ih3Im));
   imIhj1j2j1pj2pdict[1][1][0][1] = pycuda::imag(pycuda::complex<double>(Ih8Re,-Ih8Im));
   imIhj1j2j1pj2pdict[1][1][0][2] = pycuda::imag(pycuda::complex<double>(Ih13Re,-Ih13Im));
   imIhj1j2j1pj2pdict[1][1][1][0] = pycuda::imag(pycuda::complex<double>(Ih8Re,-Ih8Im));
   imIhj1j2j1pj2pdict[1][1][1][1] = pycuda::imag(pycuda::complex<double>(Ih27Re,0.));
   imIhj1j2j1pj2pdict[1][1][1][2] = pycuda::imag(pycuda::complex<double>(Ih18Re,Ih18Im));
   imIhj1j2j1pj2pdict[1][1][2][0] = pycuda::imag(pycuda::complex<double>(Ih13Re,-Ih13Im));
   imIhj1j2j1pj2pdict[1][1][2][1] = pycuda::imag(pycuda::complex<double>(Ih18Re,Ih18Im));
   imIhj1j2j1pj2pdict[1][1][2][2] = pycuda::imag(pycuda::complex<double>(Ih19Re,Ih19Im));
   imIhj1j2j1pj2pdict[1][2][0][0] = pycuda::imag(pycuda::complex<double>(Ih4Re,-Ih4Im));
   imIhj1j2j1pj2pdict[1][2][0][1] = pycuda::imag(pycuda::complex<double>(Ih9Re,-Ih9Im));
   imIhj1j2j1pj2pdict[1][2][0][2] = pycuda::imag(pycuda::complex<double>(Ih14Re,-Ih14Im));
   imIhj1j2j1pj2pdict[1][2][1][0] = pycuda::imag(pycuda::complex<double>(Ih11Re,-Ih11Im));
   imIhj1j2j1pj2pdict[1][2][1][1] = pycuda::imag(pycuda::complex<double>(Ih18Re,-Ih18Im));
   imIhj1j2j1pj2pdict[1][2][1][2] = pycuda::imag(pycuda::complex<double>(Ih28Re,0.));
   imIhj1j2j1pj2pdict[1][2][2][0] = pycuda::imag(pycuda::complex<double>(Ih16Re,-Ih16Im));
   imIhj1j2j1pj2pdict[1][2][2][1] = pycuda::imag(pycuda::complex<double>(Ih20Re,Ih20Im));
   imIhj1j2j1pj2pdict[1][2][2][2] = pycuda::imag(pycuda::complex<double>(Ih21Re,Ih21Im));
   imIhj1j2j1pj2pdict[2][0][0][0] = pycuda::imag(pycuda::complex<double>(Ih2Re,-Ih2Im));
   imIhj1j2j1pj2pdict[2][0][0][1] = pycuda::imag(pycuda::complex<double>(Ih10Re,-Ih10Im));
   imIhj1j2j1pj2pdict[2][0][0][2] = pycuda::imag(pycuda::complex<double>(Ih15Re,Ih15Im));
   imIhj1j2j1pj2pdict[2][0][1][0] = pycuda::imag(pycuda::complex<double>(Ih6Re,-Ih6Im));
   imIhj1j2j1pj2pdict[2][0][1][1] = pycuda::imag(pycuda::complex<double>(Ih13Re,Ih13Im));
   imIhj1j2j1pj2pdict[2][0][1][2] = pycuda::imag(pycuda::complex<double>(Ih16Re,Ih16Im));
   imIhj1j2j1pj2pdict[2][0][2][0] = pycuda::imag(pycuda::complex<double>(Ih26Re,0.));
   imIhj1j2j1pj2pdict[2][0][2][1] = pycuda::imag(pycuda::complex<double>(Ih14Re,Ih14Im));
   imIhj1j2j1pj2pdict[2][0][2][2] = pycuda::imag(pycuda::complex<double>(Ih17Re,Ih17Im));
   imIhj1j2j1pj2pdict[2][1][0][0] = pycuda::imag(pycuda::complex<double>(Ih4Re,-Ih4Im));
   imIhj1j2j1pj2pdict[2][1][0][1] = pycuda::imag(pycuda::complex<double>(Ih11Re,-Ih11Im));
   imIhj1j2j1pj2pdict[2][1][0][2] = pycuda::imag(pycuda::complex<double>(Ih16Re,-Ih16Im));
   imIhj1j2j1pj2pdict[2][1][1][0] = pycuda::imag(pycuda::complex<double>(Ih9Re,-Ih9Im));
   imIhj1j2j1pj2pdict[2][1][1][1] = pycuda::imag(pycuda::complex<double>(Ih18Re,-Ih18Im));
   imIhj1j2j1pj2pdict[2][1][1][2] = pycuda::imag(pycuda::complex<double>(Ih20Re,Ih20Im));
   imIhj1j2j1pj2pdict[2][1][2][0] = pycuda::imag(pycuda::complex<double>(Ih14Re,-Ih14Im));
   imIhj1j2j1pj2pdict[2][1][2][1] = pycuda::imag(pycuda::complex<double>(Ih29Re,0.));
   imIhj1j2j1pj2pdict[2][1][2][2] = pycuda::imag(pycuda::complex<double>(Ih21Re,Ih21Im));
   imIhj1j2j1pj2pdict[2][2][0][0] = pycuda::imag(pycuda::complex<double>(Ih5Re,-Ih5Im));
   imIhj1j2j1pj2pdict[2][2][0][1] = pycuda::imag(pycuda::complex<double>(Ih12Re,-Ih12Im));
   imIhj1j2j1pj2pdict[2][2][0][2] = pycuda::imag(pycuda::complex<double>(Ih17Re,-Ih17Im));
   imIhj1j2j1pj2pdict[2][2][1][0] = pycuda::imag(pycuda::complex<double>(Ih12Re,-Ih12Im));
   imIhj1j2j1pj2pdict[2][2][1][1] = pycuda::imag(pycuda::complex<double>(Ih19Re,-Ih19Im));
   imIhj1j2j1pj2pdict[2][2][1][2] = pycuda::imag(pycuda::complex<double>(Ih21Re,-Ih21Im));
   imIhj1j2j1pj2pdict[2][2][2][0] = pycuda::imag(pycuda::complex<double>(Ih17Re,-Ih17Im));
   imIhj1j2j1pj2pdict[2][2][2][1] = pycuda::imag(pycuda::complex<double>(Ih21Re,-Ih21Im));
   imIhj1j2j1pj2pdict[2][2][2][2] = pycuda::imag(pycuda::complex<double>(Ih30Re,0.));

 }

}
